#include "hip/hip_runtime.h"
//
//  KaspyCycler.cpp
//  kaspy_cuda
//
//  Created by Andrei Koulikov on 24.05.16.
//
//

#include "KaspyCycler.h"

void getbicubic(int nx, int ny, int nd, float * z, float * c);
void bcucof(float * y,float * y1,float * y2, float * y12,float d1,float d2,float * cc);




float * g_fbu;
float * g_fbv;
float * g_ffu;
float * g_ffv;

float * g_fxb;
float * g_fxf;
float * g_fyb;
float * g_fyf;

float * g_fb;
float * g_ff;

float * g_wusurf;
float * g_wvsurf;

float * g_dum;
float * g_dvm;

float * g_d;
float * g_dx;
float * g_dy;

float * g_fluxua;
float * g_fluxva;

float * g_ua;
float * g_va;

float * g_uab;
float * g_vab;

float * g_uaf;
float * g_vaf;


float * g_el;
float * g_elf;
float * g_elb;

float * g_fsm;

float * g_tps;


float * g_advua;
float * g_advva;

float * g_aru;
float * g_arv;

float * g_wubot;
float * g_wvbot;
float * g_cbc;

void KaspyCycler::findElves()
{
    //printf("arrays is set to %llxd \n", (long long)m_fArrays );
    
    
    float * elves = &(m_fArrays->elf[0][0]);
    
    float elf_min = elves[0];
    float elf_max = elves[0];
    
    for (int i=1; i<F_DATA_SIZE; i++)
    {
        if (elves[i] > elf_max)
        {
            elf_max = elves[i];
        }
        
        if (elves[i] < elf_min)
        {
            elf_min = elves[i];
        }
    }
    
    //printf("C SAYS: time is %f, elf min is %f, elf max is %f \n",m_fVars->timeh, elf_min, elf_max);
}

void KaspyCycler::sendDataToGPU()
{
    g_fbu = &m_fFloats->fbu[0][0];
    g_fbv = &m_fFloats->fbv[0][0];
    g_ffu = &m_fFloats->ffu[0][0];
    g_ffv = &m_fFloats->ffv[0][0];
    
    g_fxb = &m_fFloats->fxb[0][0];
    g_fxf = &m_fFloats->fxf[0][0];
    g_fyb = &m_fFloats->fyb[0][0];
    g_fyf = &m_fFloats->fyf[0][0];
    
    
    g_fb = &m_fFloats->fb[0][0];
    g_ff = &m_fFloats->ff[0][0];
    
    
    g_wusurf = &m_fArrays->wusurf[0][0];
    g_wvsurf = &m_fArrays->wvsurf[0][0];
    
    g_dum = &m_fArrays->dum[0][0];
    g_dvm = &m_fArrays->dvm[0][0];
    
    g_d = &m_fArrays->d[0][0];
    g_dx = &m_fArrays->dx[0];
    g_dy = &m_fArrays->dy[0];


    g_fluxua = &m_fArrays->fluxua[0][0];
    g_fluxva = &m_fArrays->fluxva[0][0];
	
	g_advua = &m_fArrays->advua[0][0];
	g_advva = &m_fArrays->advua[0][0];
	
    g_ua = &m_fArrays->ua[0][0];
    g_va = &m_fArrays->va[0][0];

	g_uab = &m_fArrays->uab[0][0];
	g_vab = &m_fArrays->vab[0][0];

	g_uaf = &m_fArrays->uaf[0][0];
	g_vaf = &m_fArrays->vaf[0][0];
	
	
    g_el = &m_fArrays->el[0][0];
    g_elf = &m_fArrays->elf[0][0];
    g_elb = &m_fArrays->elb[0][0];
	
	g_fsm = &m_fArrays->fsm[0][0];
	
	g_tps = &m_fArrays->tps[0][0];
	
	g_aru = &m_fArrays->aru[0];
	g_arv = &m_fArrays->arv[0];

	
	g_wubot = &m_fArrays->wubot[0][0];
	g_wvbot = &m_fArrays->wvbot[0][0];

	g_cbc = &m_fArrays->cbc[0][0];
}

void KaspyCycler::getDataToCPU()
{
    
}




void KaspyCycler::makeWsurf(float ro_ratio)
{
    m_fVars->timeh6 = (m_fVars->timeh / m_fVars->dht) + 1.0f;

    float timeh6 = m_fVars->timeh6;
    
    int pressSize = m_fWindData->kx * m_fWindData->ky;
    int windUSize = m_fWindData->kxu * m_fWindData->kyu;
    int windVSize = m_fWindData->kxv * m_fWindData->kyv;
    
    itime6 = (int)timeh6;

    ftim = (timeh6 - itime6);
    btim = 1.0f - ftim;
    
    if (itime6 > itime6_old)
    {
        itime6_old = itime6;
        
        memcpy(g_fxb, g_fxf, F_DATA_SIZE * sizeof(float));
        memcpy(g_fyb, g_fyf, F_DATA_SIZE * sizeof(float));
        memcpy(g_fb, g_ff, F_DATA_SIZE * sizeof(float));
        memcpy(g_fbu, g_ffu, F_DATA_SIZE * sizeof(float));
        memcpy(g_fbv, g_ffv, F_DATA_SIZE * sizeof(float));



        memcpy(m_press0, m_press + (itime6 - 1) * pressSize, pressSize * sizeof(float));
		
		getWindPressure('p');

		
        memcpy(m_uwd0, m_uwd + (itime6 - 1) * windUSize, windUSize * sizeof(float));
		
		
		getWindPressure('u');

        memcpy(m_vwd0, m_vwd + (itime6 - 1) * windVSize, windVSize * sizeof(float));
		
		getWindPressure('v');

    }
	
	
    float uw, vw, speed, windc;
    int ji, jp1i, jip1, jim1, jm1i, jp1ip1, jm1im1, jp1im1, jm1ip1;


    
    ftim = fmodf((float)m_fVars->timeh6, 1.0f);
    btim = 1.0f - ftim;
    
    for (int j=1; j<m_height; j++ )
    {
        for (int i=1; i<m_width; i++ )
        {
            if ((j<(m_height-1)) && i<(m_width-1))
            {
                ji = j * m_width + i;
                jp1i = ji + m_width;
                jip1 = ji + 1;
                jim1 = ji - 1;
                jm1i = ji - m_width;
                
                uw = btim * (g_fbu[ji]) + ftim * (g_ffu[ji]);
                vw = btim * (g_fbv[ji]) + ftim * (g_ffv[ji]);
                
                speed = sqrtf(uw*uw + vw*vw);
                windc = 0.001f * (0.8f + speed * 0.065f) * ro_ratio * speed;
                
                g_wusurf[ji] = -windc * uw *
                0.25f * (g_dum[jp1i]+g_dum[jip1]+g_dum[jim1]+g_dum[jm1i])
                + 0.5f * (g_d[ji] + g_d[jim1]) * (btim * g_fxb[ji] + ftim * g_fxf[ji]);
                
                g_wvsurf[ji] = -windc * vw *
                0.25f * (g_dvm[jp1i]+g_dvm[jip1]+g_dvm[jim1]+g_dvm[jm1i])
                + 0.5f * (g_d[ji] + g_d[jm1i]) * (btim * g_fyb[ji] + ftim * g_fyf[ji]);
            }
            


            g_fluxua[ji] = 0.25f * (g_d[ji] + g_d[jim1]) * (g_dy[j] + g_dy[j] ) * g_ua[ji];
            g_fluxva[ji] = 0.25f * (g_d[ji] + g_d[jm1i]) * (g_dx[j] + g_dx[j-1] ) * g_va[ji];
            
        }
    }
    
    
    /// HERE SHOULD START A NEW CUDA CALL TO KEEP fluxua fluxva synced
	
    
    float dte2 = m_fVars->dte * 2.0f;
    
    for (int j=1; j<(m_height-1); j++ )
    {
        float artj = m_fArrays->art[j];
        
        for (int i=1; i<(m_width-1); i++ )
        {
            ji = j * m_width + i;
            jp1i = ji + m_width;
            jip1 = ji + 1;
            
            g_elf[ji] = g_elb[ji] - dte2 *
                (g_fluxua[jip1] - g_fluxua[ji] + g_fluxva[jp1i] - g_fluxva[ji]) /  artj;
            
        }
    }
 


	/// BCOND 1
	float tide_l = m_fVars->tide_l;
	
	for (int j=1; j<m_height; j++ )
	{
		g_elf[j * m_width + 1] = tide_l;
		g_elf[j * m_width + m_width - 2] = tide_l;
		
		g_elf[j * m_width] = tide_l;
		g_elf[j * m_width + m_width - 1] = tide_l;
	}
	
	for (int i=1; i<m_width; i++ )
	{
		g_elf[i] =  g_elf[i + m_width];
		
		g_elf[i + m_width * (m_height - 1)  ] =  g_elf[i + m_width * (m_height - 2)];
	}
	
	for (int j=1; j<m_height; j++ )
	{
		for (int i=1; i<m_width; i++ )
		{
			ji = j * m_width + i;
			
			g_elf[ji] *= g_fsm[ji];
		}
	}

	
	if (m_fVars->iint % 10 == 0)
	{
		//ADVAVE()
		
		//       ADVUA=0
		//		FLUXUA=0
		
		//memset(g_advua, 0, F_DATA_SIZE * sizeof(float));
		//memset(g_fluxua, 0, F_DATA_SIZE * sizeof(float));
		
		
		float aam2d = m_fArrays->aam2d;
		
		for (int j=1; j<m_height; j++ )
		{
			for (int i=1; i<(m_width-1); i++ )
			{
				ji = j * m_width + i;
				jip1 = ji + 1;
				jim1 = ji - 1;
				
				/*g_fluxua[ji] = g_dy[j] * (.125f * ((g_d[ji + 1]+g_d[ji])*g_ua[ji + 1]
						+(g_d[ji]+g_d[ji - 1])*g_ua[ji])
										  *(g_ua[ji + 1]+g_ua[ji])
										  - g_d[ji]*2.0f*aam2d*(g_uab[ji + 1]-g_uab[ji])/g_dx[j]);*/
				g_fluxua[ji]=g_dy[j]*(.125e0*((g_d[jip1]+g_d[ji])*g_ua[jip1]
											  +(g_d[ji]+g_d[jim1])*g_ua[ji])
									  *(g_ua[jip1]+g_ua[ji])
									  -g_d[ji]*2.e0*aam2d*(g_uab[jip1]-g_uab[ji])/g_dx[j]);
				
				
			}
		}
		
		
		for (int j=1; j<m_height; j++ )
		{
			for (int i=1; i<m_width; i++ )
			{
				ji = j * m_width + i;
				jp1i = ji + m_width;
				jip1 = ji + 1;
				jim1 = ji - 1;
				jm1i = ji - m_width;
				jm1im1 = jm1i  - 1;
				
				/*g_tps[ji] =(g_d[ji]+g_d[jim1]+g_d[jm1i]+g_d[jm1im1]) *aam2d
				*((g_uab[ji]-g_uab[jm1i]) /(4.0f*g_dy[j])+(g_vab[ji]-g_vab[jim1]) /(4.0f*g_dx[j]) );
				
				g_fluxva[ji]=(.125f*((g_d[ji]+g_d[jm1i])*g_va[ji]
									 +(g_d[jim1]+g_d[jm1im1])*g_va[jim1])
							  *(g_ua[ji]+g_va[jm1i]) - g_tps[ji])*g_dx[j];*/
				
				g_tps[ji]=(g_d[ji]+g_d[jim1]+g_d[jm1i]+g_d[jm1im1])
				*aam2d
				*((g_uab[ji]-g_uab[jm1i])
				  /(4*g_dy[j])
				  +(g_vab[ji]-g_vab[jim1])
				  /(4*g_dx[j]) );
				
				g_fluxva[ji]=(.125e0*((g_d[ji]+g_d[jm1i])*g_va[ji]
									  +(g_d[jim1]+g_d[jm1im1])*g_va[jim1])
							  *(g_ua[ji]+g_ua[jm1i])
							  -g_tps[ji])*g_dx[j];
				
			}
		}

		
		for (int j=1; j<(m_height-1); j++ )
		{
			for (int i=1; i<(m_width-1); i++ )
			{
				ji = j * m_width + i;
				jim1 = ji - 1;
				jp1i = ji + m_width;
				
				g_advua[ji]=(g_fluxua[ji]-g_fluxua[jim1]
						   +g_fluxva[jp1i]-g_fluxva[ji])/g_aru[j];
			}
			
		}
		
		//memset(g_advva, 0, F_DATA_SIZE * sizeof(float));
		//memset(g_fluxva, 0, F_DATA_SIZE * sizeof(float));
		
		
		for (int j=1; j<(m_height-1); j++ )
		{
			for (int i=1; i<m_width; i++ )
			{
				ji = j * m_width + i;
				jp1i = ji + m_width;
				jip1 = ji + 1;
				jim1 = ji - 1;
				jm1i = ji - m_width;
				jm1im1 = jm1i  - 1;
				
				
			 	g_fluxva[ji]=g_dx[j]*(.125e0*((g_d[jp1i]+g_d[ji])
									       *g_va[jp1i]+(g_d[ji]+g_d[jm1i])*g_va[ji])
									      *(g_va[jp1i]+g_va[ji])
								         -g_d[ji]*2.e0*aam2d*(g_vab[jp1i]-g_vab[ji])/g_dy[j]);
				
			}
		}
		
		
		for (int j=1; j<m_height; j++ )
		{
			for (int i=1; i<m_width; i++ )
			{
				ji = j * m_width + i;
				jp1i = ji + m_width;
				jip1 = ji + 1;
				jim1 = ji - 1;
				jm1i = ji - m_width;
				jm1im1 = jm1i  - 1;
				
				
				g_fluxua[ji]=(.125e0*((g_d[ji]+g_d[jim1])*g_ua[ji]
									         +(g_d[jm1i]+g_d[jm1im1])*g_ua[jm1i])*
							                        (g_va[jim1]+g_va[ji])
							  -g_tps[ji])*g_dy[j];
			}
		}
		
		for (int j=1; j<(m_height-1); j++ )
		{
			for (int i=1; i<(m_width-1); i++ )
			{
				ji = j * m_width + i;
				jp1i = ji + m_width;
				jip1 = ji + 1;
				jim1 = ji - 1;
				jm1i = ji - m_width;
				jm1im1 = jm1i  - 1;
				
				g_advva[ji]=(g_fluxua[jip1]-g_fluxua[ji]
							         +g_fluxva[ji]-g_fluxva[jm1i])/g_arv[j];
			}
		}
	
		
		for (int j=1; j<(m_height-1); j++ )
		{
			for (int i=1; i<(m_width-1); i++ )
			{
				ji = j * m_width + i;
				jp1i = ji + m_width;
				jip1 = ji + 1;
				jim1 = ji - 1;
				jm1i = ji - m_width;
				jm1im1 = jm1i  - 1;
				
				jp1im1 = jp1i - 1;
				jm1ip1 = jm1i + 1;

				g_wubot[ji]=-0.5e0*(g_cbc[ji]+g_cbc[jim1])
				     *sqrtf(g_uab[ji]*g_uab[ji]+powf(.25e0*(g_vab[ji]
											  +g_vab[jp1i]+g_vab[jim1]+g_vab[jp1im1]), 2) )*g_uab[ji];
				
				g_wvbot[ji]=-0.5e0*(g_cbc[ji]+g_cbc[jm1i])
				    *sqrtf((.25e0*(g_uab[ji]+g_uab[jip1]
								  +g_uab[jm1i]+g_uab[jm1ip1]))**2+g_vab[ji]*g_vab[ji])*g_vab[ji];
				
			}
		}
		
		
		
	}
	
	
	
	
	
}




void KaspyCycler::getWindPressure(char uv)
{
	int kx, ky, kd, nx, ny, nd;
	float * p;
	float * px;
	float * py;
	float * pk;
	float xki, xka, yki, yka, xmi, xma, ymi, yma;
	float pkkd[50][50];
	float cd[50][50][4][4];
	
	float * pkk = &pkkd[0][0];
	float * c = &cd[0][0][0][0];
	
	if (uv == 'u')
	{
		kx = m_fWindData->kxu;
		ky = m_fWindData->kyu;
		pk = m_uwd0;
		
		xki = m_fWindData->xkui;
		xka = m_fWindData->xkua;
		yki = m_fWindData->ykui;
		yka = m_fWindData->ykua;
		
		p = g_ffu;
	}
	else if (uv == 'v')
	{
		kx = m_fWindData->kxv;
		ky = m_fWindData->kyv;
		pk = m_vwd0;
		
		xki = m_fWindData->xkvi;
		xka = m_fWindData->xkva;
		yki = m_fWindData->ykvi;
		yka = m_fWindData->ykva;
		
		p = g_ffv;
	}
	else if (uv == 'p')
	{
		kx = m_fWindData->kx;
		ky = m_fWindData->ky;
		//float kd = kx;
		pk = m_press0;
		xki = m_fWindData->xki;
		xka = m_fWindData->xka;
		yki = m_fWindData->yki;
		yka = m_fWindData->yka;

		
		p = g_ff;
		px = g_fxf;
		py = g_fyf;
	}
	else
	{
		// don't know what to do
		return;
	}
	
	kd = kx;
	
	nx = F_DATA_WIDTH;
	ny = F_DATA_HEIGHT;
	nd = F_DATA_WIDTH;
	
	xmi = m_fVars->xmi;
	xma = m_fVars->xma;
	ymi = m_fVars->ymi;
	yma = m_fVars->yma;
	
	float c1=3.1415926/180.0;
	float c2=111111.0f;
	
	
	float dky=(yka-yki)/(ky-1.0f);
	float  dkx=(xka-xki)/(kx-1.0f);
 
	float dy=(yma-ymi)/(ny-1.0f);
	float dx=(xma-xmi)/(nx-1.0f);
	
	
	for (int j=1; j<=ky; j++ )
	{
		for (int i=1; i<=kx; i++ )
		{
			pkk[j * 50 + i] = pk[(j - 1) * kd + i - 1];
		}
	}

	
	for (int j=1; j<=ky; j++ )
	{
		pkk[j*50+0] = 2.0f*pkk[j*50+1] - pkk[j*50+2];
		pkk[j*50+kx+1] = 2.0f*pkk[j*50+kx] - pkk[j*50+kx-1];
	}
	
	
	for (int i=0; i<=(kx+1); i++ )
	{
		pkk[0*50+i] = 2.0f*pkk[1*50+i] - pkk[2*50+i];
		pkk[(ky+1)*50+i] = 2.0f*pkk[ky*50+i] - pkk[(ky-1)*50+i];
	}
	
	getbicubic(kx + 2,ky + 2, 50, pkk,c);
	
	for (int j=0; j<ny; j++ )
	{
		float y = ymi + j*dy;
		int j0 = (int)((y - yki)/dky);
		
		if (j0 < 0)
		{
			j0 = 0;
		}
		
		if (j0 > ky-2)
		{
			j0 = ky-2;
		}
		
		float u = (y - (yki + j0*dky))/dky;
		
		for (int i=0; i<nx; i++ )
		{
			float x = xmi + i * dx;
			int i0 = (int)((x - xki)/dkx);
			
			if (i0 < 0) i0 = 0;
			
			if (i0 > kx-2) i0 = kx-2;
			
			float t = ( x - (xki + i0*dkx) )/dkx;
			
			float ay = 0.0f;
			float a2 = 0.0f;
			float a1 = 0.0f;
			
			int ji = j * nx + i;
			
			for (int k=3; k>=0; k-- )
			{
				ay = t*ay+((c[j0 * 800 + i0 * 16 + 3 * 4 + k] * u + c[j0 * 800 + i0 * 16 + 2 * 4 + k])*u
						   + c[j0 * 800 + i0 * 16 + 1 * 4 + k])*u + c[j0 * 800 + i0 * 16 + 0 * 4 + k];
			}
			
			if (uv == 'p')
			{
				for (int k=3; k>=0; k-- )
				{
					a2 = t*a2 + (3.0f*c[j0 * 800 + i0 * 16 + 3 * 4 + k]*u
								 + 2.0f*c[j0 * 800 + i0 * 16 + 2 * 4 + k])*u+c[j0 * 800 + i0 * 16 + 1 * 4 + k];
					
					a1 = u*a1 + (3.0f*c[j0 * 800 + i0 * 16 + k * 4 + 3]*t +
								 2.0f*c[j0 * 800 + i0 * 16 + k * 4 + 2])*t+c[j0 * 800 + i0 * 16 + k * 4 + 1];
					
				}
				
				a1 = a1/dkx/c2/cosf(c1*y);
				a2 = a2/dky/c2;
				
				px[ji] = a1;
				py[ji] = a2;
			}
			
			p[ji] = ay;
			
		}
		
	}
	

}







void getbicubic(int nx, int ny, int nd, float * z, float * c)
{
	float d1 = 1.0f;
	float d2 = 1.0f;
	
	float y[4];
	float y1[4];
	float y2[4];
	float y12[4];
	float cc[4][4];
	
	
	for (int j=1; j<ny-2; j++ )
	{
		for (int i=1; i<nx-2; i++ )
		{
			/*
			 Y(1)=Z(I,J)
			 Y(2)=Z(I+1,J)
			 Y(3)=Z(I+1,J+1)
			 Y(4)=Z(I,J+1)
			 */
			y[0] = z[j * nd + i];
			y[1] = z[j * nd + i + 1];
			y[2] = z[(j+1) * nd + i + 1];
			y[3] = z[(j+1) * nd + i];
			
			/*
			 Y1(1)=0.5*(Z(I+1,J)-Z(I-1,J))
			 Y1(4)=0.5*(Z(I+1,J+1)-Z(I-1,J+1))
			 Y1(2)=0.5*(Z(I+2,J)  -Z(I,J))
			 Y1(3)=0.5*(Z(I+2,J+1)-Z(I,J+1))
			 */
			y1[0] = 0.5f * (z[j * nd + i + 1] - z[j * nd + i - 1]);
			y1[3] = 0.5f * (z[(j+1) * nd + i + 1] - z[(j+1) * nd + i - 1]);
			y1[1] = 0.5f * (z[j * nd + i + 2] - z[j * nd + i]);
			y1[2] = 0.5f * (z[(j+1) * nd + i + 2] - z[(j+1) * nd + i]);

			
			/*
			 Y2(1)=0.5*(Z(I,J+1)  -Z(I,J-1))
			 Y2(2)=0.5*(Z(I+1,J+1)-Z(I+1,J-1))
			 Y2(3)=0.5*(Z(I+1,J+2)-Z(I+1,J))
			 Y2(4)=0.5*(Z(I,J+2)-Z(I,J))
			 */
			y2[0] = 0.5f * (z[(j+1) * nd + i] - z[(j-1) * nd + i]);
			y2[1] = 0.5f * (z[(j+1) * nd + i + 1] - z[(j-1) * nd + i + 1]);
			y2[2] = 0.5f * (z[(j+2) * nd + i + 1] - z[(j) * nd + i + 1]);
			y2[3] = 0.5f * (z[(j+2) * nd + i] - z[j * nd + i]);
			
			
			/*
			 Y12(1)=0.25*(Z(I+1,J+1)-Z(I+1,J-1)-Z(I-1,J+1)+Z(I-1,J-1))
			 Y12(2)=0.25*(Z(I+2,J+1)-Z(I+2,J-1)-Z(I,J+1)+Z(I,J-1))
			 Y12(3)=0.25*(Z(I+2,J+2)-Z(I+2,J)-Z(I,J+2)+Z(I,J))
			 Y12(4)=0.25*(Z(I+1,J+2)-Z(I+1,J)-Z(I-1,J+2)+Z(I-1,J))
			 */
			y12[0] = 0.25f * (z[(j+1) * nd + i + 1] - z[(j-1) * nd + i + 1]
							  - z[(j+1) * nd + i - 1] + z[(j-1) * nd + i - 1]);
			y12[1] = 0.25f * (z[(j+1) * nd + i + 2] - z[(j-1) * nd + i + 2]
							  - z[(j+1) * nd + i] + z[(j-1) * nd + i]);
			y12[2] = 0.25f * (z[(j+2) * nd + i + 2] - z[(j) * nd + i + 2]
							  - z[(j+2) * nd + i] + z[j * nd + i]);
			y12[3] = 0.25f * (z[(j+2) * nd + i + 1] - z[(j) * nd + i + 1]
							  - z[(j+2) * nd + i -1] + z[(j) * nd + i -1]);
	
			
			bcucof(&y[0],&y1[0],&y2[0],&y12[0],d1,d2,&cc[0][0]);
			
			for (int k=0; k<4; k++ )
			{
				for (int l=0; l<4; l++ )
				{
					//printf("\nk is %d l is %d\n", k, l);
					c[(j-1)* 800 + (i-1) * 16 + l * 4 + k ] = cc[l][k];
				}
			}
			
			
		}
	 }
	
}




void bcucof(float * y,float * y1,float * y2, float * y12,float d1,float d2,float * cc)
{
	float xx;
	float cl[16];
	
	float x[16];
	
	float wt[] = {
		1,0,-3,2,0,0,0,0,-3,0,9,-6,2,0,-6,4,
		0,0,0,0,0,0,0,0,3,0,-9,6,-2,0,6,-4,
		0,0,0,0,0,0,0,0,0,0,9,-6,0,0,-6,4,
		0,0,3,-2,0,0,0,0,0,0,-9,6,0,0,6,-4,
		0,0,0,0,1,0,-3,2,-2,0,6,-4,1,0,-3,2,
		0,0,0,0,0,0,0,0,-1,0,3,-2,1,0,-3,2,
		0,0,0,0,0,0,0,0,0,0,-3,2,0,0,3,-2,
		0,0,0,0,0,0,3,-2,0,0,-6,4,0,0,3,-2,
		0,1,-2,1,0,0,0,0,0,-3,6,-3,0,2,-4,2,
		0,0,0,0,0,0,0,0,0,3,-6,3,0,-2,4,-2,
		0,0,0,0,0,0,0,0,0,0,-3,3,0,0,2,-2,
		0,0,-1,1,0,0,0,0,0,0,3,-3,0,0,-2,2,
		0,0,0,0,0,1,-2,1,0,-2,4,-2,0,1,-2,1,
		0,0,0,0,0,0,0,0,0,-1,2,-1,0,1,-2,1,
		0,0,0,0,0,0,0,0,0,0,1,-1,0,0,-1,1,
		0,0,0,0,0,0,-1,1,0,0,2,-2,0,0,-1,1
 	};
	
	//float d1 = *pd1;
	//float d2 = *pd2;
	
	
	float d1d2 = d1 * d2;

	for (int i=0; i<4; i++ )
	{
		x[i] = y[i];
		x[i + 4] = y1[i] * d1;
		x[i + 8] = y2[i] * d2;
		x[i + 12] = y12[i] * d1d2;
	}
	
	for (int i=0; i<16; i++ )
	{
		xx = 0.0f;
		
		for (int k=0; k<16; k++ )
		{
			xx += wt[i + k*16] * x[k];
		}
		
		cl[i] = xx;
	}
	
	int l = 0;
	
	for (int i=0; i<4; i++ )
	{
		for (int j=0; j<4; j++ )
		{
			cc[j*4 + i] = cl[l++];
		}
	}
	
}






