#include "hip/hip_runtime.h"
//
//  KaspyCycler.cpp
//  kaspy_cuda
//
//  Created by Andrei Koulikov on 24.05.16.
//
//

#include "KaspyCycler.h"

// getnewpressureVAR(kx,ky,XKI,XKA,YKI,YKA,PRESS0,FF,fxf,fyf)

#ifdef _WIN64
extern "C"  void GETNEWPRESSUREVAR(int * kx, int * ky, float * xki, float * xka, float * yki, float * yka,
								   float * press0, float * ff, float * fxf, float * fyf);

extern "C"  void GETNEWWINDVAR(int * kxu, int * kyu, float * xkui, float * xkua,
							   float * ykui, float * ykua, float * uwd0, float * ffu);

#else

extern "C"  void getnewpressurevar_(int * kx, int * ky, float * xki, float * xka, float * yki, float * yka,
                              float * press0, float * ff, float * fxf, float * fyf);

extern "C"  void getnewwindvar_(int * kxu, int * kyu, float * xkui, float * xkua,
                               float * ykui, float * ykua, float * uwd0, float * ffu);

#endif





// call getnewwindVAR(kxu,kyu,XKUI,XKUA,YKUI,YKUA,uwd0,ffu)






float * g_fbu;
float * g_fbv;
float * g_ffu;
float * g_ffv;

float * g_fxb;
float * g_fxf;
float * g_fyb;
float * g_fyf;

float * g_fb;
float * g_ff;

float * g_wusurf;
float * g_wvsurf;

float * g_dum;
float * g_dvm;

float * g_d;
float * g_dx;
float * g_dy;

float * g_fluxua;
float * g_fluxva;

float * g_ua;
float * g_va;

float * g_el;
float * g_elf;
float * g_elb;





void KaspyCycler::findElves()
{
    //printf("arrays is set to %llxd \n", (long long)m_fArrays );
    
    
    float * elves = &(m_fArrays->elf[0][0]);
    
    float elf_min = elves[0];
    float elf_max = elves[0];
    
    for (int i=1; i<F_DATA_SIZE; i++)
    {
        if (elves[i] > elf_max)
        {
            elf_max = elves[i];
        }
        
        if (elves[i] < elf_min)
        {
            elf_min = elves[i];
        }
    }
    
    //printf("C SAYS: time is %f, elf min is %f, elf max is %f \n",m_fVars->timeh, elf_min, elf_max);
}

void KaspyCycler::sendDataToGPU()
{
    g_fbu = &m_fFloats->fbu[0][0];
    g_fbv = &m_fFloats->fbv[0][0];
    g_ffu = &m_fFloats->ffu[0][0];
    g_ffv = &m_fFloats->ffv[0][0];
    
    g_fxb = &m_fFloats->fxb[0][0];
    g_fxf = &m_fFloats->fxf[0][0];
    g_fyb = &m_fFloats->fyb[0][0];
    g_fyf = &m_fFloats->fyf[0][0];
    
    
    g_fb = &m_fFloats->fb[0][0];
    g_ff = &m_fFloats->ff[0][0];
    
    
    g_wusurf = &m_fArrays->wusurf[0][0];
    g_wvsurf = &m_fArrays->wvsurf[0][0];
    
    g_dum = &m_fArrays->dum[0][0];
    g_dvm = &m_fArrays->dvm[0][0];
    
    g_d = &m_fArrays->d[0][0];
    g_dx = &m_fArrays->dx[0];
    g_dy = &m_fArrays->dy[0];


    g_fluxua = &m_fArrays->fluxua[0][0];
    g_fluxva = &m_fArrays->fluxva[0][0];
    
    g_ua = &m_fArrays->ua[0][0];
    g_va = &m_fArrays->va[0][0];
    
    g_el = &m_fArrays->el[0][0];
    g_elf = &m_fArrays->elf[0][0];
    g_elb = &m_fArrays->elb[0][0];
}

void KaspyCycler::getDataToCPU()
{
    
}



/*
 DO J=2,JMM1
 DO I=2,IMM1
 uw=(btim*fbu(i,j)+ftim*ffu(i,j))
 vw=(btim*fbv(i,j)+ftim*ffv(i,j))
 speed=sqrt(uw**2+vw**2) !******************************************************
 !      speed=0
 windc=1.0e-3*(0.8+speed*0.065)*ro_ratio*speed
 WUSURF(I,J)=-windc*uw
 1 	*.25*(DUM(I,J+1)+DUM(I+1,J)+DUM(I-1,J)+DUM(I,J-1))+
 2  0.5*(d(i,j)+d(i-1,j))*(btim*FxB(i,j)+ftim*FxF(i,j))
 WVSURF(I,J)=-windc*vw
 1 	*.25*(DVM(I,J+1)+DVM(I+1,J)+DVM(I-1,J)+DVM(I,J-1))+
 2  0.5*(d(i,j)+d(i,j-1))*(btim*FyB(i,j)+ftim*FyF(i,j))
 end do
 end do
 */
void KaspyCycler::makeWsurf(float ro_ratio)
{
    m_fVars->timeh6 = (m_fVars->timeh / m_fVars->dht) + 1.0f;

    float timeh6 = m_fVars->timeh6;
    
    int pressSize = m_fWindData->kx * m_fWindData->ky;
    int windUSize = m_fWindData->kxu * m_fWindData->kyu;
    int windVSize = m_fWindData->kxv * m_fWindData->kyv;
    
    itime6 = (int)timeh6;

    ftim = (timeh6 - itime6);
    btim = 1.0f - ftim;
    
    if (itime6 > itime6_old)
    {
        itime6_old = itime6;
        
        memcpy(g_fxb, g_fxf, F_DATA_SIZE * sizeof(float));
        memcpy(g_fyb, g_fyf, F_DATA_SIZE * sizeof(float));
        memcpy(g_fb, g_ff, F_DATA_SIZE * sizeof(float));
        memcpy(g_fbu, g_ffu, F_DATA_SIZE * sizeof(float));
        memcpy(g_fbv, g_ffv, F_DATA_SIZE * sizeof(float));

        //setbuf(stdout,NULL);
        
        //printf("press size is %d\n", pressSize );
        
        //printf("press 000 is %f press0 00 is %f\n", m_press[0], m_press0[0]);

        //printf("copy pressure from %#018llx to %#018llx\n", m_press, m_press0);

        memcpy(m_press0, m_press + (itime6 - 1) * pressSize, pressSize * sizeof(float));
		
#ifdef _WIN64
		GETNEWPRESSUREVAR(&m_fWindData->kx, &m_fWindData->ky, &m_fWindData->xki, &m_fWindData->xka,
						  &m_fWindData->yki, &m_fWindData->yka, m_press0, g_ff, g_fxf, g_fyf);
#else
		getnewpressurevar_(&m_fWindData->kx, &m_fWindData->ky, &m_fWindData->xki, &m_fWindData->xka,
						   &m_fWindData->yki, &m_fWindData->yka, m_press0, g_ff, g_fxf, g_fyf);
#endif
		
		
        memcpy(m_uwd0, m_uwd + (itime6 - 1) * windUSize, windUSize * sizeof(float));
		
		
#ifdef _WIN64
		GETNEWWINDVAR(&m_fWindData->kxu, &m_fWindData->kyu, &m_fWindData->xkui, &m_fWindData->xkua,
					  &m_fWindData->ykui, &m_fWindData->ykua, m_uwd0, g_ffu);
#else
		getnewwindvar_(&m_fWindData->kxu, &m_fWindData->kyu, &m_fWindData->xkui, &m_fWindData->xkua,
		              &m_fWindData->ykui, &m_fWindData->ykua, m_uwd0, g_ffu);
#endif

		
        memcpy(m_vwd0, m_vwd + (itime6 - 1) * windVSize, windVSize * sizeof(float));
		
#ifdef _WIN64
		GETNEWWINDVAR(&m_fWindData->kxv, &m_fWindData->kyv, &m_fWindData->xkvi, &m_fWindData->xkva,
					  &m_fWindData->ykvi, &m_fWindData->ykva, m_vwd0, g_ffv);
#else
		getnewwindvar_(&m_fWindData->kxv, &m_fWindData->kyv, &m_fWindData->xkvi, &m_fWindData->xkva,
		              &m_fWindData->ykvi, &m_fWindData->ykva, m_vwd0, g_ffv);		
#endif
        
		
		
    }
	
        /*press0(:,:)=press(:,:,itime6)
        call getnewpressureVAR(kx,ky,XKI,XKA,YKI,YKA,PRESS0,
                               1 FF,fxf,fyf)
        uwd0(:,:)=uwd(:,:,itime6)
        call getnewwindVAR(kxu,kyu,XKUI,XKUA,YKUI,YKUA,uwd0,ffu)
        vwd0(:,:)=vwd(:,:,itime6)
        call getnewwindVAR(kxv,kyv,XKVI,XKVA,YKVI,YKVA,vwd0,ffv)*/


            
    float uw, vw, speed, windc;
    int ji, jp1i, jip1, jim1, jm1i;

    
    
    
    ftim = fmodf((float)m_fVars->timeh6, 1.0f);
    btim = 1.0f - ftim;
    
    for (int j=1; j<m_height; j++ )
    {
        for (int i=1; i<m_width; i++ )
        {
            if ((j<(m_height-1)) && i<(m_width-1))
            {
                ji = j * m_width + i;
                jp1i = ji + m_width;
                jip1 = ji + 1;
                jim1 = ji - 1;
                jm1i = ji - m_width;
                
                uw = btim * (g_fbu[ji]) + ftim * (g_ffu[ji]);
                vw = btim * (g_fbv[ji]) + ftim * (g_ffv[ji]);
                
                speed = sqrtf(uw*uw + vw*vw);
                windc = 0.001f * (0.8f + speed * 0.065f) * ro_ratio * speed;
                
                g_wusurf[ji] = -windc * uw *
                0.25f * (g_dum[jp1i]+g_dum[jip1]+g_dum[jim1]+g_dum[jm1i])
                + 0.5f * (g_d[ji] + g_d[jim1]) * (btim * g_fxb[ji] + ftim * g_fxf[ji]);
                
                g_wvsurf[ji] = -windc * vw *
                0.25f * (g_dvm[jp1i]+g_dvm[jip1]+g_dvm[jim1]+g_dvm[jm1i])
                + 0.5f * (g_d[ji] + g_d[jm1i]) * (btim * g_fyb[ji] + ftim * g_fyf[ji]);
            }
            
            //DO 405 J=2,JM
            //DO 405 I=2,IM
            //FLUXUA(I,J)=.25E0*(D(I,J)+D(I-1,J))*(DY(j)+DY(j))*UA(I,J)
            //405  FLUXVA(I,J)=.25E0*(D(I,J)+D(I,J-1))*(DX(j)+DX(j-1))*VA(I,J)

            g_fluxua[ji] = 0.25f * (g_d[ji] + g_d[jim1]) * (g_dy[j] + g_dy[j] /*???*/) * g_ua[ji];
            g_fluxva[ji] = 0.25f * (g_d[ji] + g_d[jm1i]) * (g_dx[j] + g_dx[j-1] ) * g_va[ji];
            
        }
    }
    
    
    /// HERE SHOULD START A NEW CUDA CALL TO KEEP fluxua fluxva synced
   
    /*DO 410 J=2,JMM1
    DO 410 I=2,IMM1
    410 ELF(I,J)=ELB(I,J)
    1    -DTE2*(FLUXUA(I+1,J)-FLUXUA(I,J)+FLUXVA(I,J+1)-FLUXVA(I,J))
    2                    / ART(J) */
    
    float dte2 = m_fVars->dte * 2.0f;
    
    for (int j=1; j<(m_height-1); j++ )
    {
        float artj = m_fArrays->art[j];
        
        for (int i=1; i<(m_width-1); i++ )
        {
            ji = j * m_width + i;
            jp1i = ji + m_width;
            jip1 = ji + 1;
            
            g_elf[ji] = g_elb[ji] - dte2 *
                (g_fluxua[jip1] - g_fluxua[ji] + g_fluxva[jp1i] - g_fluxva[ji]) /  artj;
            
        }
    }
 }








/*
 C     SURROUNDING
 DO J=2,KY+1
 DO I=2,KX+1
 PKK(I,J)=PK(I-1,J-1)
 END DO
 END DO
 DO J=2,KY+1
 PKK(1,J)=2*PKK(2,J)-PKK(3,J)
 PKK(KX+2,J)=2*PKK(KX+1,J)-PKK(KX,J)
 END DO
 DO I=1,KX+2
 PKK(I,1)=2*PKK(I,2)-PKK(I,3)
 PKK(I,KY+2)=2*PKK(I,KY+1)-PKK(I,KY)
 END DO
 CALL GETBICUBIC(KX+2,KY+2,50,PKK,C)
 
 
 
 do j=1,Ny
 y=ymi+(j-1)*dy
 j0=(y-yki)/dky+1
 if (j0<1) j0=1
 if (j0>ky-1) j0=ky-1
 u=(y-(yki+(j0-1)*dky))/dky
 
 do i=1,Nx
 x=xmi+(i-1)*dx
 i0=(x-xki)/dkx+1
 if (i0<1) I0=1
 if (i0>kx-1) i0=kx-1
 t=(x-(xki+(i0-1)*dkx))/dkx
 ay=0.
 a2=0.
 a1=0.
 DO K=4,1,-1
 ay=t*ay+((c(K,4,i0,j0)*u+c(k,3,i0,j0))*u+c(K,2,i0,j0))*u+
 1		  c(K,1,i0,j0)
 a2=t*a2+(3.*c(K,4,i0,j0)*u+2.*c(K,3,i0,j0))*u+c(K,2,i0,j0)
 a1=u*a1+(3.*c(4,K,i0,j0)*t+2.*c(3,K,i0,j0))*t+c(2,K,i0,j0)
 END DO
 a1=a1/dkx/c2/cos(c1*y)
 a2=a2/dky/c2
 
 p(i,j)=ay
 px(i,j)=a1
 py(i,j)=a2
 
 end do
 END DO
 
 CALL GETPRESScube(KX,KY,KX,PRESS0,XKI,XKA,YKI,YKA,
 1                     IM,JM,IM,FF,FXF,FYF,XMI,XMA,YMI,YMA)
 subroutine getpressCUBE(kx,ky,kd,pk,xki,xka,yki,yka,
 1                     nx,ny,nd,P,px,py,xmi,xma,ymi,yma)
 */


void KaspyCycler::getNewPressure()
{
	int kx = m_fWindData->kx;
	int ky = m_fWindData->ky;
	float kd = kx;
	float * pk = m_press0;
	float xki = m_fWindData->xki;
	float xka = m_fWindData->xka;
	float yki = m_fWindData->yki;
	float yka = m_fWindData->yka;
	int nx = F_DATA_WIDTH;
	int ny = F_DATA_HEIGHT;
	int nd = F_DATA_WIDTH;
	
	float * p = g_ff;
	float * px = g_fxf;
	float * py = g_fyf;
	
	float xmi = m_fVars->xmi;
	float xma = m_fVars->xma;
	float ymi = m_fVars->xmi;
	float yma = m_fVars->xma;
	
	float pkk[50][50];
	float c[50][50][4][4];
	
	float c1=3.1415926/180.0;
	float c2=111111.0f;
	
	float dky=(yka-yki)/(ky-1.0f);
	float  dkx=(xka-xki)/(kx-1.0f);
 
 	float dy=(yma-ymi)/(ny-1.0f);
 	float dx=(xma-xmi)/(nx-1.0f);
	
/*DO J=2,KY+1
 DO I=2,KX+1
 PKK(I,J)=PK(I-1,J-1)
 END DO
 END DO
 DO J=2,KY+1
 PKK(1,J)=2*PKK(2,J)-PKK(3,J)
 PKK(KX+2,J)=2*PKK(KX+1,J)-PKK(KX,J)
 END DO
 DO I=1,KX+2
 PKK(I,1)=2*PKK(I,2)-PKK(I,3)
 PKK(I,KY+2)=2*PKK(I,KY+1)-PKK(I,KY)
 END DO*/
	
	for (int j=1; j<ky; j++ )
	{
		for (int i=1; i<kx; i++ )
		{
			//int ji = j * kx + i;
			//int jm1i = ji - kx;
			//int jim1 = ji -  1;
			//int jm1im1 = jim1 - 1;
			
			pkk[j][i] = pk[j * (kx - 1) + i - 1];
		}
	}
	
	for (int j=1; j<ky; j++ )
	{
		pkk[j][0] = 2*pkk[j][1] - pkk[j][2];
		pkk[j][kx+1] = 2*pkk[j][kx] - pkk[j][kx-1];
	}
	
	for (int i=1; i<(kx+1); i++ )
	{
		pkk[0][i] = 2*pkk[1][i] - pkk[2][i];
		pkk[ky+1][i] = 2*pkk[ky][i] - pkk[ky-1][i];
	}
	
}






