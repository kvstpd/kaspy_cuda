#include "hip/hip_runtime.h"
//
//  KaspyCycler.cpp
//  kaspy_cuda
//
//  Created by Andrei Koulikov on 24.05.16.
//
//

#include "KaspyCycler.h"

// getnewpressureVAR(kx,ky,XKI,XKA,YKI,YKA,PRESS0,FF,fxf,fyf)

#ifdef _WIN64
extern "C"  void GETNEWPRESSUREVAR(int * kx, int * ky, float * xki, float * xka, float * yki, float * yka,
								   float * press0, float * ff, float * fxf, float * fyf);

extern "C"  void GETNEWWINDVAR(int * kxu, int * kyu, float * xkui, float * xkua,
							   float * ykui, float * ykua, float * uwd0, float * ffu);

#else

extern "C"  void getnewpressurevar_(int * kx, int * ky, float * xki, float * xka, float * yki, float * yka,
                              float * press0, float * ff, float * fxf, float * fyf);

extern "C"  void getnewwindvar_(int * kxu, int * kyu, float * xkui, float * xkua,
                               float * ykui, float * ykua, float * uwd0, float * ffu);

#endif





// call getnewwindVAR(kxu,kyu,XKUI,XKUA,YKUI,YKUA,uwd0,ffu)






float * g_fbu;
float * g_fbv;
float * g_ffu;
float * g_ffv;

float * g_fxb;
float * g_fxf;
float * g_fyb;
float * g_fyf;

float * g_fb;
float * g_ff;

float * g_wusurf;
float * g_wvsurf;

float * g_dum;
float * g_dvm;

float * g_d;
float * g_dx;
float * g_dy;

float * g_fluxua;
float * g_fluxva;

float * g_ua;
float * g_va;

float * g_el;
float * g_elf;
float * g_elb;





void KaspyCycler::findElves()
{
    //printf("arrays is set to %llxd \n", (long long)m_fArrays );
    
    
    float * elves = &(m_fArrays->elf[0][0]);
    
    float elf_min = elves[0];
    float elf_max = elves[0];
    
    for (int i=1; i<F_DATA_SIZE; i++)
    {
        if (elves[i] > elf_max)
        {
            elf_max = elves[i];
        }
        
        if (elves[i] < elf_min)
        {
            elf_min = elves[i];
        }
    }
    
    //printf("C SAYS: time is %f, elf min is %f, elf max is %f \n",m_fVars->timeh, elf_min, elf_max);
}

void KaspyCycler::sendDataToGPU()
{
    g_fbu = &m_fFloats->fbu[0][0];
    g_fbv = &m_fFloats->fbv[0][0];
    g_ffu = &m_fFloats->ffu[0][0];
    g_ffv = &m_fFloats->ffv[0][0];
    
    g_fxb = &m_fFloats->fxb[0][0];
    g_fxf = &m_fFloats->fxf[0][0];
    g_fyb = &m_fFloats->fyb[0][0];
    g_fyf = &m_fFloats->fyf[0][0];
    
    
    g_fb = &m_fFloats->fb[0][0];
    g_ff = &m_fFloats->ff[0][0];
    
    
    g_wusurf = &m_fArrays->wusurf[0][0];
    g_wvsurf = &m_fArrays->wvsurf[0][0];
    
    g_dum = &m_fArrays->dum[0][0];
    g_dvm = &m_fArrays->dvm[0][0];
    
    g_d = &m_fArrays->d[0][0];
    g_dx = &m_fArrays->dx[0];
    g_dy = &m_fArrays->dy[0];


    g_fluxua = &m_fArrays->fluxua[0][0];
    g_fluxva = &m_fArrays->fluxva[0][0];
    
    g_ua = &m_fArrays->ua[0][0];
    g_va = &m_fArrays->va[0][0];
    
    g_el = &m_fArrays->el[0][0];
    g_elf = &m_fArrays->elf[0][0];
    g_elb = &m_fArrays->elb[0][0];
}

void KaspyCycler::getDataToCPU()
{
    
}



/*
 DO J=2,JMM1
 DO I=2,IMM1
 uw=(btim*fbu(i,j)+ftim*ffu(i,j))
 vw=(btim*fbv(i,j)+ftim*ffv(i,j))
 speed=sqrt(uw**2+vw**2) !******************************************************
 !      speed=0
 windc=1.0e-3*(0.8+speed*0.065)*ro_ratio*speed
 WUSURF(I,J)=-windc*uw
 1 	*.25*(DUM(I,J+1)+DUM(I+1,J)+DUM(I-1,J)+DUM(I,J-1))+
 2  0.5*(d(i,j)+d(i-1,j))*(btim*FxB(i,j)+ftim*FxF(i,j))
 WVSURF(I,J)=-windc*vw
 1 	*.25*(DVM(I,J+1)+DVM(I+1,J)+DVM(I-1,J)+DVM(I,J-1))+
 2  0.5*(d(i,j)+d(i,j-1))*(btim*FyB(i,j)+ftim*FyF(i,j))
 end do
 end do
 */
void KaspyCycler::makeWsurf(float ro_ratio)
{
    m_fVars->timeh6 = (m_fVars->timeh / m_fVars->dht) + 1.0f;

    float timeh6 = m_fVars->timeh6;
    
    int pressSize = m_fWindData->kx * m_fWindData->ky;
    int windUSize = m_fWindData->kxu * m_fWindData->kyu;
    int windVSize = m_fWindData->kxv * m_fWindData->kyv;
    
    itime6 = (int)timeh6;

    ftim = (timeh6 - itime6);
    btim = 1.0f - ftim;
    
    if (itime6 > itime6_old)
    {
        itime6_old = itime6;
        
        memcpy(g_fxb, g_fxf, F_DATA_SIZE * sizeof(float));
        memcpy(g_fyb, g_fyf, F_DATA_SIZE * sizeof(float));
        memcpy(g_fb, g_ff, F_DATA_SIZE * sizeof(float));
        memcpy(g_fbu, g_ffu, F_DATA_SIZE * sizeof(float));
        memcpy(g_fbv, g_ffv, F_DATA_SIZE * sizeof(float));

        //setbuf(stdout,NULL);
        
        //printf("press size is %d\n", pressSize );
        
        //printf("press 000 is %f press0 00 is %f\n", m_press[0], m_press0[0]);

        //printf("copy pressure from %#018llx to %#018llx\n", m_press, m_press0);

        memcpy(m_press0, m_press + (itime6 - 1) * pressSize, pressSize * sizeof(float));
		
#ifdef _WIN64
		GETNEWPRESSUREVAR(&m_fWindData->kx, &m_fWindData->ky, &m_fWindData->xki, &m_fWindData->xka,
						  &m_fWindData->yki, &m_fWindData->yka, m_press0, g_ff, g_fxf, g_fyf);
#else
		getnewpressurevar_(&m_fWindData->kx, &m_fWindData->ky, &m_fWindData->xki, &m_fWindData->xka,
						   &m_fWindData->yki, &m_fWindData->yka, m_press0, g_ff, g_fxf, g_fyf);
#endif
		
		
        memcpy(m_uwd0, m_uwd + (itime6 - 1) * windUSize, windUSize * sizeof(float));
		
		
#ifdef _WIN64
		GETNEWWINDVAR(&m_fWindData->kxu, &m_fWindData->kyu, &m_fWindData->xkui, &m_fWindData->xkua,
					  &m_fWindData->ykui, &m_fWindData->ykua, m_uwd0, g_ffu);
#else
		getnewwindvar_(&m_fWindData->kxu, &m_fWindData->kyu, &m_fWindData->xkui, &m_fWindData->xkua,
		              &m_fWindData->ykui, &m_fWindData->ykua, m_uwd0, g_ffu);
#endif

		
        memcpy(m_vwd0, m_vwd + (itime6 - 1) * windVSize, windVSize * sizeof(float));
		
#ifdef _WIN64
		GETNEWWINDVAR(&m_fWindData->kxv, &m_fWindData->kyv, &m_fWindData->xkvi, &m_fWindData->xkva,
					  &m_fWindData->ykvi, &m_fWindData->ykva, m_vwd0, g_ffv);
#else
		getnewwindvar_(&m_fWindData->kxv, &m_fWindData->kyv, &m_fWindData->xkvi, &m_fWindData->xkva,
		              &m_fWindData->ykvi, &m_fWindData->ykva, m_vwd0, g_ffv);		
#endif
        
		
		
    }
	
        /*press0(:,:)=press(:,:,itime6)
        call getnewpressureVAR(kx,ky,XKI,XKA,YKI,YKA,PRESS0,
                               1 FF,fxf,fyf)
        uwd0(:,:)=uwd(:,:,itime6)
        call getnewwindVAR(kxu,kyu,XKUI,XKUA,YKUI,YKUA,uwd0,ffu)
        vwd0(:,:)=vwd(:,:,itime6)
        call getnewwindVAR(kxv,kyv,XKVI,XKVA,YKVI,YKVA,vwd0,ffv)*/


            
    float uw, vw, speed, windc;
    int ji, jp1i, jip1, jim1, jm1i;

    
    
    
    ftim = fmodf((float)m_fVars->timeh6, 1.0f);
    btim = 1.0f - ftim;
    
    for (int j=1; j<m_height; j++ )
    {
        for (int i=1; i<m_width; i++ )
        {
            if ((j<(m_height-1)) && i<(m_width-1))
            {
                ji = j * m_width + i;
                jp1i = ji + m_width;
                jip1 = ji + 1;
                jim1 = ji - 1;
                jm1i = ji - m_width;
                
                uw = btim * (g_fbu[ji]) + ftim * (g_ffu[ji]);
                vw = btim * (g_fbv[ji]) + ftim * (g_ffv[ji]);
                
                speed = sqrtf(uw*uw + vw*vw);
                windc = 0.001f * (0.8f + speed * 0.065f) * ro_ratio * speed;
                
                g_wusurf[ji] = -windc * uw *
                0.25f * (g_dum[jp1i]+g_dum[jip1]+g_dum[jim1]+g_dum[jm1i])
                + 0.5f * (g_d[ji] + g_d[jim1]) * (btim * g_fxb[ji] + ftim * g_fxf[ji]);
                
                g_wvsurf[ji] = -windc * vw *
                0.25f * (g_dvm[jp1i]+g_dvm[jip1]+g_dvm[jim1]+g_dvm[jm1i])
                + 0.5f * (g_d[ji] + g_d[jm1i]) * (btim * g_fyb[ji] + ftim * g_fyf[ji]);
            }
            
            //DO 405 J=2,JM
            //DO 405 I=2,IM
            //FLUXUA(I,J)=.25E0*(D(I,J)+D(I-1,J))*(DY(j)+DY(j))*UA(I,J)
            //405  FLUXVA(I,J)=.25E0*(D(I,J)+D(I,J-1))*(DX(j)+DX(j-1))*VA(I,J)

            g_fluxua[ji] = 0.25f * (g_d[ji] + g_d[jim1]) * (g_dy[j] + g_dy[j] /*???*/) * g_ua[ji];
            g_fluxva[ji] = 0.25f * (g_d[ji] + g_d[jm1i]) * (g_dx[j] + g_dx[j-1] ) * g_va[ji];
            
        }
    }
    
    
    /// HERE SHOULD START A NEW CUDA CALL TO KEEP fluxua fluxva synced
   
    /*DO 410 J=2,JMM1
    DO 410 I=2,IMM1
    410 ELF(I,J)=ELB(I,J)
    1    -DTE2*(FLUXUA(I+1,J)-FLUXUA(I,J)+FLUXVA(I,J+1)-FLUXVA(I,J))
    2                    / ART(J) */
    
    float dte2 = m_fVars->dte * 2.0f;
    
    for (int j=1; j<(m_height-1); j++ )
    {
        float artj = m_fArrays->art[j];
        
        for (int i=1; i<(m_width-1); i++ )
        {
            ji = j * m_width + i;
            jp1i = ji + m_width;
            jip1 = ji + 1;
            
            g_elf[ji] = g_elb[ji] - dte2 *
                (g_fluxua[jip1] - g_fluxua[ji] + g_fluxva[jp1i] - g_fluxva[ji]) /  artj;
            
        }
    }
    


    
    
}
