#include "hip/hip_runtime.h"
//
//  KaspyCycler.cpp
//  kaspy_cuda
//
//  Created by Andrei Koulikov on 24.05.16.
//
//

#include "KaspyCycler.h"

void getbicubic(int nx, int ny, int nd, float * z, float * c);
void bcucof(float * y,float * y1,float * y2, float * y12,float d1,float d2,float * cc);






float * g_fbu;
float * g_fbv;
float * g_ffu;
float * g_ffv;

float * g_fxb;
float * g_fxf;
float * g_fyb;
float * g_fyf;

float * g_fb;
float * g_ff;

float * g_wusurf;
float * g_wvsurf;

float * g_dum;
float * g_dvm;

float * g_d;
float * g_dx;
float * g_dy;

float * g_fluxua;
float * g_fluxva;

float * g_ua;
float * g_va;

float * g_el;
float * g_elf;
float * g_elb;





void KaspyCycler::findElves()
{
    //printf("arrays is set to %llxd \n", (long long)m_fArrays );
    
    
    float * elves = &(m_fArrays->elf[0][0]);
    
    float elf_min = elves[0];
    float elf_max = elves[0];
    
    for (int i=1; i<F_DATA_SIZE; i++)
    {
        if (elves[i] > elf_max)
        {
            elf_max = elves[i];
        }
        
        if (elves[i] < elf_min)
        {
            elf_min = elves[i];
        }
    }
    
    //printf("C SAYS: time is %f, elf min is %f, elf max is %f \n",m_fVars->timeh, elf_min, elf_max);
}

void KaspyCycler::sendDataToGPU()
{
    g_fbu = &m_fFloats->fbu[0][0];
    g_fbv = &m_fFloats->fbv[0][0];
    g_ffu = &m_fFloats->ffu[0][0];
    g_ffv = &m_fFloats->ffv[0][0];
    
    g_fxb = &m_fFloats->fxb[0][0];
    g_fxf = &m_fFloats->fxf[0][0];
    g_fyb = &m_fFloats->fyb[0][0];
    g_fyf = &m_fFloats->fyf[0][0];
    
    
    g_fb = &m_fFloats->fb[0][0];
    g_ff = &m_fFloats->ff[0][0];
    
    
    g_wusurf = &m_fArrays->wusurf[0][0];
    g_wvsurf = &m_fArrays->wvsurf[0][0];
    
    g_dum = &m_fArrays->dum[0][0];
    g_dvm = &m_fArrays->dvm[0][0];
    
    g_d = &m_fArrays->d[0][0];
    g_dx = &m_fArrays->dx[0];
    g_dy = &m_fArrays->dy[0];


    g_fluxua = &m_fArrays->fluxua[0][0];
    g_fluxva = &m_fArrays->fluxva[0][0];
    
    g_ua = &m_fArrays->ua[0][0];
    g_va = &m_fArrays->va[0][0];
    
    g_el = &m_fArrays->el[0][0];
    g_elf = &m_fArrays->elf[0][0];
    g_elb = &m_fArrays->elb[0][0];
}

void KaspyCycler::getDataToCPU()
{
    
}




void KaspyCycler::makeWsurf(float ro_ratio)
{
    m_fVars->timeh6 = (m_fVars->timeh / m_fVars->dht) + 1.0f;

    float timeh6 = m_fVars->timeh6;
    
    int pressSize = m_fWindData->kx * m_fWindData->ky;
    int windUSize = m_fWindData->kxu * m_fWindData->kyu;
    int windVSize = m_fWindData->kxv * m_fWindData->kyv;
    
    itime6 = (int)timeh6;

    ftim = (timeh6 - itime6);
    btim = 1.0f - ftim;
    
    if (itime6 > itime6_old)
    {
        itime6_old = itime6;
        
        memcpy(g_fxb, g_fxf, F_DATA_SIZE * sizeof(float));
        memcpy(g_fyb, g_fyf, F_DATA_SIZE * sizeof(float));
        memcpy(g_fb, g_ff, F_DATA_SIZE * sizeof(float));
        memcpy(g_fbu, g_ffu, F_DATA_SIZE * sizeof(float));
        memcpy(g_fbv, g_ffv, F_DATA_SIZE * sizeof(float));

        //setbuf(stdout,NULL);
        
        //printf("press size is %d\n", pressSize );
        
        //printf("press 000 is %f press0 00 is %f\n", m_press[0], m_press0[0]);

        //printf("copy pressure from %#018llx to %#018llx\n", m_press, m_press0);

        memcpy(m_press0, m_press + (itime6 - 1) * pressSize, pressSize * sizeof(float));
		
		getNewWind('p');
		
		
		
        memcpy(m_uwd0, m_uwd + (itime6 - 1) * windUSize, windUSize * sizeof(float));
		
		
		getNewWind('u');
		

		
        memcpy(m_vwd0, m_vwd + (itime6 - 1) * windVSize, windVSize * sizeof(float));
		
		getNewWind('v');
		

        
		
		
    }
	


            
    float uw, vw, speed, windc;
    int ji, jp1i, jip1, jim1, jm1i;

    
    
    
    ftim = fmodf((float)m_fVars->timeh6, 1.0f);
    btim = 1.0f - ftim;
    
    for (int j=1; j<m_height; j++ )
    {
        for (int i=1; i<m_width; i++ )
        {
            if ((j<(m_height-1)) && i<(m_width-1))
            {
                ji = j * m_width + i;
                jp1i = ji + m_width;
                jip1 = ji + 1;
                jim1 = ji - 1;
                jm1i = ji - m_width;
                
                uw = btim * (g_fbu[ji]) + ftim * (g_ffu[ji]);
                vw = btim * (g_fbv[ji]) + ftim * (g_ffv[ji]);
                
                speed = sqrtf(uw*uw + vw*vw);
                windc = 0.001f * (0.8f + speed * 0.065f) * ro_ratio * speed;
                
                g_wusurf[ji] = -windc * uw *
                0.25f * (g_dum[jp1i]+g_dum[jip1]+g_dum[jim1]+g_dum[jm1i])
                + 0.5f * (g_d[ji] + g_d[jim1]) * (btim * g_fxb[ji] + ftim * g_fxf[ji]);
                
                g_wvsurf[ji] = -windc * vw *
                0.25f * (g_dvm[jp1i]+g_dvm[jip1]+g_dvm[jim1]+g_dvm[jm1i])
                + 0.5f * (g_d[ji] + g_d[jm1i]) * (btim * g_fyb[ji] + ftim * g_fyf[ji]);
            }
            


            g_fluxua[ji] = 0.25f * (g_d[ji] + g_d[jim1]) * (g_dy[j] + g_dy[j] ) * g_ua[ji];
            g_fluxva[ji] = 0.25f * (g_d[ji] + g_d[jm1i]) * (g_dx[j] + g_dx[j-1] ) * g_va[ji];
            
        }
    }
    
    
    /// HERE SHOULD START A NEW CUDA CALL TO KEEP fluxua fluxva synced
   
    /*DO 410 J=2,JMM1
    DO 410 I=2,IMM1
    410 ELF(I,J)=ELB(I,J)
    1    -DTE2*(FLUXUA(I+1,J)-FLUXUA(I,J)+FLUXVA(I,J+1)-FLUXVA(I,J))
    2                    / ART(J) */
    
    float dte2 = m_fVars->dte * 2.0f;
    
    for (int j=1; j<(m_height-1); j++ )
    {
        float artj = m_fArrays->art[j];
        
        for (int i=1; i<(m_width-1); i++ )
        {
            ji = j * m_width + i;
            jp1i = ji + m_width;
            jip1 = ji + 1;
            
            g_elf[ji] = g_elb[ji] - dte2 *
                (g_fluxua[jip1] - g_fluxua[ji] + g_fluxva[jp1i] - g_fluxva[ji]) /  artj;
            
        }
    }
 }




void KaspyCycler::getNewWind(char uv)
{
	int kx, ky, kd, nx, ny, nd;
	float * p;
	float * px;
	float * py;
	float * pk;
	float xki, xka, yki, yka, xmi, xma, ymi, yma;
	float pkkd[50][50];
	float cd[50][50][4][4];
	
	float * pkk = &pkkd[0][0];
	float * c = &cd[0][0][0][0];
	
	if (uv == 'u')
	{
		kx = m_fWindData->kxu;
		ky = m_fWindData->kyu;
		pk = m_uwd0;
		
		xki = m_fWindData->xkui;
		xka = m_fWindData->xkua;
		yki = m_fWindData->ykui;
		yka = m_fWindData->ykua;
		
		p = g_ffu;
	}
	else if (uv == 'v')
	{
		kx = m_fWindData->kxv;
		ky = m_fWindData->kyv;
		pk = m_vwd0;
		
		xki = m_fWindData->xkvi;
		xka = m_fWindData->xkva;
		yki = m_fWindData->ykvi;
		yka = m_fWindData->ykva;
		
		p = g_ffv;
	}
	else if (uv == 'p')
	{
		kx = m_fWindData->kx;
		ky = m_fWindData->ky;
		//float kd = kx;
		pk = m_press0;
		xki = m_fWindData->xki;
		xka = m_fWindData->xka;
		yki = m_fWindData->yki;
		yka = m_fWindData->yka;

		
		p = g_ff;
		px = g_fxf;
		py = g_fyf;
	}
	else
	{
		// don't know what to do
		return;
	}
	
	kd = kx;
	
	nx = F_DATA_WIDTH;
	ny = F_DATA_HEIGHT;
	nd = F_DATA_WIDTH;
	
	xmi = m_fVars->xmi;
	xma = m_fVars->xma;
	ymi = m_fVars->ymi;
	yma = m_fVars->yma;
	
	float c1=3.1415926/180.0;
	float c2=111111.0f;
	
	
	float dky=(yka-yki)/(ky-1.0f);
	float  dkx=(xka-xki)/(kx-1.0f);
 
	float dy=(yma-ymi)/(ny-1.0f);
	float dx=(xma-xmi)/(nx-1.0f);
	
	
	for (int j=1; j<=ky; j++ )
	{
		for (int i=1; i<=kx; i++ )
		{
			pkk[j * 50 + i] = pk[(j - 1) * kd + i - 1];
		}
	}

	
	for (int j=1; j<=ky; j++ )
	{
		pkk[j*50+0] = 2.0f*pkk[j*50+1] - pkk[j*50+2];
		pkk[j*50+kx+1] = 2.0f*pkk[j*50+kx] - pkk[j*50+kx-1];
	}
	
	
	for (int i=0; i<=(kx+1); i++ )
	{
		pkk[0*50+i] = 2.0f*pkk[1*50+i] - pkk[2*50+i];
		pkk[(ky+1)*50+i] = 2.0f*pkk[ky*50+i] - pkk[(ky-1)*50+i];
	}
	
	getbicubic(kx + 2,ky + 2, 50, pkk,c);
	
	for (int j=0; j<ny; j++ )
	{
		float y = ymi + j*dy;
		int j0 = (int)((y - yki)/dky);
		
		if (j0 < 0)
		{
			j0 = 0;
		}
		
		if (j0 > ky-2)
		{
			j0 = ky-2;
		}
		
		float u = (y - (yki + j0*dky))/dky;
		
		for (int i=0; i<nx; i++ )
		{
			float x = xmi + i * dx;
			int i0 = (int)((x - xki)/dkx);
			
			if (i0 < 0) i0 = 0;
			
			if (i0 > kx-2) i0 = kx-2;
			
			float t = ( x - (xki + i0*dkx) )/dkx;
			
			float ay = 0.0f;
			float a2 = 0.0f;
			float a1 = 0.0f;
			
			int ji = j * nx + i;
			
			for (int k=3; k>=0; k-- )
			{
				ay = t*ay+((c[j0 * 800 + i0 * 16 + 3 * 4 + k] * u + c[j0 * 800 + i0 * 16 + 2 * 4 + k])*u
						   + c[j0 * 800 + i0 * 16 + 1 * 4 + k])*u + c[j0 * 800 + i0 * 16 + 0 * 4 + k];
			}
			
			if (uv == 'p')
			{
				for (int k=3; k>=0; k-- )
				{
					a2 = t*a2 + (3.0f*c[j0 * 800 + i0 * 16 + 3 * 4 + k]*u
								 + 2.0f*c[j0 * 800 + i0 * 16 + 2 * 4 + k])*u+c[j0 * 800 + i0 * 16 + 1 * 4 + k];
					
					a1 = u*a1 + (3.0f*c[j0 * 800 + i0 * 16 + k * 4 + 3]*t +
								 2.0f*c[j0 * 800 + i0 * 16 + k * 4 + 2])*t+c[j0 * 800 + i0 * 16 + k * 4 + 1];
					
				}
				
				a1 = a1/dkx/c2/cosf(c1*y);
				a2 = a2/dky/c2;
				
				px[ji] = a1;
				py[ji] = a2;
			}
			
			p[ji] = ay;
			
		}
		
	}
	

}




void KaspyCycler::getNewPressure()
{
	int kx = m_fWindData->kx;
	int ky = m_fWindData->ky;
	//float kd = kx;
	float * pk = m_press0;
	float xki = m_fWindData->xki;
	float xka = m_fWindData->xka;
	float yki = m_fWindData->yki;
	float yka = m_fWindData->yka;
	int nx = F_DATA_WIDTH;
	int ny = F_DATA_HEIGHT;
	//int nd = F_DATA_WIDTH;
	
	float * p = g_ff;
	float * px = g_fxf;
	float * py = g_fyf;
	
	float xmi = m_fVars->xmi;
	float xma = m_fVars->xma;
	float ymi = m_fVars->ymi;
	float yma = m_fVars->yma;
	
	float pkkd[50][50];
	float cd[50][50][4][4];
	
	float * pkk = &pkkd[0][0];
	float * c = &cd[0][0][0][0];
	
	float c1=3.1415926/180.0;
	float c2=111111.0f;
	
	float dky=(yka-yki)/(ky-1.0f);
	float  dkx=(xka-xki)/(kx-1.0f);
 
 	float dy=(yma-ymi)/(ny-1.0f);
 	float dx=(xma-xmi)/(nx-1.0f);
	
	
	for (int j=1; j<=ky; j++ )
	{
		for (int i=1; i<=kx; i++ )
		{
			pkk[j * 50 + i] = pk[(j - 1) * kx + i - 1];
		}
	}

	for (int j=1; j<=ky; j++ )
	{
		pkk[j*50+0] = 2.0f*pkk[j*50+1] - pkk[j*50+2];
		pkk[j*50+kx+1] = 2.0f*pkk[j*50+kx] - pkk[j*50+kx-1];
	}
	
	for (int i=0; i<=(kx+1); i++ )
	{
		pkk[0*50+i] = 2.0f*pkk[1*50+i] - pkk[2*50+i];
		pkk[(ky+1)*50+i] = 2.0f*pkk[ky*50+i] - pkk[(ky-1)*50+i];
	}
	
	
	getbicubic(kx + 2,ky + 2, 50, pkk,c);
	
	for (int j=0; j<ny; j++ )
	{
		float y = ymi + j*dy;
		int j0 = (int)((y - yki)/dky);
		
		if (j0 < 0)
		{
			j0 = 0;
		}
		
		if (j0 > ky-2)
		{
			j0 = ky-2;
		}
		
		float u = (y - (yki + j0*dky))/dky;
		
		for (int i=0; i<nx; i++ )
		{
			float x = xmi + i * dx;
			int i0 = (int)((x - xki)/dkx);
			
			if (i0 < 0) i0 = 0;
			
			if (i0 > kx-2) i0 = kx-2;
			
			float t = ( x - (xki + i0*dkx) )/dkx;
			
			float ay = 0.0f;
			float a2 = 0.0f;
			float a1 = 0.0f;
			
			for (int k=3; k>=0; k-- )
			{
				ay = t*ay+((c[j0 * 800 + i0 * 16 + 3 * 4 + k] * u + c[j0 * 800 + i0 * 16 + 2 * 4 + k])*u
						   + c[j0 * 800 + i0 * 16 + 1 * 4 + k])*u + c[j0 * 800 + i0 * 16 + 0 * 4 + k];
				
				a2 = t*a2 + (3.0f*c[j0 * 800 + i0 * 16 + 3 * 4 + k]*u
							 + 2.0f*c[j0 * 800 + i0 * 16 + 2 * 4 + k])*u+c[j0 * 800 + i0 * 16 + 1 * 4 + k];
				
				a1 = u*a1 + (3.0f*c[j0 * 800 + i0 * 16 + k * 4 + 3]*t +
							 2.0f*c[j0 * 800 + i0 * 16 + k * 4 + 2])*t+c[j0 * 800 + i0 * 16 + k * 4 + 1];
				
			}
			
			a1 = a1/dkx/c2/cosf(c1*y);
			a2 = a2/dky/c2;
			
			int ji = j * nx + i;
			
			p[ji] = ay;
			px[ji] = a1;
			py[ji] = a2;
			
		}
		
	}
	


}




void getbicubic(int nx, int ny, int nd, float * z, float * c)
{
	float d1 = 1.0f;
	float d2 = 1.0f;
	
	float y[4];
	float y1[4];
	float y2[4];
	float y12[4];
	float cc[4][4];
	
	
	for (int j=1; j<ny-2; j++ )
	{
		for (int i=1; i<nx-2; i++ )
		{
			/*
			 Y(1)=Z(I,J)
			 Y(2)=Z(I+1,J)
			 Y(3)=Z(I+1,J+1)
			 Y(4)=Z(I,J+1)
			 */
			y[0] = z[j * nd + i];
			y[1] = z[j * nd + i + 1];
			y[2] = z[(j+1) * nd + i + 1];
			y[3] = z[(j+1) * nd + i];
			
			/*
			 Y1(1)=0.5*(Z(I+1,J)-Z(I-1,J))
			 Y1(4)=0.5*(Z(I+1,J+1)-Z(I-1,J+1))
			 Y1(2)=0.5*(Z(I+2,J)  -Z(I,J))
			 Y1(3)=0.5*(Z(I+2,J+1)-Z(I,J+1))
			 */
			y1[0] = 0.5f * (z[j * nd + i + 1] - z[j * nd + i - 1]);
			y1[3] = 0.5f * (z[(j+1) * nd + i + 1] - z[(j+1) * nd + i - 1]);
			y1[1] = 0.5f * (z[j * nd + i + 2] - z[j * nd + i]);
			y1[2] = 0.5f * (z[(j+1) * nd + i + 2] - z[(j+1) * nd + i]);

			
			/*
			 Y2(1)=0.5*(Z(I,J+1)  -Z(I,J-1))
			 Y2(2)=0.5*(Z(I+1,J+1)-Z(I+1,J-1))
			 Y2(3)=0.5*(Z(I+1,J+2)-Z(I+1,J))
			 Y2(4)=0.5*(Z(I,J+2)-Z(I,J))
			 */
			y2[0] = 0.5f * (z[(j+1) * nd + i] - z[(j-1) * nd + i]);
			y2[1] = 0.5f * (z[(j+1) * nd + i + 1] - z[(j-1) * nd + i + 1]);
			y2[2] = 0.5f * (z[(j+2) * nd + i + 1] - z[(j) * nd + i + 1]);
			y2[3] = 0.5f * (z[(j+2) * nd + i] - z[j * nd + i]);
			
			
			/*
			 Y12(1)=0.25*(Z(I+1,J+1)-Z(I+1,J-1)-Z(I-1,J+1)+Z(I-1,J-1))
			 Y12(2)=0.25*(Z(I+2,J+1)-Z(I+2,J-1)-Z(I,J+1)+Z(I,J-1))
			 Y12(3)=0.25*(Z(I+2,J+2)-Z(I+2,J)-Z(I,J+2)+Z(I,J))
			 Y12(4)=0.25*(Z(I+1,J+2)-Z(I+1,J)-Z(I-1,J+2)+Z(I-1,J))
			 */
			y12[0] = 0.25f * (z[(j+1) * nd + i + 1] - z[(j-1) * nd + i + 1]
							  - z[(j+1) * nd + i - 1] + z[(j-1) * nd + i - 1]);
			y12[1] = 0.25f * (z[(j+1) * nd + i + 2] - z[(j-1) * nd + i + 2]
							  - z[(j+1) * nd + i] + z[(j-1) * nd + i]);
			y12[2] = 0.25f * (z[(j+2) * nd + i + 2] - z[(j) * nd + i + 2]
							  - z[(j+2) * nd + i] + z[j * nd + i]);
			y12[3] = 0.25f * (z[(j+2) * nd + i + 1] - z[(j) * nd + i + 1]
							  - z[(j+2) * nd + i -1] + z[(j) * nd + i -1]);
	
			
			bcucof(&y[0],&y1[0],&y2[0],&y12[0],d1,d2,&cc[0][0]);
			
			for (int k=0; k<4; k++ )
			{
				for (int l=0; l<4; l++ )
				{
					//printf("\nk is %d l is %d\n", k, l);
					c[(j-1)* 800 + (i-1) * 16 + l * 4 + k ] = cc[l][k];
				}
			}
			
			
		}
	 }
	
}




void bcucof(float * y,float * y1,float * y2, float * y12,float d1,float d2,float * cc)
{
	float xx;
	float cl[16];
	
	float x[16];
	
	float wt[] = {
		1,0,-3,2,0,0,0,0,-3,0,9,-6,2,0,-6,4,
		0,0,0,0,0,0,0,0,3,0,-9,6,-2,0,6,-4,
		0,0,0,0,0,0,0,0,0,0,9,-6,0,0,-6,4,
		0,0,3,-2,0,0,0,0,0,0,-9,6,0,0,6,-4,
		0,0,0,0,1,0,-3,2,-2,0,6,-4,1,0,-3,2,
		0,0,0,0,0,0,0,0,-1,0,3,-2,1,0,-3,2,
		0,0,0,0,0,0,0,0,0,0,-3,2,0,0,3,-2,
		0,0,0,0,0,0,3,-2,0,0,-6,4,0,0,3,-2,
		0,1,-2,1,0,0,0,0,0,-3,6,-3,0,2,-4,2,
		0,0,0,0,0,0,0,0,0,3,-6,3,0,-2,4,-2,
		0,0,0,0,0,0,0,0,0,0,-3,3,0,0,2,-2,
		0,0,-1,1,0,0,0,0,0,0,3,-3,0,0,-2,2,
		0,0,0,0,0,1,-2,1,0,-2,4,-2,0,1,-2,1,
		0,0,0,0,0,0,0,0,0,-1,2,-1,0,1,-2,1,
		0,0,0,0,0,0,0,0,0,0,1,-1,0,0,-1,1,
		0,0,0,0,0,0,-1,1,0,0,2,-2,0,0,-1,1
 	};
	
	//float d1 = *pd1;
	//float d2 = *pd2;
	
	
	float d1d2 = d1 * d2;

	for (int i=0; i<4; i++ )
	{
		x[i] = y[i];
		x[i + 4] = y1[i] * d1;
		x[i + 8] = y2[i] * d2;
		x[i + 12] = y12[i] * d1d2;
	}
	
	for (int i=0; i<16; i++ )
	{
		xx = 0.0f;
		
		for (int k=0; k<16; k++ )
		{
			xx += wt[i + k*16] * x[k];
		}
		
		cl[i] = xx;
	}
	
	int l = 0;
	
	for (int i=0; i<4; i++ )
	{
		for (int j=0; j<4; j++ )
		{
			cc[j*4 + i] = cl[l++];
		}
	}
	
}






