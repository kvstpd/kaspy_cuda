#include "hip/hip_runtime.h"

#include <GL/glew.h>

#include <GL/freeglut.h>


// CUDA utilities and system includes
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Includes
#include <stdlib.h>
#include <stdio.h>



#include "DrawArrayWindow.h"


void display(void);
void keyboard(unsigned char key, int /*x*/, int /*y*/);
void reshape(int x, int y);
void cleanup(void);


static DrawArrayWindow * currentWindow = 0;



__global__ void gpu_gen_buffer(int width, int height, unsigned char * buf, float kw, float kh, int zw, const float * z, const float * h, float brighten)//,  float m_b2y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	int zi  = int( kw * i);
	int zj  =  int( kh * j);

	int ind;
	float zval, hval;

	if ((i < width) && (j < height))
	{
		ind = (i + (height - j - 1) * width) * 4;

		zval = z[zi + zw * zj] * brighten * 255.0f;

		hval = h[zi + zw * zj];

		if (hval <= 0.f)
		{
			if (hval <= -2500.f)
			{
				buf[ind] =  140;//int(fminf(zval, 255.0f)); //r
				buf[ind + 1] = 100; //g
				buf[ind + 2] = 0;
			}
			else if (hval <= -1000.f)
			{
				buf[ind] =  150;//int(fminf(zval, 255.0f)); //r
				buf[ind + 1] = 130; //g
				buf[ind + 2] = 0;
			}
			else if (hval <= -250.f)
			{
				buf[ind] =  40;//int(fminf(zval, 255.0f)); //r
				buf[ind + 1] = 120; //g
				buf[ind + 2] = 30;
			}
			else
			{
				buf[ind] =  20;//int(fminf(zval, 255.0f)); //r
				buf[ind + 1] = 100; //g
				buf[ind + 2] = 20;
			}

		}
		else if (zval >=0.0f)
		{
			int zvv = int(fminf(zval, 255.0f));

			buf[ind] =  0xff;//int(fminf(zval, 255.0f)); //r
			buf[ind + 1] = 0xff - zvv; //g
			buf[ind + 2] = 0xff - zvv; //b
		}
		else
		{
			int zvv = int(fminf(-zval, 255.0f));

			buf[ind] = 0xff - zvv;  //r
			buf[ind + 1] = 0xff - zvv; //g
			buf[ind + 2] = 0xff; //b
		}



		//buf[ind] += i % 0xff; //r
		//buf[ind + 1] = 0x00; //g
		//buf[ind + 2] += ind % 0xff; //b
		buf[ind + 3] = 0xff; //a
	}
	

}



DrawArrayWindow::DrawArrayWindow()
{
	m_cuda_gResource = 0;
	//m_texArray = 0;
	m_pixData = 0;

	m_gl_texId = 0;
	m_gl_bufId = 0;

	m_width = DAW_START_WINDOW_WIDTH;
	m_height = DAW_START_WINDOW_HEIGHT;


	m_gpu_original = 0;
	m_original_width  = 0;
	m_original_height = 0;

	m_brighten = 1.0f;

}

void DrawArrayWindow::set_data_to_display(float * gpu_zdata, float * gpu_hdata, int w, int h, int p_w)
{
	m_gpu_original = gpu_zdata;
	m_gpu_h = gpu_hdata;

	m_original_width  = w;
	m_original_height = h;

	m_original_pitched_width = p_w;
}


int DrawArrayWindow::gl_init(int device)
{
	int argc = 0;

	setbuf(stdout,NULL);
    printf("starting to init GL\n");

    glutInit(&argc, 0);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(m_width, m_height);
    glutCreateWindow("Ocean");

    glewInit();

    if (!glewIsSupported("GL_VERSION_1_5 GL_ARB_vertex_buffer_object GL_ARB_pixel_buffer_object"))
    {
        fprintf(stderr, "Error: no GL extensions found!\n");

		return -1;
    }
	
	hipError_t err = hipGetLastError();

	cudaGLSetGLDevice(device);

	err = hipGetLastError();


    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutReshapeFunc(reshape);


	 glutCloseFunc(cleanup);


    glDisable(GL_DEPTH_TEST);
	glEnable(GL_TEXTURE_2D);  


	 if (this->gl_rebuild_texture(m_width, m_height) < 0 )
	 {
        fprintf(stderr, "Error: texture creation error!\n");

		return -1;
	 }
	 
 

	currentWindow = this;

	setbuf(stdout,NULL);
    printf("GL init finished\n");
    
	return 0;
}


void DrawArrayWindow::clear_gl_objects()
{
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

		

	if (m_cuda_gResource)
	{
		hipGraphicsUnregisterResource(m_cuda_gResource);

		m_cuda_gResource = 0;
	}


	if (m_gl_texId)
	{
		glDeleteTextures(1, &m_gl_texId);
		m_gl_texId = 0;
	}


	if (m_gl_bufId)
	{
		glDeleteBuffers(1, &m_gl_bufId);
		m_gl_bufId = 0;	
	}


	if (m_pixData)
	{
		delete m_pixData;
		m_pixData = 0;
	}
}

int DrawArrayWindow::gl_rebuild_texture(int width, int height)
{
	this->clear_gl_objects();


	m_width = width;
	m_height = height;


	// 4 bytes per pixel
	int dataSize = sizeof(unsigned char) * m_width * m_height * 4;
	 
	m_pixData = (unsigned char *) malloc(dataSize);
 
  

	glGenTextures(1, &m_gl_texId);        

	glBindTexture(GL_TEXTURE_2D, m_gl_texId);

	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, m_width, m_height, 0, GL_RGBA, GL_UNSIGNED_BYTE, m_pixData);


	glGenBuffers(1, &m_gl_bufId);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB,m_gl_bufId);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, dataSize, m_pixData, GL_STREAM_COPY);
	  

	if (hipGraphicsGLRegisterBuffer(&m_cuda_gResource, m_gl_bufId, cudaGraphicsMapFlagsWriteDiscard)  == hipSuccess)
	{
		fprintf(stderr, "GL texture linked to CUDA\n");
	}
	else
	{
		fprintf(stderr, "Error: unable to link GL texture to CUDA!\n");

		return -1;
	}

	return 0;
}


void DrawArrayWindow::gl_draw_frame()
{
	display();
	glutMainLoopEvent();
}



void DrawArrayWindow::gl_deinit()
{
	this->clear_gl_objects();

	if (currentWindow)
	{
		currentWindow = 0;
	}
}



void DrawArrayWindow::fill_texture()
{
	if (m_cuda_gResource && m_gpu_original)
	{
		size_t num_bytes;
		unsigned char * devPixData;

		hipError_t err = hipSuccess;


		dim3 threadsPerSquareBlock(16, 16);

		dim3 numSquareBlocks((m_width + threadsPerSquareBlock.x - 1) / threadsPerSquareBlock.x, (m_height + threadsPerSquareBlock.y - 1) / threadsPerSquareBlock.y);


		hipGraphicsMapResources(1, &m_cuda_gResource, 0);
  
		hipGraphicsResourceGetMappedPointer((void**)&devPixData, &num_bytes, m_cuda_gResource);
 

		gpu_gen_buffer<<<numSquareBlocks, threadsPerSquareBlock>>>(m_width, m_height, devPixData, (0.0f + m_original_width) / m_width, (0.0f + m_original_height) / m_height, m_original_pitched_width, m_gpu_original, m_gpu_h, m_brighten);//,  m_gpu_b2x);


		err = hipGetLastError();

		if (err != hipSuccess)
		{
			fprintf(stderr, "DRAW: Failed to launch GPU algorithm  (error code %s)!\n", hipGetErrorString(err));
			//exit(EXIT_FAILURE);
		}
   
		//hipDeviceSynchronize();

		hipGraphicsUnmapResources(1, &m_cuda_gResource, 0);


	}

}



DrawArrayWindow::~DrawArrayWindow()
{
	this->gl_deinit();
}


void display(void)
{
	if (currentWindow)
	{
		currentWindow->fill_texture();
	
		glClear(GL_COLOR_BUFFER_BIT);

		glColor3f(1.0f, 1.0f, 1.0f);
		glBindTexture(GL_TEXTURE_2D, currentWindow->m_gl_texId);
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB,currentWindow->m_gl_bufId);
 
		glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, currentWindow->m_width, currentWindow->m_height, GL_RGBA, GL_UNSIGNED_BYTE, 0);



		glBegin(GL_QUADS);
		glVertex2f(0, 0);
		glTexCoord2f(0, 0);
		glVertex2f(0, 1);
		glTexCoord2f(1, 0);
		glVertex2f(1, 1);
		glTexCoord2f(1, 1);
		glVertex2f(1, 0);
		glTexCoord2f(0, 1);
		glEnd();
		glBindTexture(GL_TEXTURE_2D, 0);
		glutSwapBuffers();

	}
}



void keyboard(unsigned char key, int /*x*/, int /*y*/)
{

    switch (key)
    {
        case 27:
        case 'q':
        case 'Q':
            printf("Shutting down...\n");

            glutDestroyWindow(glutGetWindow());
            break;

        default:
            break;
    }
}

void reshape(int x, int y)
{
    glViewport(0, 0, x, y);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0, 1, 0, 1, 0, 1);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

	if (currentWindow)
	{
		currentWindow->gl_rebuild_texture(x, y);
	}	
}

void cleanup(void)
{
	if (currentWindow)
	{
		currentWindow->clear_gl_objects();
	}
	
}