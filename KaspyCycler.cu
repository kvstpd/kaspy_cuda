#include "hip/hip_runtime.h"
//
//  KaspyCycler.cpp
//  kaspy_cuda
//
//  Created by Andrei Koulikov on 24.05.16.
//
//

#include "KaspyCycler.h"


#include "hip/hip_runtime.h"
#include ""



void getbicubic(int nx, int ny, int nd, float * z, float * c);
void bcucof(float * y,float * y1,float * y2, float * y12,float d1,float d2,float * cc);


float grav = 9.806;

__device__ float * dev_fbu = 0;
__device__ float * dev_fbv = 0;
__device__ float * dev_ffu = 0;
__device__ float * dev_ffv = 0;

__device__ float * dev_fxb = 0;
__device__ float * dev_fxf = 0;
__device__ float * dev_fyb = 0;
__device__ float * dev_fyf = 0;

__device__ float * dev_fb = 0;
__device__ float * dev_ff = 0;

__device__ float * dev_wusurf = 0;
__device__ float * dev_wvsurf = 0;

__device__ float * dev_dum = 0;
__device__ float * dev_dvm = 0;

__device__ float * dev_d = 0;
__device__ float * dev_dx = 0;
__device__ float * dev_dy = 0;

__device__ float * dev_fluxua = 0;
__device__ float * dev_fluxva = 0;

__device__ float * dev_ua = 0;
__device__ float * dev_va = 0;

__device__ float * dev_uab = 0;
__device__ float * dev_vab = 0;

__device__ float * dev_uaf = 0;
__device__ float * dev_vaf = 0;


__device__ float * dev_el = 0;
__device__ float * dev_elf = 0;
__device__ float * dev_elb = 0;

__device__ float * dev_fsm = 0;

__device__ float * dev_tps = 0;


__device__ float * dev_advua = 0;
__device__ float * dev_advva = 0;

__device__ float * dev_aru = 0;
__device__ float * dev_arv = 0;

__device__ float * dev_wubot = 0;
__device__ float * dev_wvbot = 0;
__device__ float * dev_cbc = 0;

__device__ float * dev_cor = 0;

__device__ float * dev_h = 0;

__device__ float * dev_press0 = 0;
__device__ float * dev_uwd0 = 0;
__device__ float * dev_vwd0 = 0;

__device__ float * dev_art = 0;


__device__ float * dev_temp = 0;


float * g_fbu = 0;
float * g_fbv = 0;
float * g_ffu = 0;
float * g_ffv = 0;

float * g_fxb = 0;
float * g_fxf = 0;
float * g_fyb = 0;
float * g_fyf = 0;

float * g_fb = 0;
float * g_ff = 0;

float * g_wusurf = 0;
float * g_wvsurf = 0;

float * g_dum = 0;
float * g_dvm = 0;

float * g_d = 0;
float * g_dx = 0;
float * g_dy = 0;

float * g_fluxua = 0;
float * g_fluxva = 0;

float * g_ua = 0;
float * g_va = 0;

float * g_uab = 0;
float * g_vab = 0;

float * g_uaf = 0;
float * g_vaf = 0;


float * g_el = 0;
float * g_elf = 0;
float * g_elb = 0;

float * g_fsm = 0;

float * g_tps = 0;


float * g_advua = 0;
float * g_advva = 0;

float * g_aru = 0;
float * g_arv = 0;

float * g_wubot = 0;
float * g_wvbot = 0;
float * g_cbc = 0;

float * g_cor = 0;

float * g_h = 0;

float * g_press0 = 0;
float * g_uwd0 = 0;
float * g_vwd0 = 0;

float * g_art = 0;


__constant__ __device__  float dev_grav = 9.806f;
__constant__ __device__  float dev_ro_ratio = 1.29f/1020.0f;

__constant__ __device__  int  dev_width;
__constant__ __device__  int  dev_height;
__constant__ __device__  int  dev_widthm1;
__constant__ __device__  int  dev_heightm1;

__constant__ __device__ int dev_ewidth;

__constant__ __device__ float dev_dte;
__constant__ __device__ float dev_dte2;
__constant__ __device__ float dev_aam2d;

__constant__ __device__ float dev_tide_l = 0.0f;

__constant__ __device__ float dev_alpha = 0.225f;

__constant__ __device__ float dev_vmaxl = 100.0f;;

__device__ int dev_should_stop = 0;

__constant__ __device__ float dev_smoth = 0.10f;


__constant__ __device__ float dev_wt[] = {
	1,0,-3,2,0,0,0,0,-3,0,9,-6,2,0,-6,4,
	0,0,0,0,0,0,0,0,3,0,-9,6,-2,0,6,-4,
	0,0,0,0,0,0,0,0,0,0,9,-6,0,0,-6,4,
	0,0,3,-2,0,0,0,0,0,0,-9,6,0,0,6,-4,
	0,0,0,0,1,0,-3,2,-2,0,6,-4,1,0,-3,2,
	0,0,0,0,0,0,0,0,-1,0,3,-2,1,0,-3,2,
	0,0,0,0,0,0,0,0,0,0,-3,2,0,0,3,-2,
	0,0,0,0,0,0,3,-2,0,0,-6,4,0,0,3,-2,
	0,1,-2,1,0,0,0,0,0,-3,6,-3,0,2,-4,2,
	0,0,0,0,0,0,0,0,0,3,-6,3,0,-2,4,-2,
	0,0,0,0,0,0,0,0,0,0,-3,3,0,0,2,-2,
	0,0,-1,1,0,0,0,0,0,0,3,-3,0,0,-2,2,
	0,0,0,0,0,1,-2,1,0,-2,4,-2,0,1,-2,1,
	0,0,0,0,0,0,0,0,0,-1,2,-1,0,1,-2,1,
	0,0,0,0,0,0,0,0,0,0,1,-1,0,0,-1,1,
	0,0,0,0,0,0,-1,1,0,0,2,-2,0,0,-1,1
};





__device__ void dev_bcucof(float * y,float * y1,float * y2, float * y12,float d1,float d2,float * cc)
{
	float xx;
	float cl[16];
	
	float x[16];
	
	
	float d1d2 = d1 * d2;
	
	for (int i=0; i<4; i++ )
	{
		x[i] = y[i];
		x[i + 4] = y1[i] * d1;
		x[i + 8] = y2[i] * d2;
		x[i + 12] = y12[i] * d1d2;
	}
	
	for (int i=0; i<16; i++ )
	{
		xx = 0.0f;
		
		for (int k=0; k<16; k++ )
		{
			xx += dev_wt[i + k*16] * x[k];
		}
		
		cl[i] = xx;
	}
	
	int l = 0;
	
	for (int i=0; i<4; i++ )
	{
		for (int j=0; j<4; j++ )
		{
			cc[j*4 + i] = cl[l++];
		}
	}
	
}


__global__ void dev_bucubic(int nx, int ny, int nd, float * z, float * c)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	
	float d1 = 1.0f;
	float d2 = 1.0f;
	
	float y[4];
	float y1[4];
	float y2[4];
	float y12[4];
	float cc[4][4];
	
	
	
	
	if (i > 0 && j > 0 && i < (nx - 2) && j < (ny - 2))
	{
		
		y[0] = z[j * nd + i];
		y[1] = z[j * nd + i + 1];
		y[2] = z[(j+1) * nd + i + 1];
		y[3] = z[(j+1) * nd + i];
		
		y1[0] = 0.5f * (z[j * nd + i + 1] - z[j * nd + i - 1]);
		y1[3] = 0.5f * (z[(j+1) * nd + i + 1] - z[(j+1) * nd + i - 1]);
		y1[1] = 0.5f * (z[j * nd + i + 2] - z[j * nd + i]);
		y1[2] = 0.5f * (z[(j+1) * nd + i + 2] - z[(j+1) * nd + i]);
		
		
		y2[0] = 0.5f * (z[(j+1) * nd + i] - z[(j-1) * nd + i]);
		y2[1] = 0.5f * (z[(j+1) * nd + i + 1] - z[(j-1) * nd + i + 1]);
		y2[2] = 0.5f * (z[(j+2) * nd + i + 1] - z[(j) * nd + i + 1]);
		y2[3] = 0.5f * (z[(j+2) * nd + i] - z[j * nd + i]);
		
		
		y12[0] = 0.25f * (z[(j+1) * nd + i + 1] - z[(j-1) * nd + i + 1]
						  - z[(j+1) * nd + i - 1] + z[(j-1) * nd + i - 1]);
		y12[1] = 0.25f * (z[(j+1) * nd + i + 2] - z[(j-1) * nd + i + 2]
						  - z[(j+1) * nd + i] + z[(j-1) * nd + i]);
		y12[2] = 0.25f * (z[(j+2) * nd + i + 2] - z[(j) * nd + i + 2]
						  - z[(j+2) * nd + i] + z[j * nd + i]);
		y12[3] = 0.25f * (z[(j+2) * nd + i + 1] - z[(j) * nd + i + 1]
						  - z[(j+2) * nd + i -1] + z[(j) * nd + i -1]);
		
		
		dev_bcucof(&y[0],&y1[0],&y2[0],&y12[0],d1,d2,&cc[0][0]);
		
		for (int k=0; k<4; k++ )
		{
			for (int l=0; l<4; l++ )
			{
				//printf("\nk is %d l is %d\n", k, l);
				c[(j-1)* 800 + (i-1) * 16 + l * 4 + k ] = cc[l][k];
			}
		}
	}
}



/**/

__global__ void surf_and_flux_1(float ftim)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	int jp1i = ji + dev_width;
	int jip1 = ji + 1;
	int jim1 = ji - 1;
	int jm1i = ji - dev_width;
	
	float btim = 1.0f - ftim;
	
	
	if (i < dev_widthm1 && j < dev_heightm1)
	{
		float uw = btim * (dev_fbu[ji]) + ftim * (dev_ffu[ji]);
		float vw = btim * (dev_fbv[ji]) + ftim * (dev_ffv[ji]);
		
		float speed = sqrtf(uw*uw + vw*vw);
		float windc = 0.001f * (0.8f + speed * 0.065f) * dev_ro_ratio * speed;
		
		dev_wusurf[ji] = -windc * uw *
		0.25f * (dev_dum[jp1i]+dev_dum[jip1]+dev_dum[jim1]+dev_dum[jm1i])
		+ 0.5f * (dev_d[ji] + dev_d[jim1]) * (btim * dev_fxb[ji] + ftim * dev_fxf[ji]);
		
		dev_wvsurf[ji] = -windc * vw *
		0.25f * (dev_dvm[jp1i]+dev_dvm[jip1]+dev_dvm[jim1]+dev_dvm[jm1i])
		+ 0.5f * (dev_d[ji] + dev_d[jm1i]) * (btim * dev_fyb[ji] + ftim * dev_fyf[ji]);
	}
	
	if (i < dev_width && j < dev_height)
	{
		dev_fluxua[ji] = 0.25f * (dev_d[ji] + dev_d[jim1]) * (dev_dy[j] + dev_dy[j] ) * dev_ua[ji];
		dev_fluxva[ji] = 0.25f * (dev_d[ji] + dev_d[jm1i]) * (dev_dx[j] + dev_dx[j-1] ) * dev_va[ji];
	}
}


__global__ void elf_and_flux_2()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	int jp1i = ji + dev_width;
	int jip1 = ji + 1;
	
	if (i > 0 && j > 0 && i < dev_widthm1 && j < dev_heightm1)
	{
		dev_elf[ji] = dev_elb[ji] - dev_dte2 *
		(dev_fluxua[jip1] - dev_fluxua[ji] + dev_fluxva[jp1i] - dev_fluxva[ji]) / dev_art[j];
	}

}


__global__ void bcond_1_j()
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (j > 0 && j < dev_height)
	{
		dev_elf[j * dev_width + 1] = dev_tide_l;
		dev_elf[j * dev_width + dev_width - 2] = dev_tide_l;
		
		dev_elf[j * dev_width] = dev_tide_l;
		dev_elf[j * dev_width + dev_width - 1] = dev_tide_l;
	}
}

__global__ void bcond_1_i()
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i > 0 && i< dev_width)
	{
		dev_elf[i] =  dev_elf[i + dev_width];
		
		dev_elf[i + dev_width * (dev_height - 1)  ] =  dev_elf[i + dev_width * (dev_height - 2)];
	}
}


__global__ void bcond_1_ji()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	
	
	if (i > 0 && j > 0 && i < dev_width && j < dev_height)
	{
		dev_elf[ji] *= dev_fsm[ji];
	}
	
}

__global__ void uaf_and_vaf_3()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	
 	int jp1i = ji + dev_width;
 	int jip1 = ji + 1;
 	int jim1 = ji - 1;
 	int jm1i = ji - dev_width;
 	//int jm1im1 = jm1i  - 1;
 	int jp1im1 = jp1i - 1;
 	int jm1ip1 = jm1i + 1;
	
	
	

	if (i > 0 && j > 0)
	{
		if (i < dev_width && j < dev_heightm1)
		{

			
			
			float uaf1= dev_advua[ji]   -0.25f*(dev_cor[j]*dev_d[ji]*(dev_va[jp1i]+dev_va[ji])
												+dev_cor[j]*dev_d[jim1]*(dev_va[jp1im1]+dev_va[jim1]) )
			+0.5f*dev_grav*dev_dy[j]/dev_aru[j]*(dev_d[ji]+dev_d[jim1])
			*( (1.0f-2.0f*dev_alpha)*(dev_el[ji]-dev_el[jim1])
			  +dev_alpha*(dev_elb[ji]-dev_elb[jim1]+dev_elf[ji]-dev_elf[jim1]) )
			+dev_wusurf[ji]-dev_wubot[ji];
			
			dev_uaf[ji]=
			((dev_h[ji]+dev_elb[ji]+dev_h[jim1]+dev_elb[jim1])*dev_uab[ji]
			 -4.e0*dev_dte*uaf1)  /(dev_h[ji]+dev_elf[ji]+dev_h[jim1]+dev_elf[jim1]);
		}
		
		if (i < dev_widthm1 && j < dev_height)
		{
			float vaf1=dev_advva[ji]
			+.25f*(  dev_cor[j]*dev_d[ji]*(dev_ua[jip1]+dev_ua[ji])
				   +dev_cor[j-1]*dev_d[jm1i]*(dev_ua[jm1ip1]+dev_ua[jm1i]) )
			+0.5f*dev_grav*dev_dx[j]/dev_arv[j]*(dev_d[ji]+dev_d[jm1i])
			*( (1.0f-2.0f*dev_alpha)*(dev_el[ji]-dev_el[jm1i])
			  +dev_alpha*(dev_elb[ji]-dev_elb[jm1i]+dev_elf[ji]-dev_elf[jm1i]) )
			+ dev_wvsurf[ji]-dev_wvbot[ji];
			
			dev_vaf[ji]= ((dev_h[ji]+dev_elb[ji]+dev_h[jm1i]+dev_elb[jm1i])*dev_vab[ji]
						  -4.0f*dev_dte*vaf1) /(dev_h[ji]+dev_elf[ji]+dev_h[jm1i]+dev_elf[jm1i]);
			
		}
		
	}
	
}


__global__ void bcond_2_j()
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	
	int j1 =  j * dev_width;
	int j2 =  j1 + 1;
	int j3 =  j1 + 2;
	int jl = j1 + dev_widthm1;
	int jlm1 = jl - 1;
	
	float gae;
	
	if (j > 0 && j < dev_heightm1)
	{
		if(dev_dum[jl] > 0.5f)
		{
			gae = dev_dte*sqrtf(dev_grav*dev_h[jl])/dev_dx[j];
			
			dev_uaf[jl] = gae*dev_ua[jlm1]+(1.0f-gae)*dev_ua[jl];
		}
		else
		{
			dev_uaf[jl] = 0.0f;
		}
		
		dev_vaf[jl]=0.0f;
		
		if(dev_dum[j2] > 0.5f)
		{
			gae = dev_dte*sqrtf(dev_grav*dev_h[j2])/dev_dx[j];
			dev_uaf[j2]=gae*dev_ua[j3]+(1.0f-gae)*dev_ua[j2];
		}
		else
		{
			dev_uaf[j2]=0.0f;
		}
		
		dev_uaf[j1]=dev_uaf[j2];
		dev_vaf[j1]=0.0f;

	}
}

__global__ void bcond_2_i()
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	int jli = dev_width * (dev_heightm1) + i;
	int jlm1i = jli - dev_width;
	
	int j1i = i;
	
	int j2i = dev_width + j1i;
	
	int j3i = dev_width + j2i;
	
	float gae;
	
	if (i > 0 && i< dev_widthm1)
	{
		if (dev_dvm[jli] > 0.5f)
		{
			gae = dev_dte * sqrtf(dev_grav * dev_h[jli]) / dev_dy[dev_heightm1];
			
			dev_vaf[jli] = gae * dev_va[jlm1i]+(1.0f-gae)*dev_va[jli];
		}
		else
		{
			dev_vaf[jli]=0.0f;
		}
		
		dev_uaf[jli]=0.0;
		
		if (dev_dvm[j2i] > 0.5f)
		{
			gae=dev_dte*sqrtf(dev_grav*dev_h[j2i])/dev_dy[0];
			
			dev_vaf[j2i]=gae*dev_va[j3i]+(1.-gae)*dev_va[j2i];
		}
		else
		{
			dev_vaf[j2i]=0.0f;
		}
		
		
		dev_vaf[j1i]=dev_vaf[j1i];
		dev_uaf[j1i]=0.0f;
	}
}


__global__ void bcond_2_ji()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	
	
	if (i > 0 && j > 0 && i < dev_width && j < dev_height)
	{
		dev_uaf[ji] = dev_uaf[ji] * dev_dum[ji];
		dev_vaf[ji] = dev_vaf[ji] * dev_dvm[ji];
	}
 

}

__global__ void tps_and_other_arrays_4()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	
	
	if (i > 0 && j > 0 && i < dev_width && j < dev_height)
	{
		dev_tps[ji] = sqrtf(dev_uaf[ji]*dev_uaf[ji] + dev_vaf[ji]*dev_vaf[ji]);
		
		/*if (dev_tps[ji] > dev_vmaxl)
		{
			dev_should_stop = 1;
		}*/
		
		dev_ua[ji]=dev_ua[ji]+0.5f*dev_smoth*(dev_uab[ji]-2.0f*dev_ua[ji]+dev_uaf[ji]);
		dev_va[ji]=dev_va[ji]+0.5f*dev_smoth*(dev_vab[ji]-2.0f*dev_va[ji]+dev_vaf[ji]);
		dev_el[ji]=dev_el[ji]+0.5f*dev_smoth*(dev_elb[ji]-2.0f*dev_el[ji]+dev_elf[ji]);
		dev_elb[ji]=dev_el[ji];  // OP
		dev_el[ji]=dev_elf[ji];  // OP
		dev_d[ji]=dev_h[ji]+dev_elf[ji];
		dev_uab[ji]=dev_ua[ji];  // OP
		dev_ua[ji]=dev_uaf[ji];  // OP
		dev_vab[ji]=dev_va[ji];  // OP
		dev_va[ji]=dev_vaf[ji];  // OP
		
	}
	
}



__global__ void adv_fluxes_1()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	int jip1 = ji + 1;
	int jim1 = ji - 1;
	int jm1i = ji - dev_width;
	int jm1im1 = jm1i  - 1;
 
	
	
	if (i > 0 && j > 0)
	{
		if (i < dev_widthm1 && j < dev_height)
		{
			dev_fluxua[ji]=dev_dy[j]*(.125f*((dev_d[jip1]+dev_d[ji])*dev_ua[jip1]
										 +(dev_d[ji]+dev_d[jim1])*dev_ua[ji])
								  *(dev_ua[jip1]+dev_ua[ji])
								  -dev_d[ji]*2.0f*dev_aam2d*(dev_uab[jip1]-dev_uab[ji])/dev_dx[j]);
		}
		
		
		if (i < dev_width && j < dev_height)
		{
			dev_tps[ji]=(dev_d[ji]+dev_d[jim1]+dev_d[jm1i]+dev_d[jm1im1])
			*dev_aam2d
			*((dev_uab[ji]-dev_uab[jm1i])
			  /(4.0f*dev_dy[j])
			  +(dev_vab[ji]-dev_vab[jim1])
			  /(4.0f*dev_dx[j]) );
			
			dev_fluxva[ji]=(.125f*((dev_d[ji]+dev_d[jm1i])*dev_va[ji]
								 +(dev_d[jim1]+dev_d[jm1im1])*dev_va[jim1])
						  *(dev_ua[ji]+dev_ua[jm1i])
						  -dev_tps[ji])*dev_dx[j];
			
		}

	}
}




__global__ void adv_advua_1()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	int jim1 = ji - 1;
 	int jp1i = ji + dev_width;
	
	if (i > 0 && j > 0 && i < dev_widthm1 && j < dev_heightm1)
	{
		dev_advua[ji]=(dev_fluxua[ji]-dev_fluxua[jim1]
					 +dev_fluxva[jp1i]-dev_fluxva[ji])/dev_aru[j];
		
	}
}

__global__ void adv_fluxes_2()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	int jim1 = ji - 1;
	int jp1i = ji + dev_width;
	int jm1i = ji - dev_width;
	int jm1im1 = jm1i  - 1;
 
	
	
	if (i > 0 && j > 0)
	{
		if (i < dev_width && j < dev_heightm1)
		{
			dev_fluxva[ji]=dev_dx[j]*(.125f*((dev_d[jp1i]+dev_d[ji])
										 *dev_va[jp1i]+(dev_d[ji]+dev_d[jm1i])*dev_va[ji])
								  *(dev_va[jp1i]+dev_va[ji])
								  -dev_d[ji]*2.0f*dev_aam2d*(dev_vab[jp1i]-dev_vab[ji])/dev_dy[j]);
		}
		
		
		if (i < dev_width && j < dev_height)
		{
			dev_fluxua[ji]=(.125f*((dev_d[ji]+dev_d[jim1])*dev_ua[ji]
								 +(dev_d[jm1i]+dev_d[jm1im1])*dev_ua[jm1i])*
						  (dev_va[jim1]+dev_va[ji])
						  -dev_tps[ji])*dev_dy[j];
			
		}
		
	}
}


__global__ void adv_advva_2()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	int jip1 = ji + 1;
	int jm1i = ji - dev_width;
	
	if (i > 0 && j > 0 && i < dev_widthm1 && j < dev_heightm1)
	{
		dev_advva[ji]=(dev_fluxua[jip1]-dev_fluxua[ji]
					 +dev_fluxva[ji]-dev_fluxva[jm1i])/dev_arv[j];
		
	}
}

__global__ void adv_bot_3()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
 	int jp1i = ji + dev_width;
 	int jip1 = ji + 1;
 	int jim1 = ji - 1;
 	int jm1i = ji - dev_width;
 
 	int jp1im1 = jp1i - 1;
 	int jm1ip1 = jm1i + 1;
	
	if (i > 0 && j > 0 && i < dev_widthm1 && j < dev_heightm1)
	{
		dev_wubot[ji]=-0.5f*(dev_cbc[ji]+dev_cbc[jim1])
		*sqrtf(dev_uab[ji]*dev_uab[ji]+powf(0.25f*(dev_vab[ji]
											   +dev_vab[jp1i]+dev_vab[jim1]+dev_vab[jp1im1]), 2) )*dev_uab[ji];
		
		dev_wvbot[ji]=-0.5f*(dev_cbc[ji]+dev_cbc[jm1i])
		*sqrtf(powf(.25e0*(dev_uab[ji]+dev_uab[jip1]
						   +dev_uab[jm1i]+dev_uab[jm1ip1]), 2)+dev_vab[ji]*dev_vab[ji])*dev_vab[ji];
	}
}




void KaspyCycler::findElves()
{
	/// DO CUDA REDUCTION instead of copying back to host mem
	
	//hipDeviceSynchronize();
	
	float * h_elf =  &m_fArrays->elf[0][0];
	
	hipError_t err = hipMemcpy(h_elf, g_elf,  m_height * m_width * sizeof(float), hipMemcpyDeviceToHost);
	
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to update host array ELF  (error code %s)!\n", hipGetErrorString(err));
	}
	

	
	float elf_min = h_elf[0];
    float elf_max = h_elf[0];
    
    for (int i=1; i<F_DATA_SIZE; i++)
    {
        if (h_elf[i] > elf_max)
        {
            elf_max = h_elf[i];
        }
        
        if (h_elf[i] < elf_min)
        {
            elf_min = h_elf[i];
        }
    }
	
	m_fVars->elfmin =  elf_min;
	m_fVars->elfmax =  elf_max;
}


void KaspyCycler::sendDataToGPU()
{
	//int ewidth = ((int)m_pitch) / sizeof(float);
	int wm1 = m_width - 1 ;
	int hm1 = m_height - 1 ;
	float dte = (float)m_fVars->dte;
	float dte2 = (float)m_fVars->dte * 2.0f;
	float tide_l = (float)m_fVars->tide_l;
	
	float aam2d = m_fArrays->aam2d;
	
	
	if ( (hipMemcpyToSymbol(HIP_SYMBOL(dev_width), &m_width, sizeof(int))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_height), &m_height, sizeof(int))  == hipSuccess)
		&&(hipMemcpyToSymbol(HIP_SYMBOL(dev_widthm1), &wm1, sizeof(int))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_heightm1), &hm1, sizeof(int))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_dte), &dte, sizeof(float))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_dte2), &dte2, sizeof(float))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_tide_l), &tide_l, sizeof(float))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_aam2d), &aam2d, sizeof(float))  == hipSuccess)
		//&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ewidth), &ewidth,  sizeof(int))  == hipSuccess)
		)
	{
		printf("GPU constant memory filled\n");
		
		
		//int test_i = 0;
		//int test_f = 0;
		
		
	}
	else
	{
		printf("GPU memory copy error (error code %s)!\n", hipGetErrorString(hipGetLastError()));
	}
	
	
	size_t s_data_size =  m_height * m_width *  sizeof(float);
	
	if ( (hipMemcpy(g_fbu,&m_fFloats->fbu[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fbv,&m_fFloats->fbv[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_ffu,&m_fFloats->ffu[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_ffv,&m_fFloats->ffv[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fb,&m_fFloats->fb[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_ff,&m_fFloats->ff[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fxb,&m_fFloats->fxb[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fxf,&m_fFloats->fxf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fyb,&m_fFloats->fyb[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fyf,&m_fFloats->fyf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_wusurf,&m_fArrays->wusurf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_wvsurf,&m_fArrays->wvsurf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_dum,&m_fArrays->dum[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_dvm,&m_fArrays->dvm[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_d, &m_fArrays->d[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		
		&& (hipMemcpy(g_fluxua,&m_fArrays->fluxua[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fluxva,&m_fArrays->fluxva[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		
		&& (hipMemcpy(g_ua,&m_fArrays->ua[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_va,&m_fArrays->va[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_uab,&m_fArrays->uab[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_vab,&m_fArrays->vab[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_uaf,&m_fArrays->uaf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_vaf,&m_fArrays->vaf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_el,&m_fArrays->el[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_elb,&m_fArrays->elb[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_elf,&m_fArrays->elf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fsm,&m_fArrays->fsm[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_tps,&m_fArrays->tps[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_advua,&m_fArrays->advua[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_advva,&m_fArrays->advva[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_wubot,&m_fArrays->wubot[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_wvbot,&m_fArrays->wvbot[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_cbc,&m_fArrays->cbc[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_h,&m_fArrays->h[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		
		
		&& (hipMemcpy(g_cor, &m_fArrays->cor[0], m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_aru, &m_fArrays->aru[0], m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_arv, &m_fArrays->arv[0],  m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_art, &m_fArrays->art[0],  m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_dx, &m_fArrays->dx[0], m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_dy, &m_fArrays->dy[0], m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess))
		
	{
		printf("GPU memory filled\n");
	}
	else
	{
		printf("GPU memory copy error!\n");

	}
	
	
}

void KaspyCycler::getDataToCPU()
{
    hipDeviceSynchronize();
}




void KaspyCycler::makeWsurf()
{
    m_fVars->timeh6 = (m_fVars->timeh / m_fVars->dht) + 1.0f;

    float timeh6 = (float)m_fVars->timeh6;
	
    int pressSize = m_fWindData->kx * m_fWindData->ky;
    int windUSize = m_fWindData->kxu * m_fWindData->kyu;
    int windVSize = m_fWindData->kxv * m_fWindData->kyv;
	
	
	float ftim = fmodf(timeh6, 1.0f);
	
	
	int threadsPerBlock = 64;
	
	int blocksPerGridJ = (m_height + threadsPerBlock - 1) / threadsPerBlock;
	int blocksPerGridI = (m_width + threadsPerBlock - 1) / threadsPerBlock;
	
	dim3 threadsPerSquareBlock(16, 16);
	
	dim3 numSquareBlocks((m_width + threadsPerSquareBlock.x - 1) / threadsPerSquareBlock.x, (m_height + threadsPerSquareBlock.y - 1) / threadsPerSquareBlock.y);
	
	
    
    itime6 = (int)timeh6;

    if (itime6 > itime6_old)
    {
		hipDeviceSynchronize();
		
        itime6_old = itime6;
		
		
		if ( (hipMemcpy(g_fxb,g_fxf, F_DATA_SIZE * sizeof(float), hipMemcpyDeviceToDevice) == hipSuccess)
			&& (hipMemcpy(g_fyb,g_fyf, F_DATA_SIZE * sizeof(float), hipMemcpyDeviceToDevice) == hipSuccess)
			&& (hipMemcpy(g_fb,g_ff, F_DATA_SIZE * sizeof(float), hipMemcpyDeviceToDevice) == hipSuccess)
			&& (hipMemcpy(g_fbu,g_ffu, F_DATA_SIZE * sizeof(float), hipMemcpyDeviceToDevice) == hipSuccess)
			&& (hipMemcpy(g_fbv,g_ffv, F_DATA_SIZE * sizeof(float), hipMemcpyDeviceToDevice) == hipSuccess)
			)
		{
			//printf("ff arrays reset\n");
		}
		else
		{
			printf("GPU memory copy error!\n");
		}
			
			

		//size_t s_p_width = m_fWindData->kx * sizeof(float);

       // memcpy(g_press0, m_press + (itime6 - 1) * pressSize, pressSize * sizeof(float));
		
		if ( (hipMemcpy(g_press0, m_press + (itime6 - 1) * pressSize, pressSize * sizeof(float), hipMemcpyHostToDevice) == hipSuccess) )
		{
			//printf("pressure data copied \n");
		}
		else
		{
			printf("GPU memory copy error!\n");
		}
		
		
		
		getWindPressure('p');

		

		//size_t s_wu_width = m_fWindData->kxu * sizeof(float);
		
		//memcpy(g_uwd0, m_uwd + (itime6 - 1) * windUSize, windUSize * sizeof(float));
		
		if ( (hipMemcpy(g_uwd0, m_uwd + (itime6 - 1) * windUSize, windUSize * sizeof(float), hipMemcpyHostToDevice) == hipSuccess) )
		{
			//printf("wind U data copied \n");
		}
		else
		{
			printf("GPU memory copy error!\n");
		}
		
		
		getWindPressure('u');

        //memcpy(g_vwd0, m_vwd + (itime6 - 1) * windVSize, windVSize * sizeof(float));
		//size_t s_wv_width = m_fWindData->kxv * sizeof(float);
		
		
		if ( (hipMemcpy(g_vwd0, m_vwd + (itime6 - 1) * windVSize, windVSize * sizeof(float), hipMemcpyHostToDevice) == hipSuccess) )
		{
			//printf("wind V data copied \n");
		}
		else
		{
			printf("GPU memory copy error!\n");
		}
		
		
		getWindPressure('v');

		hipDeviceSynchronize();
	}

	
	surf_and_flux_1<<<numSquareBlocks, threadsPerSquareBlock>>>(ftim);

	if (hipGetLastError() != hipSuccess)
	{
		printf("error calling surf_and_flux_1 kernel! \n");
	}
	
	elf_and_flux_2<<<numSquareBlocks, threadsPerSquareBlock>>>();
	
	if (hipGetLastError() != hipSuccess)
	{
		printf("error calling elf_and_flux_2 kernel! \n");
	}

	/// BCOND 1

	
	bcond_1_j<<< blocksPerGridJ, threadsPerBlock>>>();
	
	if (hipGetLastError() != hipSuccess)
	{
		printf("error calling bcond_1_j kernel! \n");
	}
	
	bcond_1_i<<< blocksPerGridI, threadsPerBlock>>>();
	
	if (hipGetLastError() != hipSuccess)
	{
		printf("error calling bcond_1_i kernel! \n");
	}
	
	bcond_1_ji<<< numSquareBlocks, threadsPerSquareBlock>>>();
	
	if (hipGetLastError() != hipSuccess)
	{
		printf("error calling bcond_1_ji kernel! \n");
	}


	
	if (m_fVars->iint % 10 == 0)
	{
		//ADVAVE()
		//       ADVUA=0 ?
		//		FLUXUA=0 ?

		
		adv_fluxes_1<<< numSquareBlocks, threadsPerSquareBlock>>>();
		
		if (hipGetLastError() != hipSuccess)
		{
			printf("error calling adv_fluxes_1 kernel! \n");
		}
		
		adv_advua_1<<< numSquareBlocks, threadsPerSquareBlock>>>();
		
		if (hipGetLastError() != hipSuccess)
		{
			printf("error calling adv_advua_1 kernel! \n");
		}
		
		adv_fluxes_2<<< numSquareBlocks, threadsPerSquareBlock>>>();
		
		if (hipGetLastError() != hipSuccess)
		{
			printf("error calling adv_fluxes_2 kernel! \n");
		}
		
		adv_advva_2<<< numSquareBlocks, threadsPerSquareBlock>>>();
		
		if (hipGetLastError() != hipSuccess)
		{
			printf("error calling adv_advva_2 kernel! \n");
		}
		
		
		 adv_bot_3<<< numSquareBlocks, threadsPerSquareBlock>>>();
		
		if (hipGetLastError() != hipSuccess)
		{
			printf("error calling adv_bot_3 kernel! \n");
		}
	

		// END ADVAVE();
	}
	
	
	uaf_and_vaf_3<<<numSquareBlocks, threadsPerSquareBlock>>>();
	
	if (hipGetLastError() != hipSuccess)
	{
		printf("error calling uaf_and_vaf_3 kernel! \n");
	}

	
	 bcond_2_j<<< blocksPerGridJ, threadsPerBlock>>>();
	
	if (hipGetLastError() != hipSuccess)
	{
		printf("error calling bcond_2_j kernel! \n");
	}
	
	 bcond_2_i<<< blocksPerGridI, threadsPerBlock>>>();
	
	if (hipGetLastError() != hipSuccess)
	{
		printf("error calling bcond_2_i kernel! \n");
	}
	
	bcond_2_ji<<< numSquareBlocks, threadsPerSquareBlock>>>();
	
	if (hipGetLastError() != hipSuccess)
	{
		printf("error calling bcond_2_ji kernel! \n");
	}
	
	tps_and_other_arrays_4<<<numSquareBlocks, threadsPerSquareBlock>>>();
	
	if (hipGetLastError() != hipSuccess)
	{
		printf("error calling tps_and_other_arrays_4 kernel! \n");
	}


}




void KaspyCycler::getWindPressure(char uv)
{
	int kx, ky, kd, nx, ny;//, nd;
	float * p;
	float * px;
	float * py;
	float * pk;
	float xki, xka, yki, yka, xmi, xma, ymi, yma;
	float pkkd[50][50];
	float cd[50][50][4][4];
	
	float * pkk = &pkkd[0][0];
	float * c = &cd[0][0][0][0];
	
	if (uv == 'u')
	{
		kx = m_fWindData->kxu;
		ky = m_fWindData->kyu;
		pk = g_uwd0;
		
		xki = m_fWindData->xkui;
		xka = m_fWindData->xkua;
		yki = m_fWindData->ykui;
		yka = m_fWindData->ykua;
		
		p = g_ffu;
	}
	else if (uv == 'v')
	{
		kx = m_fWindData->kxv;
		ky = m_fWindData->kyv;
		pk = g_vwd0;
		
		xki = m_fWindData->xkvi;
		xka = m_fWindData->xkva;
		yki = m_fWindData->ykvi;
		yka = m_fWindData->ykva;
		
		p = g_ffv;
	}
	else if (uv == 'p')
	{
		kx = m_fWindData->kx;
		ky = m_fWindData->ky;
		//float kd = kx;
		pk = g_press0;
		xki = m_fWindData->xki;
		xka = m_fWindData->xka;
		yki = m_fWindData->yki;
		yka = m_fWindData->yka;

		
		p = g_ff;
		px = g_fxf;
		py = g_fyf;
	}
	else
	{
		// don't know what to do
		return;
	}
	
	kd = kx;
	
	nx = F_DATA_WIDTH;
	ny = F_DATA_HEIGHT;
	//nd = F_DATA_WIDTH;
	
	xmi = m_fVars->xmi;
	xma = m_fVars->xma;
	ymi = m_fVars->ymi;
	yma = m_fVars->yma;
	
	float c1=3.1415926/180.0;
	float c2=111111.0f;
	
	
	float dky=(yka-yki)/(ky-1.0f);
	float  dkx=(xka-xki)/(kx-1.0f);
 
	float dy=(yma-ymi)/(ny-1.0f);
	float dx=(xma-xmi)/(nx-1.0f);
	
	
	for (int j=1; j<=ky; j++ )
	{
		for (int i=1; i<=kx; i++ )
		{
			pkk[j * 50 + i] = pk[(j - 1) * kd + i - 1];
		}
	}

	
	for (int j=1; j<=ky; j++ )
	{
		pkk[j*50+0] = 2.0f*pkk[j*50+1] - pkk[j*50+2];
		pkk[j*50+kx+1] = 2.0f*pkk[j*50+kx] - pkk[j*50+kx-1];
	}
	
	
	for (int i=0; i<=(kx+1); i++ )
	{
		pkk[0*50+i] = 2.0f*pkk[1*50+i] - pkk[2*50+i];
		pkk[(ky+1)*50+i] = 2.0f*pkk[ky*50+i] - pkk[(ky-1)*50+i];
	}
	
	getbicubic(kx + 2,ky + 2, 50, pkk,c);
	
	for (int j=0; j<ny; j++ )
	{
		float y = ymi + j*dy;
		int j0 = (int)((y - yki)/dky);
		
		if (j0 < 0)
		{
			j0 = 0;
		}
		
		if (j0 > ky-2)
		{
			j0 = ky-2;
		}
		
		float u = (y - (yki + j0*dky))/dky;
		
		for (int i=0; i<nx; i++ )
		{
			float x = xmi + i * dx;
			int i0 = (int)((x - xki)/dkx);
			
			if (i0 < 0) i0 = 0;
			
			if (i0 > kx-2) i0 = kx-2;
			
			float t = ( x - (xki + i0*dkx) )/dkx;
			
			float ay = 0.0f;
			float a2 = 0.0f;
			float a1 = 0.0f;
			
			int ji = j * nx + i;
			
			for (int k=3; k>=0; k-- )
			{
				ay = t*ay+((c[j0 * 800 + i0 * 16 + 3 * 4 + k] * u + c[j0 * 800 + i0 * 16 + 2 * 4 + k])*u
						   + c[j0 * 800 + i0 * 16 + 1 * 4 + k])*u + c[j0 * 800 + i0 * 16 + 0 * 4 + k];
			}
			
			if (uv == 'p')
			{
				for (int k=3; k>=0; k-- )
				{
					a2 = t*a2 + (3.0f*c[j0 * 800 + i0 * 16 + 3 * 4 + k]*u
								 + 2.0f*c[j0 * 800 + i0 * 16 + 2 * 4 + k])*u+c[j0 * 800 + i0 * 16 + 1 * 4 + k];
					
					a1 = u*a1 + (3.0f*c[j0 * 800 + i0 * 16 + k * 4 + 3]*t +
								 2.0f*c[j0 * 800 + i0 * 16 + k * 4 + 2])*t+c[j0 * 800 + i0 * 16 + k * 4 + 1];
					
				}
				
				a1 = a1/dkx/c2/cosf(c1*y);
				a2 = a2/dky/c2;
				
				px[ji] = a1;
				py[ji] = a2;
			}
			
			p[ji] = ay;
			
		}
		
	}
	

}







void getbicubic(int nx, int ny, int nd, float * z, float * c)
{
	float d1 = 1.0f;
	float d2 = 1.0f;
	
	float y[4];
	float y1[4];
	float y2[4];
	float y12[4];
	float cc[4][4];
	
	
	for (int j=1; j<ny-2; j++ )
	{
		for (int i=1; i<nx-2; i++ )
		{
			/*
			 Y(1)=Z(I,J)
			 Y(2)=Z(I+1,J)
			 Y(3)=Z(I+1,J+1)
			 Y(4)=Z(I,J+1)
			 */
			y[0] = z[j * nd + i];
			y[1] = z[j * nd + i + 1];
			y[2] = z[(j+1) * nd + i + 1];
			y[3] = z[(j+1) * nd + i];
			
			/*
			 Y1(1)=0.5*(Z(I+1,J)-Z(I-1,J))
			 Y1(4)=0.5*(Z(I+1,J+1)-Z(I-1,J+1))
			 Y1(2)=0.5*(Z(I+2,J)  -Z(I,J))
			 Y1(3)=0.5*(Z(I+2,J+1)-Z(I,J+1))
			 */
			y1[0] = 0.5f * (z[j * nd + i + 1] - z[j * nd + i - 1]);
			y1[3] = 0.5f * (z[(j+1) * nd + i + 1] - z[(j+1) * nd + i - 1]);
			y1[1] = 0.5f * (z[j * nd + i + 2] - z[j * nd + i]);
			y1[2] = 0.5f * (z[(j+1) * nd + i + 2] - z[(j+1) * nd + i]);

			
			/*
			 Y2(1)=0.5*(Z(I,J+1)  -Z(I,J-1))
			 Y2(2)=0.5*(Z(I+1,J+1)-Z(I+1,J-1))
			 Y2(3)=0.5*(Z(I+1,J+2)-Z(I+1,J))
			 Y2(4)=0.5*(Z(I,J+2)-Z(I,J))
			 */
			y2[0] = 0.5f * (z[(j+1) * nd + i] - z[(j-1) * nd + i]);
			y2[1] = 0.5f * (z[(j+1) * nd + i + 1] - z[(j-1) * nd + i + 1]);
			y2[2] = 0.5f * (z[(j+2) * nd + i + 1] - z[(j) * nd + i + 1]);
			y2[3] = 0.5f * (z[(j+2) * nd + i] - z[j * nd + i]);
			
			
			/*
			 Y12(1)=0.25*(Z(I+1,J+1)-Z(I+1,J-1)-Z(I-1,J+1)+Z(I-1,J-1))
			 Y12(2)=0.25*(Z(I+2,J+1)-Z(I+2,J-1)-Z(I,J+1)+Z(I,J-1))
			 Y12(3)=0.25*(Z(I+2,J+2)-Z(I+2,J)-Z(I,J+2)+Z(I,J))
			 Y12(4)=0.25*(Z(I+1,J+2)-Z(I+1,J)-Z(I-1,J+2)+Z(I-1,J))
			 */
			y12[0] = 0.25f * (z[(j+1) * nd + i + 1] - z[(j-1) * nd + i + 1]
							  - z[(j+1) * nd + i - 1] + z[(j-1) * nd + i - 1]);
			y12[1] = 0.25f * (z[(j+1) * nd + i + 2] - z[(j-1) * nd + i + 2]
							  - z[(j+1) * nd + i] + z[(j-1) * nd + i]);
			y12[2] = 0.25f * (z[(j+2) * nd + i + 2] - z[(j) * nd + i + 2]
							  - z[(j+2) * nd + i] + z[j * nd + i]);
			y12[3] = 0.25f * (z[(j+2) * nd + i + 1] - z[(j) * nd + i + 1]
							  - z[(j+2) * nd + i -1] + z[(j) * nd + i -1]);
	
			
			bcucof(&y[0],&y1[0],&y2[0],&y12[0],d1,d2,&cc[0][0]);
			
			for (int k=0; k<4; k++ )
			{
				for (int l=0; l<4; l++ )
				{
					//printf("\nk is %d l is %d\n", k, l);
					c[(j-1)* 800 + (i-1) * 16 + l * 4 + k ] = cc[l][k];
				}
			}
			
			
		}
	 }
	
}



void getbicubic_g(int nx, int ny, int nd, float * z, float * c)
{	
	
	dim3 threadsPerSquareBlock(8, 8);
	
	dim3 numSquareBlocks(((nx - 2) + threadsPerSquareBlock.x - 1) / threadsPerSquareBlock.x, ((ny - 2) + threadsPerSquareBlock.y - 1) / threadsPerSquareBlock.y);
	
	
	dev_bucubic<<<numSquareBlocks, threadsPerSquareBlock>>>(nx, ny, nd, z, c);
	
}




void bcucof(float * y,float * y1,float * y2, float * y12,float d1,float d2,float * cc)
{
	float xx;
	float cl[16];
	
	float x[16];
	
	float wt[] = {
		1,0,-3,2,0,0,0,0,-3,0,9,-6,2,0,-6,4,
		0,0,0,0,0,0,0,0,3,0,-9,6,-2,0,6,-4,
		0,0,0,0,0,0,0,0,0,0,9,-6,0,0,-6,4,
		0,0,3,-2,0,0,0,0,0,0,-9,6,0,0,6,-4,
		0,0,0,0,1,0,-3,2,-2,0,6,-4,1,0,-3,2,
		0,0,0,0,0,0,0,0,-1,0,3,-2,1,0,-3,2,
		0,0,0,0,0,0,0,0,0,0,-3,2,0,0,3,-2,
		0,0,0,0,0,0,3,-2,0,0,-6,4,0,0,3,-2,
		0,1,-2,1,0,0,0,0,0,-3,6,-3,0,2,-4,2,
		0,0,0,0,0,0,0,0,0,3,-6,3,0,-2,4,-2,
		0,0,0,0,0,0,0,0,0,0,-3,3,0,0,2,-2,
		0,0,-1,1,0,0,0,0,0,0,3,-3,0,0,-2,2,
		0,0,0,0,0,1,-2,1,0,-2,4,-2,0,1,-2,1,
		0,0,0,0,0,0,0,0,0,-1,2,-1,0,1,-2,1,
		0,0,0,0,0,0,0,0,0,0,1,-1,0,0,-1,1,
		0,0,0,0,0,0,-1,1,0,0,2,-2,0,0,-1,1
 	};
	
	//float d1 = *pd1;
	//float d2 = *pd2;
	
	
	float d1d2 = d1 * d2;

	for (int i=0; i<4; i++ )
	{
		x[i] = y[i];
		x[i + 4] = y1[i] * d1;
		x[i + 8] = y2[i] * d2;
		x[i + 12] = y12[i] * d1d2;
	}
	
	for (int i=0; i<16; i++ )
	{
		xx = 0.0f;
		
		for (int k=0; k<16; k++ )
		{
			xx += wt[i + k*16] * x[k];
		}
		
		cl[i] = xx;
	}
	
	int l = 0;
	
	for (int i=0; i<4; i++ )
	{
		for (int j=0; j<4; j++ )
		{
			cc[j*4 + i] = cl[l++];
		}
	}
	
}






int KaspyCycler::init_device()
{
	int device_count = 0;
	
	if (m_gpu_device >= 0)
	{
		// already initialized
		printf("CUDA device is already initiaized\n");
		
		return m_gpu_device;
	}
	
	
	hipGetDeviceCount(&device_count);
	
	for (int i = 0 ; i < device_count ; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		
		if (properties.major > 1 || (properties.major == 1 && properties.minor >= 1))
		{
			m_gpu_device = i;
			
			printf("Running on GPU %d (%s) \n",i ,properties.name);
			break;
		}
		else
		{
			printf("GPU %d (%s) does not support CUDA Dynamic Parallelism\n", i ,properties.name);
		}
	}
	
	
	if (m_gpu_device == -1)
	{
		printf("No suitable device found!\n");
		return m_gpu_device;
	}
	
	if (hipSetDevice(m_gpu_device) == hipSuccess)
	{
		printf("device set OK\n");
	}
	else
	{
		printf("unable to set device!\n");
		m_gpu_device = -1;
	}
	
	
	
	// Allocate GPU memory.
	if ( (hipMallocManaged((void **)&g_fbu, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fbv, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_ffu, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_ffv, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fb, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_ff, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fxb, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fxf, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fyb, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fyf, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_wusurf, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_wvsurf, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_dum, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_dvm, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_d, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		
		&& (hipMallocManaged((void **)&g_fluxua, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fluxva, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		
		&& (hipMallocManaged((void **)&g_ua, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_va, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_uab, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_vab, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_uaf, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_vaf, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_el, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_elb, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_elf, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fsm, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_tps, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_advua, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_advva, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_wubot, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_wvbot, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_cbc, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_h, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		
		&& (hipMallocManaged((void **)&g_press0,  m_fWindData->ky *  m_fWindData->kx * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		
		&& (hipMallocManaged((void **)&g_uwd0, m_fWindData->kyu * m_fWindData->kxu * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_vwd0, m_fWindData->kyv * m_fWindData->kxv * sizeof(float), hipMemAttachGlobal) == hipSuccess)

		
		&& (hipMallocManaged((void **)&g_cor, m_height * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_aru, m_height * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_arv, m_height * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_art, m_height * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		
		&& (hipMallocManaged((void **)&g_dx, m_height * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_dy, m_height * sizeof(float), hipMemAttachGlobal) == hipSuccess))
	{
		printf("GPU memory allocated\n");
		
	}
	else
	{
		printf("GPU memory allocation error!\n");
		deinit_device();
		return m_gpu_device;
	}
	
	
	
	if ( (hipMemcpyToSymbol(HIP_SYMBOL(dev_fbu), &g_fbu, sizeof(g_fbu)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fbv), &g_fbv, sizeof(g_fbv)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ffu), &g_ffu, sizeof(g_ffu)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ffv), &g_ffv, sizeof(g_ffv)) == hipSuccess)
		
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fxb), &g_fxb, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fxf), &g_fxf, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fyb), &g_fyb, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fyf), &g_fyf, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fb), &g_fb, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ff), &g_ff, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_wusurf), &g_wusurf, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_wvsurf), &g_wvsurf, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_dum), &g_dum, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_dvm), &g_dvm, sizeof(float *)) == hipSuccess)

		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_d), &g_d, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_dx), &g_dx, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_dy), &g_dy, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fluxua), &g_fluxua, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fluxva), &g_fluxva, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ua), &g_ua, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_va), &g_va, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_uab), &g_uab, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_vab), &g_vab, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_uaf), &g_uaf, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_vaf), &g_vaf, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_el), &g_el, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_elf), &g_elf, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_elb), &g_elb, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fsm), &g_fsm, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_tps), &g_tps, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_advua), &g_advua, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_advva), &g_advva, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_aru), &g_aru, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_arv), &g_arv, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_art), &g_art, sizeof(float *)) == hipSuccess)

		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_wubot), &g_wubot, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_wvbot), &g_wvbot, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_cbc), &g_cbc, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_cor), &g_cor, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_h), &g_h, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_press0), &g_press0, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_uwd0), &g_uwd0, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_vwd0), &g_vwd0, sizeof(float *)) == hipSuccess)
		)
	{
		printf("Device pointers initialized\n");
	}
	else
	{
		printf("Pointer init error!\n");
		deinit_device();
		return m_gpu_device;
	}
	
	
	
	return m_gpu_device;
}


void KaspyCycler::deinit_device()
{
	if (m_gpu_device >=0)
	{
		
		if (g_fbu)
		{
			hipFree(g_fbu);
		}
		
		if (g_fbv)
		{
			hipFree(g_fbv);
		}
		
		if (g_ffu)
		{
			hipFree(g_ffu);
		}
		
		if (g_ffv)
		{
			hipFree(g_ffv);
		}
		
		
		if (g_fb)
		{
			hipFree(g_fb);
		}
		
		if (g_ff)
		{
			hipFree(g_ff);
		}
		
		
		if (g_fxb)
		{
			hipFree(g_fxb);
		}
		
		if (g_fxf)
		{
			hipFree(g_fxf);
		}
		
		if (g_fyb)
		{
			hipFree(g_fyb);
		}
		
		if (g_fyf)
		{
			hipFree(g_fyf);
		}
		
		if (g_wusurf)
		{
			hipFree(g_wusurf);
		}
		
		if (g_wvsurf)
		{
			hipFree(g_wvsurf);
		}
		
		if (g_dum)
		{
			hipFree(g_dum);
		}
		
		if (g_dvm)
		{
			hipFree(g_dvm);
		}
		
		if (g_d)
		{
			hipFree(g_d);
		}
		
		if (g_fluxua)
		{
			hipFree(g_fluxua);
		}
		
		if (g_fluxva)
		{
			hipFree(g_fluxva);
		}
		
		if (g_ua)
		{
			hipFree(g_ua);
		}
		
		if (g_va)
		{
			hipFree(g_va);
		}
		
		if (g_uaf)
		{
			hipFree(g_uaf);
		}
		
		if (g_vaf)
		{
			hipFree(g_vaf);
		}
		
		if (g_uab)
		{
			hipFree(g_uab);
		}
		
		if (g_vab)
		{
			hipFree(g_vab);
		}
		
		if (g_el)
		{
			hipFree(g_el);
		}
		
		if (g_elb)
		{
			hipFree(g_elb);
		}
		
		if (g_elf)
		{
			hipFree(g_elf);
		}
		
		if (g_fsm)
		{
			hipFree(g_fsm);
		}
		
		if (g_tps)
		{
			hipFree(g_tps);
		}
		
		if (g_advua)
		{
			hipFree(g_advua);
		}
		
		if (g_advva)
		{
			hipFree(g_advva);
		}
		
		if (g_aru)
		{
			hipFree(g_aru);
		}
		
		if (g_arv)
		{
			hipFree(g_arv);
		}
		
		if (g_art)
		{
			hipFree(g_art);
		}
		
		if (g_wubot)
		{
			hipFree(g_wubot);
		}
		
		if (g_wvbot)
		{
			hipFree(g_wvbot);
		}
		
		if (g_cbc)
		{
			hipFree(g_cbc);
		}
		
		if (g_h)
		{
			hipFree(g_h);
		}
		
		if (g_cor)
		{
			hipFree(g_cor);
		}
		
		if (g_press0)
		{
			hipFree(g_press0);
		}
		
		
		if (g_uwd0)
		{
			hipFree(g_uwd0);
		}
		
		if (g_vwd0)
		{
			hipFree(g_vwd0);
		}
		
		if (hipDeviceReset() == hipSuccess)
		{
			printf("GPU device reset ok\n");
		}
		
		m_gpu_device = -1;
	}
}




