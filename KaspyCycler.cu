#include "hip/hip_runtime.h"
//
//  KaspyCycler.cpp
//  kaspy_cuda
//
//  Created by Andrei Koulikov on 24.05.16.
//
//

#include "KaspyCycler.h"


#include "hip/hip_runtime.h"
#include ""



void getbicubic(int nx, int ny, int nd, float * z, float * c);
void bcucof(float * y,float * y1,float * y2, float * y12,float d1,float d2,float * cc);


float grav = 9.806;


float * g_fbu = 0;
float * g_fbv = 0;
float * g_ffu = 0;
float * g_ffv = 0;

float * g_fxb = 0;
float * g_fxf = 0;
float * g_fyb = 0;
float * g_fyf = 0;

float * g_fb = 0;
float * g_ff = 0;

float * g_wusurf = 0;
float * g_wvsurf = 0;

float * g_dum = 0;
float * g_dvm = 0;

float * g_d = 0;
float * g_dx = 0;
float * g_dy = 0;

float * g_fluxua = 0;
float * g_fluxva = 0;

float * g_ua = 0;
float * g_va = 0;

float * g_uab = 0;
float * g_vab = 0;

float * g_uaf = 0;
float * g_vaf = 0;


float * g_el = 0;
float * g_elf = 0;
float * g_elb = 0;

float * g_fsm = 0;

float * g_tps = 0;


float * g_advua = 0;
float * g_advva = 0;

float * g_aru = 0;
float * g_arv = 0;

float * g_wubot = 0;
float * g_wvbot = 0;
float * g_cbc = 0;

float * g_cor = 0;

float * g_h = 0;

float * g_press0 = 0;
float * g_uwd0 = 0;
float * g_vwd0 = 0;


__constant__ __device__  float g_grav = 9.806;

__constant__ __device__  int  g_width;
__constant__ __device__  int  g_height;
__constant__ __device__  int  g_widthm1;
__constant__ __device__  int  g_heightm1;

__constant__ __device__ int g_ewidth;


/*, float * g_fbu, float * g_ffu, float * g_fbv, float * g_ffv, float * g_dum, float * g_dvm, float * g_d, float * g_wusurf, float * g_wvsurf, float * g_fluxua, float * g_fluxva, float * g_dx, float * g_dy, float * g_ua, float * g_va, float * g_fxf, float * g_fyf, float * g_fxb, float *  g_fyb*/

__global__ void surf_and_flux_1(float ftim, float ro_ratio)
{
	/*int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * g_width + i;
 	int jp1i = ji + g_width;
 	int jip1 = ji + 1;
 	int jim1 = ji - 1;
 	int jm1i = ji - g_width;*/
	
	float btim = 1.0f - ftim;
	
	
	/*if (i < g_widthm1 && j < g_heightm1)
	{
		float uw = btim * (g_fbu[ji]) + ftim * (g_ffu[ji]);
		float vw = btim * (g_fbv[ji]) + ftim * (g_ffv[ji]);
		
		float speed = sqrtf(uw*uw + vw*vw);
		float windc = 0.001f * (0.8f + speed * 0.065f) * ro_ratio * speed;
		
		g_wusurf[ji] = -windc * uw *
		0.25f * (g_dum[jp1i]+g_dum[jip1]+g_dum[jim1]+g_dum[jm1i])
		+ 0.5f * (g_d[ji] + g_d[jim1]) * (btim * g_fxb[ji] + ftim * g_fxf[ji]);
		
		g_wvsurf[ji] = -windc * vw *
		0.25f * (g_dvm[jp1i]+g_dvm[jip1]+g_dvm[jim1]+g_dvm[jm1i])
		+ 0.5f * (g_d[ji] + g_d[jm1i]) * (btim * g_fyb[ji] + ftim * g_fyf[ji]);
	}
	
	if (i < g_width && j < g_height)
	{
		g_fluxua[ji] = 0.25f * (g_d[ji] + g_d[jim1]) * (g_dy[j] + g_dy[j] ) * g_ua[ji];
		g_fluxva[ji] = 0.25f * (g_d[ji] + g_d[jm1i]) * (g_dx[j] + g_dx[j-1] ) * g_va[ji];
	}*/
}


/*
 btim = 1.0f - ftim;
 
 for (int j=1; j<m_height; j++ )
 {
 for (int i=1; i<m_width; i++ )
 {
 if ((j<(m_height-1)) && i<(m_width-1))
 {
 ji = j * m_width + i;
 jp1i = ji + m_width;
 jip1 = ji + 1;
 jim1 = ji - 1;
 jm1i = ji - m_width;
 
 uw = btim * (g_fbu[ji]) + ftim * (g_ffu[ji]);
 vw = btim * (g_fbv[ji]) + ftim * (g_ffv[ji]);
 
 speed = sqrtf(uw*uw + vw*vw);
 windc = 0.001f * (0.8f + speed * 0.065f) * ro_ratio * speed;
 
 g_wusurf[ji] = -windc * uw *
 0.25f * (g_dum[jp1i]+g_dum[jip1]+g_dum[jim1]+g_dum[jm1i])
 + 0.5f * (g_d[ji] + g_d[jim1]) * (btim * g_fxb[ji] + ftim * g_fxf[ji]);
 
 g_wvsurf[ji] = -windc * vw *
 0.25f * (g_dvm[jp1i]+g_dvm[jip1]+g_dvm[jim1]+g_dvm[jm1i])
 + 0.5f * (g_d[ji] + g_d[jm1i]) * (btim * g_fyb[ji] + ftim * g_fyf[ji]);
 }
 
 
 
 g_fluxua[ji] = 0.25f * (g_d[ji] + g_d[jim1]) * (g_dy[j] + g_dy[j] ) * g_ua[ji];
 g_fluxva[ji] = 0.25f * (g_d[ji] + g_d[jm1i]) * (g_dx[j] + g_dx[j-1] ) * g_va[ji];
 
 }
 }*/





void KaspyCycler::findElves()
{
	/// DO CUDA REDUCTION instead of copying back to host mem
	
	float * h_elf =  &m_fArrays->elf[0][0];
	
	hipError_t err = hipMemcpy(h_elf, g_elf,  m_height * m_width * sizeof(float), hipMemcpyDeviceToHost);
	
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to update host array ELF  (error code %s)!\n", hipGetErrorString(err));
	}
	

	
	float elf_min = h_elf[0];
    float elf_max = h_elf[0];
    
    for (int i=1; i<F_DATA_SIZE; i++)
    {
        if (h_elf[i] > elf_max)
        {
            elf_max = h_elf[i];
        }
        
        if (h_elf[i] < elf_min)
        {
            elf_min = h_elf[i];
        }
    }
	
	m_fVars->elfmin =  elf_min;
	m_fVars->elfmax =  elf_max;
}


void KaspyCycler::sendDataToGPU()
{
	//int ewidth = ((int)m_pitch) / sizeof(float);
	int wm1 = m_width - 1 ;
	int hm1 = m_height - 1 ;
	
	if ( (hipMemcpyToSymbol(HIP_SYMBOL(g_width), &m_width, sizeof(int))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(g_height), &m_height, sizeof(int))  == hipSuccess)
		&&(hipMemcpyToSymbol(HIP_SYMBOL(g_widthm1), &wm1, sizeof(int))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(g_heightm1), &hm1, sizeof(int))  == hipSuccess)
		//&& (hipMemcpyToSymbol(HIP_SYMBOL(g_ewidth), &ewidth,  sizeof(int))  == hipSuccess)
		)
	{
		printf("GPU constant memory filled\n");
		
		//int test_ewidth = 0;
		
		//hipMemcpyFromSymbol(&test_ewidth, HIP_SYMBOL(g_ewidth), sizeof(int));
		
		
		//printf("pitched width is now %d\n", test_ewidth);
	}
	else
	{
		printf("GPU memory copy error (error code %s)!\n", hipGetErrorString(hipGetLastError()));
	}
	
	
	size_t s_data_size =  m_height * m_width *  sizeof(float);
	
	if ( (hipMemcpy(g_fbu,&m_fFloats->fbu[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fbv,&m_fFloats->fbv[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_ffu,&m_fFloats->ffu[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_ffv,&m_fFloats->ffv[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fb,&m_fFloats->fb[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_ff,&m_fFloats->ff[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fxb,&m_fFloats->fxb[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fxf,&m_fFloats->fxf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fyb,&m_fFloats->fyb[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fyf,&m_fFloats->fyf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_wusurf,&m_fArrays->wusurf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_wvsurf,&m_fArrays->wvsurf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_dum,&m_fArrays->dum[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_dvm,&m_fArrays->dvm[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_d, &m_fArrays->d[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		
		&& (hipMemcpy(g_fluxua,&m_fArrays->fluxua[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fluxva,&m_fArrays->fluxva[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		
		&& (hipMemcpy(g_ua,&m_fArrays->ua[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_va,&m_fArrays->va[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_uab,&m_fArrays->uab[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_vab,&m_fArrays->vab[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_uaf,&m_fArrays->uaf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_vaf,&m_fArrays->vaf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_el,&m_fArrays->el[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_elb,&m_fArrays->elb[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_elf,&m_fArrays->elf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fsm,&m_fArrays->fsm[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_tps,&m_fArrays->tps[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_advua,&m_fArrays->advua[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_advva,&m_fArrays->advva[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_wubot,&m_fArrays->wubot[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_wvbot,&m_fArrays->wvbot[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_cbc,&m_fArrays->cbc[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_h,&m_fArrays->h[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		
		
		&& (hipMemcpy(g_cor, &m_fArrays->cor[0], m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_aru, &m_fArrays->aru[0], m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_arv, &m_fArrays->arv[0],  m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_dx, &m_fArrays->dx[0], m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_dy, &m_fArrays->dy[0], m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess))
		
	{
		printf("GPU memory filled\n");
	}
	else
	{
		printf("GPU memory copy error!\n");

	}
	
	/*g_fbu = &m_fFloats->fbu[0][0];
    g_fbv = &m_fFloats->fbv[0][0];
    g_ffu = &m_fFloats->ffu[0][0];
    g_ffv = &m_fFloats->ffv[0][0];
    
    g_fxb = &m_fFloats->fxb[0][0];
    g_fxf = &m_fFloats->fxf[0][0];
    g_fyb = &m_fFloats->fyb[0][0];
    g_fyf = &m_fFloats->fyf[0][0];
    
    
    g_fb = &m_fFloats->fb[0][0];
    g_ff = &m_fFloats->ff[0][0];
	
	g_h = &m_fArrays->h[0][0];
    
    g_wusurf = &m_fArrays->wusurf[0][0];
    g_wvsurf = &m_fArrays->wvsurf[0][0];
    
    g_dum = &m_fArrays->dum[0][0];
    g_dvm = &m_fArrays->dvm[0][0];
    
    g_d = &m_fArrays->d[0][0];
    g_dx = &m_fArrays->dx[0];
    g_dy = &m_fArrays->dy[0];


    g_fluxua = &m_fArrays->fluxua[0][0];
    g_fluxva = &m_fArrays->fluxva[0][0];
	
	g_advua = &m_fArrays->advua[0][0];
	g_advva = &m_fArrays->advva[0][0];
	
    g_ua = &m_fArrays->ua[0][0];
    g_va = &m_fArrays->va[0][0];

	g_uab = &m_fArrays->uab[0][0];
	g_vab = &m_fArrays->vab[0][0];

	g_uaf = &m_fArrays->uaf[0][0];
	g_vaf = &m_fArrays->vaf[0][0];
	
	
    g_el = &m_fArrays->el[0][0];
    g_elf = &m_fArrays->elf[0][0];
    g_elb = &m_fArrays->elb[0][0];
	
	g_fsm = &m_fArrays->fsm[0][0];
	
	g_tps = &m_fArrays->tps[0][0];
	
	g_aru = &m_fArrays->aru[0];
	g_arv = &m_fArrays->arv[0];

	
	g_wubot = &m_fArrays->wubot[0][0];
	g_wvbot = &m_fArrays->wvbot[0][0];

	g_cbc = &m_fArrays->cbc[0][0];
	
	g_cor = &m_fArrays->cor[0];*/
}

void KaspyCycler::getDataToCPU()
{
    
}




void KaspyCycler::makeWsurf(float ro_ratio)
{
    m_fVars->timeh6 = (m_fVars->timeh / m_fVars->dht) + 1.0f;

    float timeh6 = (float)m_fVars->timeh6;
	
    int pressSize = m_fWindData->kx * m_fWindData->ky;
    int windUSize = m_fWindData->kxu * m_fWindData->kyu;
    int windVSize = m_fWindData->kxv * m_fWindData->kyv;
	
	size_t s_width =  m_width *  sizeof(float);
    
    itime6 = (int)timeh6;

    //ftim = (timeh6 - itime6);
    //btim = 1.0f - ftim;
    
    if (itime6 > itime6_old)
    {
        itime6_old = itime6;
        
        /*memcpy(g_fxb, g_fxf, F_DATA_SIZE * sizeof(float));
        memcpy(g_fyb, g_fyf, F_DATA_SIZE * sizeof(float));
        memcpy(g_fb, g_ff, F_DATA_SIZE * sizeof(float));
        memcpy(g_fbu, g_ffu, F_DATA_SIZE * sizeof(float));
        memcpy(g_fbv, g_ffv, F_DATA_SIZE * sizeof(float));*/
		
		if ( (hipMemcpy(g_fxb,g_fxf, F_DATA_SIZE * sizeof(float), hipMemcpyDeviceToDevice) == hipSuccess)
			&& (hipMemcpy(g_fyb,g_fyf, F_DATA_SIZE * sizeof(float), hipMemcpyDeviceToDevice) == hipSuccess)
			&& (hipMemcpy(g_fb,g_ff, F_DATA_SIZE * sizeof(float), hipMemcpyDeviceToDevice) == hipSuccess)
			&& (hipMemcpy(g_fbu,g_ffu, F_DATA_SIZE * sizeof(float), hipMemcpyDeviceToDevice) == hipSuccess)
			&& (hipMemcpy(g_fbv,g_ffv, F_DATA_SIZE * sizeof(float), hipMemcpyDeviceToDevice) == hipSuccess)
			)
		{
			printf("ff arrays reset\n");
		}
		else
		{
			printf("GPU memory copy error!\n");
		}
			
			

		size_t s_p_width = m_fWindData->kx * sizeof(float);

       // memcpy(g_press0, m_press + (itime6 - 1) * pressSize, pressSize * sizeof(float));
		
		if ( (hipMemcpy(g_press0, m_press + (itime6 - 1) * pressSize, pressSize * sizeof(float), hipMemcpyHostToDevice) == hipSuccess) )
		{
			printf("pressure data copied \n");
		}
		else
		{
			printf("GPU memory copy error!\n");
		}
		
		
		
		getWindPressure('p');

		

		size_t s_wu_width = m_fWindData->kxu * sizeof(float);
		
		//memcpy(g_uwd0, m_uwd + (itime6 - 1) * windUSize, windUSize * sizeof(float));
		
		if ( (hipMemcpy(g_uwd0, m_uwd + (itime6 - 1) * windUSize, windUSize * sizeof(float), hipMemcpyHostToDevice) == hipSuccess) )
		{
			printf("wind U data copied \n");
		}
		else
		{
			printf("GPU memory copy error!\n");
		}
		
		
		getWindPressure('u');

        //memcpy(g_vwd0, m_vwd + (itime6 - 1) * windVSize, windVSize * sizeof(float));
		size_t s_wv_width = m_fWindData->kxv * sizeof(float);
		
		
		if ( (hipMemcpy(g_vwd0, m_vwd + (itime6 - 1) * windVSize, windVSize * sizeof(float), hipMemcpyHostToDevice) == hipSuccess) )
		{
			printf("wind V data copied \n");
		}
		else
		{
			printf("GPU memory copy error!\n");
		}
		
		
		getWindPressure('v');

    }
	
	
    float uw, vw, speed, windc;
    int ji, jp1i, jip1, jim1, jm1i, jp1ip1, jm1im1, jp1im1, jm1ip1, jl, jlm1, j1, j2, j3, jli, jlm1i, j1i, j2i, j3i;
	
	
	float ftim = fmodf(timeh6, 1.0f);
	
	//int threadsPerBlock = 64;
	
	//int blocksPerGridJ = (m_height + threadsPerBlock - 1) / threadsPerBlock;
	//int blocksPerGridI = (m_width + threadsPerBlock - 1) / threadsPerBlock;
	
	/*dim3 threadsPerSquareBlock(4, 4);
	
	dim3 numSquareBlocks((m_width + threadsPerSquareBlock.x - 1) / threadsPerSquareBlock.x, (m_height + threadsPerSquareBlock.y - 1) / threadsPerSquareBlock.y);
	
	hipError_t err = hipSuccess;*/
	
	
	//surf_and_flux_1<<<numSquareBlocks, threadsPerSquareBlock>>>(ftim, ro_ratio);
	setbuf(stdout,NULL);
	printf("before call\n");
	setbuf(stdout,NULL);
	
	//surf_and_flux_1<<<4, 8>>>(ftim, ro_ratio);
	
	setbuf(stdout,NULL);
	printf("after call\n");
	setbuf(stdout,NULL);
	
	/*,  g_fbu,  g_ffu,  g_fbv,  g_ffv,  g_dum,  g_dvm,  g_d,  g_wusurf,  g_wvsurf,  g_fluxua,  g_fluxva,  g_dx,  g_dy,  g_ua,  g_va,  g_fxf,  g_fyf, g_fxb,  g_fyb);*/

    
    /*
    btim = 1.0f - ftim;
    
    for (int j=1; j<m_height; j++ )
    {
        for (int i=1; i<m_width; i++ )
        {
            if ((j<(m_height-1)) && i<(m_width-1))
            {
                ji = j * m_width + i;
                jp1i = ji + m_width;
                jip1 = ji + 1;
                jim1 = ji - 1;
                jm1i = ji - m_width;
                
                uw = btim * (g_fbu[ji]) + ftim * (g_ffu[ji]);
                vw = btim * (g_fbv[ji]) + ftim * (g_ffv[ji]);
                
                speed = sqrtf(uw*uw + vw*vw);
                windc = 0.001f * (0.8f + speed * 0.065f) * ro_ratio * speed;
                
                g_wusurf[ji] = -windc * uw *
                0.25f * (g_dum[jp1i]+g_dum[jip1]+g_dum[jim1]+g_dum[jm1i])
                + 0.5f * (g_d[ji] + g_d[jim1]) * (btim * g_fxb[ji] + ftim * g_fxf[ji]);
                
                g_wvsurf[ji] = -windc * vw *
                0.25f * (g_dvm[jp1i]+g_dvm[jip1]+g_dvm[jim1]+g_dvm[jm1i])
                + 0.5f * (g_d[ji] + g_d[jm1i]) * (btim * g_fyb[ji] + ftim * g_fyf[ji]);
            }
            


            g_fluxua[ji] = 0.25f * (g_d[ji] + g_d[jim1]) * (g_dy[j] + g_dy[j] ) * g_ua[ji];
            g_fluxva[ji] = 0.25f * (g_d[ji] + g_d[jm1i]) * (g_dx[j] + g_dx[j-1] ) * g_va[ji];
            
        }
    }*/
    
    
    /// HERE SHOULD START A NEW CUDA CALL TO KEEP fluxua fluxva synced
	
	
	setbuf(stdout,NULL);
	printf("b call 2\n");
	setbuf(stdout,NULL);
	
    float dte2 = m_fVars->dte * 2.0f;
    
    for (int j=1; j<(m_height-1); j++ )
    {
        float artj = m_fArrays->art[j];
        
        for (int i=1; i<(m_width-1); i++ )
        {
            ji = j * m_width + i;
            jp1i = ji + m_width;
            jip1 = ji + 1;
            
            g_elf[ji] = g_elb[ji] - dte2 *
                (g_fluxua[jip1] - g_fluxua[ji] + g_fluxva[jp1i] - g_fluxva[ji]) /  artj;
            
        }
    }
	
	
	setbuf(stdout,NULL);
	printf("a call 2\n");
	setbuf(stdout,NULL);
 


	/// BCOND 1
	float tide_l = m_fVars->tide_l;
	
	for (int j=1; j<m_height; j++ )
	{
		g_elf[j * m_width + 1] = tide_l;
		g_elf[j * m_width + m_width - 2] = tide_l;
		
		g_elf[j * m_width] = tide_l;
		g_elf[j * m_width + m_width - 1] = tide_l;
	}
	
	for (int i=1; i<m_width; i++ )
	{
		g_elf[i] =  g_elf[i + m_width];
		
		g_elf[i + m_width * (m_height - 1)  ] =  g_elf[i + m_width * (m_height - 2)];
	}
	
	for (int j=1; j<m_height; j++ )
	{
		for (int i=1; i<m_width; i++ )
		{
			ji = j * m_width + i;
			
			g_elf[ji] *= g_fsm[ji];
		}
	}

	
	if (m_fVars->iint % 10 == 0)
	{//ADVAVE()
		//       ADVUA=0
		//		FLUXUA=0
		
		//memset(g_advua, 0, F_DATA_SIZE * sizeof(float));
		//memset(g_fluxua, 0, F_DATA_SIZE * sizeof(float));
		
		
		float aam2d = m_fArrays->aam2d;
		
		for (int j=1; j<m_height; j++ )
		{
			for (int i=1; i<(m_width-1); i++ )
			{
				ji = j * m_width + i;
				jip1 = ji + 1;
				jim1 = ji - 1;
				
				/*g_fluxua[ji] = g_dy[j] * (.125f * ((g_d[ji + 1]+g_d[ji])*g_ua[ji + 1]
						+(g_d[ji]+g_d[ji - 1])*g_ua[ji])
										  *(g_ua[ji + 1]+g_ua[ji])
										  - g_d[ji]*2.0f*aam2d*(g_uab[ji + 1]-g_uab[ji])/g_dx[j]);*/
				g_fluxua[ji]=g_dy[j]*(.125f*((g_d[jip1]+g_d[ji])*g_ua[jip1]
											  +(g_d[ji]+g_d[jim1])*g_ua[ji])
									  *(g_ua[jip1]+g_ua[ji])
									  -g_d[ji]*2.0f*aam2d*(g_uab[jip1]-g_uab[ji])/g_dx[j]);
				
				
			}
		}
		
		
		for (int j=1; j<m_height; j++ )
		{
			for (int i=1; i<m_width; i++ )
			{
				ji = j * m_width + i;
				jp1i = ji + m_width;
				jip1 = ji + 1;
				jim1 = ji - 1;
				jm1i = ji - m_width;
				jm1im1 = jm1i  - 1;
				
				/*g_tps[ji] =(g_d[ji]+g_d[jim1]+g_d[jm1i]+g_d[jm1im1]) *aam2d
				*((g_uab[ji]-g_uab[jm1i]) /(4.0f*g_dy[j])+(g_vab[ji]-g_vab[jim1]) /(4.0f*g_dx[j]) );
				
				g_fluxva[ji]=(.125f*((g_d[ji]+g_d[jm1i])*g_va[ji]
									 +(g_d[jim1]+g_d[jm1im1])*g_va[jim1])
							  *(g_ua[ji]+g_va[jm1i]) - g_tps[ji])*g_dx[j];*/
				
				g_tps[ji]=(g_d[ji]+g_d[jim1]+g_d[jm1i]+g_d[jm1im1])
				*aam2d
				*((g_uab[ji]-g_uab[jm1i])
				  /(4*g_dy[j])
				  +(g_vab[ji]-g_vab[jim1])
				  /(4*g_dx[j]) );
				
				g_fluxva[ji]=(.125f*((g_d[ji]+g_d[jm1i])*g_va[ji]
									  +(g_d[jim1]+g_d[jm1im1])*g_va[jim1])
							  *(g_ua[ji]+g_ua[jm1i])
							  -g_tps[ji])*g_dx[j];
				
			}
		}

		
		for (int j=1; j<(m_height-1); j++ )
		{
			for (int i=1; i<(m_width-1); i++ )
			{
				ji = j * m_width + i;
				jim1 = ji - 1;
				jp1i = ji + m_width;
				
				g_advua[ji]=(g_fluxua[ji]-g_fluxua[jim1]
						   +g_fluxva[jp1i]-g_fluxva[ji])/g_aru[j];
			}
			
		}
		
		//memset(g_advva, 0, F_DATA_SIZE * sizeof(float));
		//memset(g_fluxva, 0, F_DATA_SIZE * sizeof(float));
		
		
		for (int j=1; j<(m_height-1); j++ )
		{
			for (int i=1; i<m_width; i++ )
			{
				ji = j * m_width + i;
				jp1i = ji + m_width;
				jip1 = ji + 1;
				jim1 = ji - 1;
				jm1i = ji - m_width;
				jm1im1 = jm1i  - 1;
				
				
			 	g_fluxva[ji]=g_dx[j]*(.125f*((g_d[jp1i]+g_d[ji])
									       *g_va[jp1i]+(g_d[ji]+g_d[jm1i])*g_va[ji])
									      *(g_va[jp1i]+g_va[ji])
								         -g_d[ji]*2.0f*aam2d*(g_vab[jp1i]-g_vab[ji])/g_dy[j]);
				
			}
		}
		
		
		for (int j=1; j<m_height; j++ )
		{
			for (int i=1; i<m_width; i++ )
			{
				ji = j * m_width + i;
				jp1i = ji + m_width;
				jip1 = ji + 1;
				jim1 = ji - 1;
				jm1i = ji - m_width;
				jm1im1 = jm1i  - 1;
				
				
				g_fluxua[ji]=(.125f*((g_d[ji]+g_d[jim1])*g_ua[ji]
									         +(g_d[jm1i]+g_d[jm1im1])*g_ua[jm1i])*
							                        (g_va[jim1]+g_va[ji])
							  -g_tps[ji])*g_dy[j];
			}
		}

		
		for (int j=1; j<(m_height-1); j++ )
		{
			for (int i=1; i<(m_width-1); i++ )
			{
				ji = j * m_width + i;
				jp1i = ji + m_width;
				jip1 = ji + 1;
				jim1 = ji - 1;
				jm1i = ji - m_width;
				jm1im1 = jm1i  - 1;
				
				g_advva[ji]=(g_fluxua[jip1]-g_fluxua[ji]
							         +g_fluxva[ji]-g_fluxva[jm1i])/g_arv[j];
			}
		}
		
	
		
		for (int j=1; j<(m_height-1); j++ )
		{
			for (int i=1; i<(m_width-1); i++ )
			{
				ji = j * m_width + i;
				jp1i = ji + m_width;
				jip1 = ji + 1;
				jim1 = ji - 1;
				jm1i = ji - m_width;
				jm1im1 = jm1i  - 1;
				
				jp1im1 = jp1i - 1;
				jm1ip1 = jm1i + 1;

				g_wubot[ji]=-0.5e0*(g_cbc[ji]+g_cbc[jim1])
				     *sqrtf(g_uab[ji]*g_uab[ji]+powf(.25e0*(g_vab[ji]
											  +g_vab[jp1i]+g_vab[jim1]+g_vab[jp1im1]), 2) )*g_uab[ji];
				
				g_wvbot[ji]=-0.5e0*(g_cbc[ji]+g_cbc[jm1i])
				    *sqrtf(powf(.25e0*(g_uab[ji]+g_uab[jip1]
								  +g_uab[jm1i]+g_uab[jm1ip1]), 2)+g_vab[ji]*g_vab[ji])*g_vab[ji];
				
			}
		}
		
		
		// END ADVAVE();
	}
	
	float alpha =  0.225f;
	float dte = m_fVars->dte;
	
	for (int j=1; j<(m_height-1); j++ )
	{
		for (int i=1; i<m_width; i++ )
		{
			ji = j * m_width + i;
			jp1i = ji + m_width;
			jip1 = ji + 1;
			jim1 = ji - 1;
			jm1i = ji - m_width;
			jm1im1 = jm1i  - 1;
			jp1im1 = jp1i - 1;
			jm1ip1 = jm1i + 1;
			
			float uaf1= g_advua[ji]   -0.25f*(g_cor[j]*g_d[ji]*(g_va[jp1i]+g_va[ji])
					                 +g_cor[j]*g_d[jim1]*(g_va[jp1im1]+g_va[jim1]) )
			         +0.5f*grav*g_dy[j]/g_aru[j]*(g_d[ji]+g_d[jim1])
			             *( (1.0f-2.0f*alpha)*(g_el[ji]-g_el[jim1])
							            +alpha*(g_elb[ji]-g_elb[jim1]+g_elf[ji]-g_elf[jim1]) )
			+g_wusurf[ji]-g_wubot[ji];
			
			g_uaf[ji]=
			         ((g_h[ji]+g_elb[ji]+g_h[jim1]+g_elb[jim1])*g_uab[ji]
					                   -4.e0*dte*uaf1)  /(g_h[ji]+g_elf[ji]+g_h[jim1]+g_elf[jim1]);
			
		}
	}
	
	
	for (int j=1; j<m_height; j++ )
	{
		for (int i=1; i<(m_width-1); i++ )
		{
			ji = j * m_width + i;
			jp1i = ji + m_width;
			jip1 = ji + 1;
			jim1 = ji - 1;
			jm1i = ji - m_width;
			jm1im1 = jm1i  - 1;
			jp1im1 = jp1i - 1;
			jm1ip1 = jm1i + 1;
			
			float vaf1=g_advva[ji]
			+.25f*(  g_cor[j]*g_d[ji]*(g_ua[jip1]+g_ua[ji])
				  +g_cor[j-1]*g_d[jm1i]*(g_ua[jm1ip1]+g_ua[jm1i]) )
			+0.5f*grav*g_dx[j]/g_arv[j]*(g_d[ji]+g_d[jm1i])
			*( (1.0f-2.0f*alpha)*(g_el[ji]-g_el[jm1i])
			  +alpha*(g_elb[ji]-g_elb[jm1i]+g_elf[ji]-g_elf[jm1i]) )
			+ g_wvsurf[ji]-g_wvbot[ji];
			
			g_vaf[ji]= ((g_h[ji]+g_elb[ji]+g_h[jm1i]+g_elb[jm1i])*g_vab[ji]
						-4.0f*dte*vaf1) /(g_h[ji]+g_elf[ji]+g_h[jm1i]+g_elf[jm1i]);
			
		}
		
	}
	
	
	
		/// BCOND 2
	float gae;
	
	for (int j=1; j<(m_height-1); j++ )
	{
		j1 =  j * m_width;
		j2 =  j1 + 1;
		j3 =  j1 + 2;
		
		jl = j1 + m_width -1;
		jlm1 = jl - 1;
		
		if(g_dum[jl] > 0.5f)
		{
			gae = dte*sqrtf(grav*g_h[jl])/g_dx[j];
			
			g_uaf[jl] = gae*g_ua[jlm1]+(1.0f-gae)*g_ua[jl];
		}
		else
		{
			g_uaf[jl] = 0.0f;
		}

		g_vaf[jl]=0.0f;
		
		if(g_dum[j2] > 0.5f)
		{
			gae = dte*sqrtf(grav*g_h[j2])/g_dx[j];
			g_uaf[j2]=gae*g_ua[j3]+(1.-gae)*g_ua[j2];
		}
		else
		{
			g_uaf[j2]=0.0f;
		}
		
		g_uaf[j1]=g_uaf[j2];
		g_vaf[j1]=0.0;
		
	}
	
	
	

	for (int i=1; i<(m_width-1); i++ )
	{
		jli = m_width * (m_height - 1) + i;
		jlm1i = jli - m_width;
		
		j1i = i;
		
		j2i = m_width + j1i;
		
		j3i = m_width + j2i;
		
		
		if (g_dvm[jli] > 0.5f)
		{
			gae = dte*sqrtf(grav*g_h[jli])/g_dy[m_height-1];
			
			g_vaf[jli] = gae*g_va[jlm1i]+(1.0f-gae)*g_va[jli];
		}
		else
		{
			g_vaf[jli]=0.0f;
		}

		g_uaf[jli]=0.0;

		if (g_dvm[j2i] > 0.5f)
		{
			gae=dte*sqrtf(grav*g_h[j2i])/g_dy[0];
			
			g_vaf[j2i]=gae*g_va[j3i]+(1.-gae)*g_va[j2i];
		}
		else
		{
			g_vaf[j2i]=0.0f;
		}
		

		g_vaf[j1i]=g_vaf[j1i];
		g_uaf[j1i]=0.0f;
	}
	
	/// must separate cuda calls here
	
	for (int j=1; j<m_height; j++ )
	{
		for (int i=1; i<m_width; i++ )
		{
			ji = j * m_width + i;
			
			g_uaf[ji] = g_uaf[ji] * g_dum[ji];
			g_vaf[ji] = g_vaf[ji] * g_dvm[ji];
		}
	}
	// END BCOND 2
	
	
	
	float vmaxl = 100.0f;
	
	
	float tpsmax = 0.0f;
	
	int imax = 0;
	int jmax = 0;
	
	for (int j=1; j<m_height; j++ )
	{
		for (int i=1; i<m_width; i++ )
		{
			ji = j * m_width + i;
			
			g_tps[ji] = sqrtf(g_uaf[ji]*g_uaf[ji] + g_vaf[ji]*g_vaf[ji]);
			
			if (g_tps[ji] > tpsmax)
			{
				tpsmax = g_tps[ji];
				imax = i;
				jmax = j;
			}
		}
	}
	
	
	if (tpsmax > vmaxl)
	{
		setbuf(stdout,NULL);
		
		printf("vamax>vmax!!! at i=%d, j=%d \n", imax,jmax);
		
		exit(-1);
	}

	
	
	float smoth = 0.10f;
	
	
	for (int j=1; j<m_height; j++ )
	{
		for (int i=1; i<m_width; i++ )
		{
			ji = j * m_width + i;
			
			g_ua[ji]=g_ua[ji]+0.5f*smoth*(g_uab[ji]-2.0f*g_ua[ji]+g_uaf[ji]);
			g_va[ji]=g_va[ji]+0.5f*smoth*(g_vab[ji]-2.0f*g_va[ji]+g_vaf[ji]);
			g_el[ji]=g_el[ji]+0.5f*smoth*(g_elb[ji]-2.0f*g_el[ji]+g_elf[ji]);
			g_elb[ji]=g_el[ji];  // OP
			g_el[ji]=g_elf[ji];  // OP
			g_d[ji]=g_h[ji]+g_el[ji];
			g_uab[ji]=g_ua[ji];  // OP
			g_ua[ji]=g_uaf[ji];  // OP
			g_vab[ji]=g_va[ji];  // OP
			g_va[ji]=g_vaf[ji];  // OP
		}
	}
	
	
}




void KaspyCycler::getWindPressure(char uv)
{
	int kx, ky, kd, nx, ny, nd;
	float * p;
	float * px;
	float * py;
	float * pk;
	float xki, xka, yki, yka, xmi, xma, ymi, yma;
	float pkkd[50][50];
	float cd[50][50][4][4];
	
	float * pkk = &pkkd[0][0];
	float * c = &cd[0][0][0][0];
	
	if (uv == 'u')
	{
		kx = m_fWindData->kxu;
		ky = m_fWindData->kyu;
		pk = g_uwd0;
		
		xki = m_fWindData->xkui;
		xka = m_fWindData->xkua;
		yki = m_fWindData->ykui;
		yka = m_fWindData->ykua;
		
		p = g_ffu;
	}
	else if (uv == 'v')
	{
		kx = m_fWindData->kxv;
		ky = m_fWindData->kyv;
		pk = g_vwd0;
		
		xki = m_fWindData->xkvi;
		xka = m_fWindData->xkva;
		yki = m_fWindData->ykvi;
		yka = m_fWindData->ykva;
		
		p = g_ffv;
	}
	else if (uv == 'p')
	{
		kx = m_fWindData->kx;
		ky = m_fWindData->ky;
		//float kd = kx;
		pk = g_press0;
		xki = m_fWindData->xki;
		xka = m_fWindData->xka;
		yki = m_fWindData->yki;
		yka = m_fWindData->yka;

		
		p = g_ff;
		px = g_fxf;
		py = g_fyf;
	}
	else
	{
		// don't know what to do
		return;
	}
	
	kd = kx;
	
	nx = F_DATA_WIDTH;
	ny = F_DATA_HEIGHT;
	nd = F_DATA_WIDTH;
	
	xmi = m_fVars->xmi;
	xma = m_fVars->xma;
	ymi = m_fVars->ymi;
	yma = m_fVars->yma;
	
	float c1=3.1415926/180.0;
	float c2=111111.0f;
	
	
	float dky=(yka-yki)/(ky-1.0f);
	float  dkx=(xka-xki)/(kx-1.0f);
 
	float dy=(yma-ymi)/(ny-1.0f);
	float dx=(xma-xmi)/(nx-1.0f);
	
	
	for (int j=1; j<=ky; j++ )
	{
		for (int i=1; i<=kx; i++ )
		{
			pkk[j * 50 + i] = pk[(j - 1) * kd + i - 1];
		}
	}

	
	for (int j=1; j<=ky; j++ )
	{
		pkk[j*50+0] = 2.0f*pkk[j*50+1] - pkk[j*50+2];
		pkk[j*50+kx+1] = 2.0f*pkk[j*50+kx] - pkk[j*50+kx-1];
	}
	
	
	for (int i=0; i<=(kx+1); i++ )
	{
		pkk[0*50+i] = 2.0f*pkk[1*50+i] - pkk[2*50+i];
		pkk[(ky+1)*50+i] = 2.0f*pkk[ky*50+i] - pkk[(ky-1)*50+i];
	}
	
	getbicubic(kx + 2,ky + 2, 50, pkk,c);
	
	for (int j=0; j<ny; j++ )
	{
		float y = ymi + j*dy;
		int j0 = (int)((y - yki)/dky);
		
		if (j0 < 0)
		{
			j0 = 0;
		}
		
		if (j0 > ky-2)
		{
			j0 = ky-2;
		}
		
		float u = (y - (yki + j0*dky))/dky;
		
		for (int i=0; i<nx; i++ )
		{
			float x = xmi + i * dx;
			int i0 = (int)((x - xki)/dkx);
			
			if (i0 < 0) i0 = 0;
			
			if (i0 > kx-2) i0 = kx-2;
			
			float t = ( x - (xki + i0*dkx) )/dkx;
			
			float ay = 0.0f;
			float a2 = 0.0f;
			float a1 = 0.0f;
			
			int ji = j * nx + i;
			
			for (int k=3; k>=0; k-- )
			{
				ay = t*ay+((c[j0 * 800 + i0 * 16 + 3 * 4 + k] * u + c[j0 * 800 + i0 * 16 + 2 * 4 + k])*u
						   + c[j0 * 800 + i0 * 16 + 1 * 4 + k])*u + c[j0 * 800 + i0 * 16 + 0 * 4 + k];
			}
			
			if (uv == 'p')
			{
				for (int k=3; k>=0; k-- )
				{
					a2 = t*a2 + (3.0f*c[j0 * 800 + i0 * 16 + 3 * 4 + k]*u
								 + 2.0f*c[j0 * 800 + i0 * 16 + 2 * 4 + k])*u+c[j0 * 800 + i0 * 16 + 1 * 4 + k];
					
					a1 = u*a1 + (3.0f*c[j0 * 800 + i0 * 16 + k * 4 + 3]*t +
								 2.0f*c[j0 * 800 + i0 * 16 + k * 4 + 2])*t+c[j0 * 800 + i0 * 16 + k * 4 + 1];
					
				}
				
				a1 = a1/dkx/c2/cosf(c1*y);
				a2 = a2/dky/c2;
				
				px[ji] = a1;
				py[ji] = a2;
			}
			
			p[ji] = ay;
			
		}
		
	}
	

}







void getbicubic(int nx, int ny, int nd, float * z, float * c)
{
	float d1 = 1.0f;
	float d2 = 1.0f;
	
	float y[4];
	float y1[4];
	float y2[4];
	float y12[4];
	float cc[4][4];
	
	
	for (int j=1; j<ny-2; j++ )
	{
		for (int i=1; i<nx-2; i++ )
		{
			/*
			 Y(1)=Z(I,J)
			 Y(2)=Z(I+1,J)
			 Y(3)=Z(I+1,J+1)
			 Y(4)=Z(I,J+1)
			 */
			y[0] = z[j * nd + i];
			y[1] = z[j * nd + i + 1];
			y[2] = z[(j+1) * nd + i + 1];
			y[3] = z[(j+1) * nd + i];
			
			/*
			 Y1(1)=0.5*(Z(I+1,J)-Z(I-1,J))
			 Y1(4)=0.5*(Z(I+1,J+1)-Z(I-1,J+1))
			 Y1(2)=0.5*(Z(I+2,J)  -Z(I,J))
			 Y1(3)=0.5*(Z(I+2,J+1)-Z(I,J+1))
			 */
			y1[0] = 0.5f * (z[j * nd + i + 1] - z[j * nd + i - 1]);
			y1[3] = 0.5f * (z[(j+1) * nd + i + 1] - z[(j+1) * nd + i - 1]);
			y1[1] = 0.5f * (z[j * nd + i + 2] - z[j * nd + i]);
			y1[2] = 0.5f * (z[(j+1) * nd + i + 2] - z[(j+1) * nd + i]);

			
			/*
			 Y2(1)=0.5*(Z(I,J+1)  -Z(I,J-1))
			 Y2(2)=0.5*(Z(I+1,J+1)-Z(I+1,J-1))
			 Y2(3)=0.5*(Z(I+1,J+2)-Z(I+1,J))
			 Y2(4)=0.5*(Z(I,J+2)-Z(I,J))
			 */
			y2[0] = 0.5f * (z[(j+1) * nd + i] - z[(j-1) * nd + i]);
			y2[1] = 0.5f * (z[(j+1) * nd + i + 1] - z[(j-1) * nd + i + 1]);
			y2[2] = 0.5f * (z[(j+2) * nd + i + 1] - z[(j) * nd + i + 1]);
			y2[3] = 0.5f * (z[(j+2) * nd + i] - z[j * nd + i]);
			
			
			/*
			 Y12(1)=0.25*(Z(I+1,J+1)-Z(I+1,J-1)-Z(I-1,J+1)+Z(I-1,J-1))
			 Y12(2)=0.25*(Z(I+2,J+1)-Z(I+2,J-1)-Z(I,J+1)+Z(I,J-1))
			 Y12(3)=0.25*(Z(I+2,J+2)-Z(I+2,J)-Z(I,J+2)+Z(I,J))
			 Y12(4)=0.25*(Z(I+1,J+2)-Z(I+1,J)-Z(I-1,J+2)+Z(I-1,J))
			 */
			y12[0] = 0.25f * (z[(j+1) * nd + i + 1] - z[(j-1) * nd + i + 1]
							  - z[(j+1) * nd + i - 1] + z[(j-1) * nd + i - 1]);
			y12[1] = 0.25f * (z[(j+1) * nd + i + 2] - z[(j-1) * nd + i + 2]
							  - z[(j+1) * nd + i] + z[(j-1) * nd + i]);
			y12[2] = 0.25f * (z[(j+2) * nd + i + 2] - z[(j) * nd + i + 2]
							  - z[(j+2) * nd + i] + z[j * nd + i]);
			y12[3] = 0.25f * (z[(j+2) * nd + i + 1] - z[(j) * nd + i + 1]
							  - z[(j+2) * nd + i -1] + z[(j) * nd + i -1]);
	
			
			bcucof(&y[0],&y1[0],&y2[0],&y12[0],d1,d2,&cc[0][0]);
			
			for (int k=0; k<4; k++ )
			{
				for (int l=0; l<4; l++ )
				{
					//printf("\nk is %d l is %d\n", k, l);
					c[(j-1)* 800 + (i-1) * 16 + l * 4 + k ] = cc[l][k];
				}
			}
			
			
		}
	 }
	
}




void bcucof(float * y,float * y1,float * y2, float * y12,float d1,float d2,float * cc)
{
	float xx;
	float cl[16];
	
	float x[16];
	
	float wt[] = {
		1,0,-3,2,0,0,0,0,-3,0,9,-6,2,0,-6,4,
		0,0,0,0,0,0,0,0,3,0,-9,6,-2,0,6,-4,
		0,0,0,0,0,0,0,0,0,0,9,-6,0,0,-6,4,
		0,0,3,-2,0,0,0,0,0,0,-9,6,0,0,6,-4,
		0,0,0,0,1,0,-3,2,-2,0,6,-4,1,0,-3,2,
		0,0,0,0,0,0,0,0,-1,0,3,-2,1,0,-3,2,
		0,0,0,0,0,0,0,0,0,0,-3,2,0,0,3,-2,
		0,0,0,0,0,0,3,-2,0,0,-6,4,0,0,3,-2,
		0,1,-2,1,0,0,0,0,0,-3,6,-3,0,2,-4,2,
		0,0,0,0,0,0,0,0,0,3,-6,3,0,-2,4,-2,
		0,0,0,0,0,0,0,0,0,0,-3,3,0,0,2,-2,
		0,0,-1,1,0,0,0,0,0,0,3,-3,0,0,-2,2,
		0,0,0,0,0,1,-2,1,0,-2,4,-2,0,1,-2,1,
		0,0,0,0,0,0,0,0,0,-1,2,-1,0,1,-2,1,
		0,0,0,0,0,0,0,0,0,0,1,-1,0,0,-1,1,
		0,0,0,0,0,0,-1,1,0,0,2,-2,0,0,-1,1
 	};
	
	//float d1 = *pd1;
	//float d2 = *pd2;
	
	
	float d1d2 = d1 * d2;

	for (int i=0; i<4; i++ )
	{
		x[i] = y[i];
		x[i + 4] = y1[i] * d1;
		x[i + 8] = y2[i] * d2;
		x[i + 12] = y12[i] * d1d2;
	}
	
	for (int i=0; i<16; i++ )
	{
		xx = 0.0f;
		
		for (int k=0; k<16; k++ )
		{
			xx += wt[i + k*16] * x[k];
		}
		
		cl[i] = xx;
	}
	
	int l = 0;
	
	for (int i=0; i<4; i++ )
	{
		for (int j=0; j<4; j++ )
		{
			cc[j*4 + i] = cl[l++];
		}
	}
	
}






int KaspyCycler::init_device()
{
	int device_count = 0;
	
	if (m_gpu_device >= 0)
	{
		// already initialized
		printf("CUDA device is already initiaized\n");
		
		return m_gpu_device;
	}
	
	
	hipGetDeviceCount(&device_count);
	
	for (int i = 0 ; i < device_count ; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		
		if (properties.major > 1 || (properties.major == 1 && properties.minor >= 1))
		{
			m_gpu_device = i;
			
			printf("Running on GPU %d (%s) \n",i ,properties.name);
			break;
		}
		else
		{
			printf("GPU %d (%s) does not support CUDA Dynamic Parallelism\n", i ,properties.name);
		}
	}
	
	
	if (m_gpu_device == -1)
	{
		printf("No suitable device found!\n");
		return m_gpu_device;
	}
	
	if (hipSetDevice(m_gpu_device) == hipSuccess)
	{
		printf("device set OK\n");
	}
	else
	{
		printf("unable to set device!\n");
		m_gpu_device = -1;
	}
	
	
	
	// Allocate GPU memory.
	if ( (hipMallocManaged((void **)&g_fbu, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fbv, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_ffu, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_ffv, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fb, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_ff, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fxb, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fxf, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fyb, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fyf, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_wusurf, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_wvsurf, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_dum, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_dvm, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_d, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		
		&& (hipMallocManaged((void **)&g_fluxua, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fluxva, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		
		&& (hipMallocManaged((void **)&g_ua, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_va, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_uab, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_vab, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_uaf, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_vaf, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_el, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_elb, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_elf, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_fsm, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_tps, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_advua, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_advva, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_wubot, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_wvbot, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_cbc, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_h, m_height*m_width * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		
		&& (hipMallocManaged((void **)&g_press0,  m_fWindData->ky *  m_fWindData->kx * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		
		&& (hipMallocManaged((void **)&g_uwd0, m_fWindData->kyu * m_fWindData->kxu * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_vwd0, m_fWindData->kyv * m_fWindData->kxv * sizeof(float), hipMemAttachGlobal) == hipSuccess)

		
		&& (hipMallocManaged((void **)&g_cor, m_height * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_aru, m_height * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_arv, m_height * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_dx, m_height * sizeof(float), hipMemAttachGlobal) == hipSuccess)
		&& (hipMallocManaged((void **)&g_dy, m_height * sizeof(float), hipMemAttachGlobal) == hipSuccess))
	{
		printf("GPU memory allocated\n");
		
	}
	else
	{
		printf("GPU memory allocation error!\n");
		deinit_device();
		return m_gpu_device;
	}
	
	
	
	
	return m_gpu_device;
}


void KaspyCycler::deinit_device()
{
	if (m_gpu_device >=0)
	{
		
		if (g_fbu)
		{
			hipFree(g_fbu);
		}
		
		if (g_fbv)
		{
			hipFree(g_fbv);
		}
		
		if (g_ffu)
		{
			hipFree(g_ffu);
		}
		
		if (g_ffv)
		{
			hipFree(g_ffv);
		}
		
		
		if (g_fb)
		{
			hipFree(g_fb);
		}
		
		if (g_ff)
		{
			hipFree(g_ff);
		}
		
		
		if (g_fxb)
		{
			hipFree(g_fxb);
		}
		
		if (g_fxf)
		{
			hipFree(g_fxf);
		}
		
		if (g_fyb)
		{
			hipFree(g_fyb);
		}
		
		if (g_fyf)
		{
			hipFree(g_fyf);
		}
		
		if (g_wusurf)
		{
			hipFree(g_wusurf);
		}
		
		if (g_wvsurf)
		{
			hipFree(g_wvsurf);
		}
		
		if (g_dum)
		{
			hipFree(g_dum);
		}
		
		if (g_dvm)
		{
			hipFree(g_dvm);
		}
		
		if (g_d)
		{
			hipFree(g_d);
		}
		
		if (g_fluxua)
		{
			hipFree(g_fluxua);
		}
		
		if (g_fluxva)
		{
			hipFree(g_fluxva);
		}
		
		if (g_ua)
		{
			hipFree(g_ua);
		}
		
		if (g_va)
		{
			hipFree(g_va);
		}
		
		if (g_uaf)
		{
			hipFree(g_uaf);
		}
		
		if (g_vaf)
		{
			hipFree(g_vaf);
		}
		
		if (g_uab)
		{
			hipFree(g_uab);
		}
		
		if (g_vab)
		{
			hipFree(g_vab);
		}
		
		if (g_el)
		{
			hipFree(g_el);
		}
		
		if (g_elb)
		{
			hipFree(g_elb);
		}
		
		if (g_elf)
		{
			hipFree(g_elf);
		}
		
		if (g_fsm)
		{
			hipFree(g_fsm);
		}
		
		if (g_tps)
		{
			hipFree(g_tps);
		}
		
		if (g_advua)
		{
			hipFree(g_advua);
		}
		
		if (g_advva)
		{
			hipFree(g_advva);
		}
		
		if (g_aru)
		{
			hipFree(g_aru);
		}
		
		if (g_arv)
		{
			hipFree(g_arv);
		}
		
		if (g_wubot)
		{
			hipFree(g_wubot);
		}
		
		if (g_wvbot)
		{
			hipFree(g_wvbot);
		}
		
		if (g_cbc)
		{
			hipFree(g_cbc);
		}
		
		if (g_h)
		{
			hipFree(g_h);
		}
		
		if (g_cor)
		{
			hipFree(g_cor);
		}
		
		if (g_press0)
		{
			hipFree(g_press0);
		}
		
		
		if (g_uwd0)
		{
			hipFree(g_uwd0);
		}
		
		if (g_vwd0)
		{
			hipFree(g_vwd0);
		}
		
		if (hipDeviceReset() == hipSuccess)
		{
			printf("GPU device reset ok\n");
		}
		
		m_gpu_device = -1;
	}
}




