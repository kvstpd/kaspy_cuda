#include "hip/hip_runtime.h"
//
//  KaspyCycler.cpp
//  kaspy_cuda
//
//  Created by Andrei Koulikov on 24.05.16.
//
//

#define HIPCUB_STDERR

#include "InitValues.h"

#include "KaspyCycler.h"


#include "hip/hip_runtime.h"
#include ""


#include "hipcub/hipcub.hpp"


using namespace hipcub;


extern InitValues * initValues;

extern "C" void WRITEGRD(int * NX, int * NY, int * NDX, float * Z, float * XMI, float * XMA, float * YMI, float * YMA,const char * NAME);





void getbicubic(int nx, int ny, int nd, float * z, float * c);
void bcucof(float * y,float * y1,float * y2, float * y12,float d1,float d2,float * cc);


float grav = 9.806;

CachingDeviceAllocator  g_allocator(true);



int * g_stations_x = 0;
int * g_stations_y = 0;

float * g_station_elves = 0;

__device__ int * dev_stations_x = 0;
__device__ int * dev_stations_y = 0;

__device__ int dev_nstations = 0;

__device__ float * dev_station_elves = 0;



__device__ float * dev_fbu = 0;
__device__ float * dev_fbv = 0;
__device__ float * dev_ffu = 0;
__device__ float * dev_ffv = 0;

__device__ float * dev_fxb = 0;
__device__ float * dev_fxf = 0;
__device__ float * dev_fyb = 0;
__device__ float * dev_fyf = 0;

__device__ float * dev_fb = 0;
__device__ float * dev_ff = 0;

__device__ float * dev_wusurf = 0;
__device__ float * dev_wvsurf = 0;

__device__ float * dev_dum = 0;
__device__ float * dev_dvm = 0;

__device__ float * dev_d = 0;
__device__ float * dev_dx = 0;
__device__ float * dev_dy = 0;

__device__ float * dev_fluxua = 0;
__device__ float * dev_fluxva = 0;

__device__ float * dev_ua = 0;
__device__ float * dev_va = 0;

__device__ float * dev_uab = 0;
__device__ float * dev_vab = 0;

__device__ float * dev_uaf = 0;
__device__ float * dev_vaf = 0;


__device__ float * dev_el = 0;
__device__ float * dev_elf = 0;
__device__ float * dev_elb = 0;

__device__ float * dev_elf_r = 0;

__device__ float * dev_fsm = 0;

__device__ float * dev_tps = 0;


__device__ float * dev_advua = 0;
__device__ float * dev_advva = 0;

__device__ float * dev_aru = 0;
__device__ float * dev_arv = 0;

__device__ float * dev_wubot = 0;
__device__ float * dev_wvbot = 0;
__device__ float * dev_cbc = 0;

__device__ float * dev_cor = 0;

__device__ float * dev_h = 0;

__device__ float * dev_press = 0;
__device__ float * dev_uwd = 0;
__device__ float * dev_vwd = 0;

//__device__ float * dev_press0 = 0;
//__device__ float * dev_uwd0 = 0;
//__device__ float * dev_vwd0 = 0;

__device__ float * dev_art = 0;


__device__ float * dev_p = 0;
__device__ float * dev_pk = 0;
__device__ float * dev_px = 0;
__device__ float * dev_py = 0;


__device__ float * dev_temp = 0;




__device__ float * dev_sel = 0;
__device__ float * dev_ssel = 0;
__device__ float * dev_sfel = 0;
__device__ float * dev_sfa = 0;
__device__ float * dev_ssfa = 0;
__device__ float * dev_sfar = 0;
__device__ float * dev_ssfar = 0;
__device__ float * dev_sfelr = 0;
__device__ float * dev_su = 0;
__device__ float * dev_sv = 0;
__device__ float * dev_ssu = 0;
__device__ float * dev_ssv = 0;
__device__ float * dev_ssuv = 0;
__device__ float * dev_ssue = 0;
__device__ float * dev_ssve = 0;



float * g_sel = 0;
float * g_ssel = 0;
float * g_sfel = 0;
float * g_sfa = 0;
float * g_ssfa = 0;
float * g_sfar = 0;
float * g_ssfar = 0;
float * g_sfelr = 0;
float * g_su = 0;
float * g_sv = 0;
float * g_ssu = 0;
float * g_ssv = 0;
float * g_ssuv = 0;
float * g_ssue = 0;
float * g_ssve = 0;


float * g_fbu = 0;
float * g_fbv = 0;
float * g_ffu = 0;
float * g_ffv = 0;

float * g_fxb = 0;
float * g_fxf = 0;
float * g_fyb = 0;
float * g_fyf = 0;

float * g_fb = 0;
float * g_ff = 0;

float * g_wusurf = 0;
float * g_wvsurf = 0;

float * g_dum = 0;
float * g_dvm = 0;

float * g_d = 0;
float * g_dx = 0;
float * g_dy = 0;

float * g_fluxua = 0;
float * g_fluxva = 0;

float * g_ua = 0;
float * g_va = 0;

float * g_uab = 0;
float * g_vab = 0;

float * g_uaf = 0;
float * g_vaf = 0;


float * g_el = 0;
float * g_elf = 0;
float * g_elb = 0;

float * g_elf_r = 0;

float * g_fsm = 0;

float * g_tps = 0;


float * g_advua = 0;
float * g_advva = 0;

float * g_aru = 0;
float * g_arv = 0;

float * g_wubot = 0;
float * g_wvbot = 0;
float * g_cbc = 0;

float * g_cor = 0;

float * g_h = 0;

float * g_press = 0;
float * g_uwd = 0;
float * g_vwd = 0;

float * g_press0 = 0;
float * g_uwd0 = 0;
float * g_vwd0 = 0;

float * g_art = 0;


__constant__ __device__  float dev_grav = 9.806f;
__constant__ __device__  float dev_ro_ratio = 1.29f/1020.0f;

__constant__ __device__  int  dev_width;
__constant__ __device__  int  dev_height;
__constant__ __device__  int  dev_widthm1;
__constant__ __device__  int  dev_heightm1;

__constant__ __device__ int dev_ewidth;

__constant__ __device__ float dev_dte;
__constant__ __device__ float dev_dte2;
__constant__ __device__ float dev_aam2d;

__constant__ __device__ float dev_tide_l = 0.0f;

__constant__ __device__ float dev_alpha = 0.225f;

__constant__ __device__ float dev_vmaxl = 100.0f;;

__device__ int dev_should_stop = 0;

__constant__ __device__ float dev_smoth = 0.10f;

__constant__ __device__ float dev_xmi;// = m_fVars->xmi;
__constant__ __device__ float dev_xma;// = m_fVars->xma;
__constant__ __device__ float dev_ymi;// = m_fVars->ymi;
__constant__ __device__ float dev_yma;// = m_fVars->yma;



__constant__ __device__ float dev_c1 = 3.1415926/180.0;
__constant__ __device__ float dev_c2 = 111111.0f;


__constant__ __device__ float dev_wt[] = {
	1,0,-3,2,0,0,0,0,-3,0,9,-6,2,0,-6,4,
	0,0,0,0,0,0,0,0,3,0,-9,6,-2,0,6,-4,
	0,0,0,0,0,0,0,0,0,0,9,-6,0,0,-6,4,
	0,0,3,-2,0,0,0,0,0,0,-9,6,0,0,6,-4,
	0,0,0,0,1,0,-3,2,-2,0,6,-4,1,0,-3,2,
	0,0,0,0,0,0,0,0,-1,0,3,-2,1,0,-3,2,
	0,0,0,0,0,0,0,0,0,0,-3,2,0,0,3,-2,
	0,0,0,0,0,0,3,-2,0,0,-6,4,0,0,3,-2,
	0,1,-2,1,0,0,0,0,0,-3,6,-3,0,2,-4,2,
	0,0,0,0,0,0,0,0,0,3,-6,3,0,-2,4,-2,
	0,0,0,0,0,0,0,0,0,0,-3,3,0,0,2,-2,
	0,0,-1,1,0,0,0,0,0,0,3,-3,0,0,-2,2,
	0,0,0,0,0,1,-2,1,0,-2,4,-2,0,1,-2,1,
	0,0,0,0,0,0,0,0,0,-1,2,-1,0,1,-2,1,
	0,0,0,0,0,0,0,0,0,0,1,-1,0,0,-1,1,
	0,0,0,0,0,0,-1,1,0,0,2,-2,0,0,-1,1
};


__device__ float dev_pkk[50 * 50];
__device__ float dev_c[50 * 50 * 4 * 4];



__device__ void dev_bcucof(float * y,float * y1,float * y2, float * y12,float d1,float d2,float * cc)
{
	float xx;
	float cl[16];
	
	float x[16];
	
	
	float d1d2 = d1 * d2;
	
	for (int i=0; i<4; i++ )
	{
		x[i] = y[i];
		x[i + 4] = y1[i] * d1;
		x[i + 8] = y2[i] * d2;
		x[i + 12] = y12[i] * d1d2;
	}
	
	for (int i=0; i<16; i++ )
	{
		xx = 0.0f;
		
		for (int k=0; k<16; k++ )
		{
			xx += dev_wt[i + k*16] * x[k];
		}
		
		cl[i] = xx;
	}
	
	int l = 0;
	
	for (int i=0; i<4; i++ )
	{
		for (int j=0; j<4; j++ )
		{
			cc[j*4 + i] = cl[l++];
		}
	}
	
}


__global__ void dev_make_p(int nx, int ny, int kx, int ky, float dx, float dy, float dkx, float dky, float xki, float yki)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	
	
	if (i < nx && j < ny)
	{
		float y = dev_ymi + j*dy;
		int j0 = (int)((y - yki)/dky);
		
		if (j0 < 0)
		{
			j0 = 0;
		}
		
		if (j0 > ky-2)
		{
			j0 = ky-2;
		}
		
		float u = (y - (yki + j0*dky))/dky;
		
		
		float x = dev_xmi + i * dx;
		
		int i0 = (int)((x - xki)/dkx);
		
		if (i0 < 0) i0 = 0;
		
		if (i0 > kx-2) i0 = kx-2;
		
		float t = ( x - (xki + i0*dkx) )/dkx;
		
		float ay = 0.0f;
		float a2 = 0.0f;
		float a1 = 0.0f;
		
		int ji = j * nx + i;
		
		for (int k=3; k>=0; k-- )
		{
			ay = t*ay+((dev_c[j0 * 800 + i0 * 16 + 3 * 4 + k] * u + dev_c[j0 * 800 + i0 * 16 + 2 * 4 + k])*u
					   + dev_c[j0 * 800 + i0 * 16 + 1 * 4 + k])*u + dev_c[j0 * 800 + i0 * 16 + 0 * 4 + k];
		}
		
		
		if (dev_px != 0)
		{
			for (int k=3; k>=0; k-- )
			{
				a2 = t*a2 + (3.0f*dev_c[j0 * 800 + i0 * 16 + 3 * 4 + k]*u
							 + 2.0f*dev_c[j0 * 800 + i0 * 16 + 2 * 4 + k])*u+dev_c[j0 * 800 + i0 * 16 + 1 * 4 + k];
				
				a1 = u*a1 + (3.0f*dev_c[j0 * 800 + i0 * 16 + k * 4 + 3]*t +
							 2.0f*dev_c[j0 * 800 + i0 * 16 + k * 4 + 2])*t+dev_c[j0 * 800 + i0 * 16 + k * 4 + 1];
				
			}
			
			a1 = a1/dkx/dev_c2/cosf(dev_c1*y);
			a2 = a2/dky/dev_c2;
			
			dev_px[ji] = a1;
			dev_py[ji] = a2;
		}
		
		dev_p[ji] = ay;
		
	}
}




__global__ void dev_bicubic(int nx, int ny, int nd)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	
	float d1 = 1.0f;
	float d2 = 1.0f;
	
	float y[4];
	float y1[4];
	float y2[4];
	float y12[4];
	float cc[4][4];
	
	
	
	
	if (i > 0 && j > 0 && i < (nx - 2) && j < (ny - 2))
	{
		
		y[0] = dev_pkk[j * nd + i];
		y[1] = dev_pkk[j * nd + i + 1];
		y[2] = dev_pkk[(j+1) * nd + i + 1];
		y[3] = dev_pkk[(j+1) * nd + i];
		
		y1[0] = 0.5f * (dev_pkk[j * nd + i + 1] - dev_pkk[j * nd + i - 1]);
		y1[3] = 0.5f * (dev_pkk[(j+1) * nd + i + 1] - dev_pkk[(j+1) * nd + i - 1]);
		y1[1] = 0.5f * (dev_pkk[j * nd + i + 2] - dev_pkk[j * nd + i]);
		y1[2] = 0.5f * (dev_pkk[(j+1) * nd + i + 2] - dev_pkk[(j+1) * nd + i]);
		
		
		y2[0] = 0.5f * (dev_pkk[(j+1) * nd + i] - dev_pkk[(j-1) * nd + i]);
		y2[1] = 0.5f * (dev_pkk[(j+1) * nd + i + 1] - dev_pkk[(j-1) * nd + i + 1]);
		y2[2] = 0.5f * (dev_pkk[(j+2) * nd + i + 1] - dev_pkk[(j) * nd + i + 1]);
		y2[3] = 0.5f * (dev_pkk[(j+2) * nd + i] - dev_pkk[j * nd + i]);
		
		
		y12[0] = 0.25f * (dev_pkk[(j+1) * nd + i + 1] - dev_pkk[(j-1) * nd + i + 1]
						  - dev_pkk[(j+1) * nd + i - 1] + dev_pkk[(j-1) * nd + i - 1]);
		y12[1] = 0.25f * (dev_pkk[(j+1) * nd + i + 2] - dev_pkk[(j-1) * nd + i + 2]
						  - dev_pkk[(j+1) * nd + i] + dev_pkk[(j-1) * nd + i]);
		y12[2] = 0.25f * (dev_pkk[(j+2) * nd + i + 2] - dev_pkk[(j) * nd + i + 2]
						  - dev_pkk[(j+2) * nd + i] + dev_pkk[j * nd + i]);
		y12[3] = 0.25f * (dev_pkk[(j+2) * nd + i + 1] - dev_pkk[(j) * nd + i + 1]
						  - dev_pkk[(j+2) * nd + i -1] + dev_pkk[(j) * nd + i -1]);
		
		
		dev_bcucof(&y[0],&y1[0],&y2[0],&y12[0],d1,d2,&cc[0][0]);
		
		for (int k=0; k<4; k++ )
		{
			for (int l=0; l<4; l++ )
			{
				//printf("\nk is %d l is %d\n", k, l);
				dev_c[(j-1)* 800 + (i-1) * 16 + l * 4 + k ] = cc[l][k];
			}
		}
	}
}




__global__ void dev_pkk_ij(int kx, int ky, int kd)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (i > 0 && j > 0 && i <= kx && j <= ky)
	{
		dev_pkk[j * 50 + i] = dev_pk[(j - 1) * kd + i - 1];
	}
}


__global__ void dev_pkk_j(int kx, int ky)
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (j > 0 && j <= ky)
	{
		dev_pkk[j*50+0] = 2.0f*dev_pkk[j*50+1] - dev_pkk[j*50+2];
		dev_pkk[j*50+kx+1] = 2.0f*dev_pkk[j*50+kx] - dev_pkk[j*50+kx-1];
	}
}


__global__ void dev_pkk_i(int kx, int ky)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i > 0 && i <= kx+1)
	{
		dev_pkk[0*50+i] = 2.0f*dev_pkk[1*50+i] - dev_pkk[2*50+i];
		dev_pkk[(ky+1)*50+i] = 2.0f*dev_pkk[ky*50+i] - dev_pkk[(ky-1)*50+i];
	}
}


/**/

__global__ void surf_and_flux_1(float ftim)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	int jp1i = ji + dev_width;
	int jip1 = ji + 1;
	int jim1 = ji - 1;
	int jm1i = ji - dev_width;
	
	float btim = 1.0f - ftim;
	
	
	if (i < dev_widthm1 && j < dev_heightm1)
	{
		float uw = btim * (dev_fbu[ji]) + ftim * (dev_ffu[ji]);
		float vw = btim * (dev_fbv[ji]) + ftim * (dev_ffv[ji]);
		
		float speed =  hypotf(uw, vw); //sqrtf(uw*uw + vw*vw);
		float windc = 0.001f * (0.8f + speed * 0.065f) * dev_ro_ratio * speed;
		
		dev_wusurf[ji] = -windc * uw *
		0.25f * (dev_dum[jp1i]+dev_dum[jip1]+dev_dum[jim1]+dev_dum[jm1i])
		+ 0.5f * (dev_d[ji] + dev_d[jim1]) * (btim * dev_fxb[ji] + ftim * dev_fxf[ji]);
		
		dev_wvsurf[ji] = -windc * vw *
		0.25f * (dev_dvm[jp1i]+dev_dvm[jip1]+dev_dvm[jim1]+dev_dvm[jm1i])
		+ 0.5f * (dev_d[ji] + dev_d[jm1i]) * (btim * dev_fyb[ji] + ftim * dev_fyf[ji]);
	}
	
	if (i < dev_width && j < dev_height)
	{
		dev_fluxua[ji] = 0.25f * (dev_d[ji] + dev_d[jim1]) * (dev_dy[j] + dev_dy[j] ) * dev_ua[ji];
		dev_fluxva[ji] = 0.25f * (dev_d[ji] + dev_d[jm1i]) * (dev_dx[j] + dev_dx[j-1] ) * dev_va[ji];
	}
}


__global__ void elf_and_flux_2()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	int jp1i = ji + dev_width;
	int jip1 = ji + 1;
	
	if (i > 0 && j > 0 && i < dev_widthm1 && j < dev_heightm1)
	{
		dev_elf[ji] = dev_elb[ji] - dev_dte2 *
		(dev_fluxua[jip1] - dev_fluxua[ji] + dev_fluxva[jp1i] - dev_fluxva[ji]) / dev_art[j];
	}

}


__global__ void bcond_1_j()
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (j > 0 && j < dev_height)
	{
		dev_elf[j * dev_width + 1] = dev_tide_l;
		dev_elf[j * dev_width + dev_width - 2] = dev_tide_l;
		
		dev_elf[j * dev_width] = dev_tide_l;
		dev_elf[j * dev_width + dev_width - 1] = dev_tide_l;
	}
}

__global__ void bcond_1_i()
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if (i > 0 && i< dev_width)
	{
		dev_elf[i] =  dev_elf[i + dev_width];
		
		dev_elf[i + dev_width * (dev_height - 1)  ] =  dev_elf[i + dev_width * (dev_height - 2)];
	}
}


__global__ void bcond_1_ji()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	
	
	if (i > 0 && j > 0 && i < dev_width && j < dev_height)
	{
		dev_elf[ji] *= dev_fsm[ji];
	}
	
}

__global__ void uaf_and_vaf_3()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	
 	int jp1i = ji + dev_width;
 	int jip1 = ji + 1;
 	int jim1 = ji - 1;
 	int jm1i = ji - dev_width;
 	//int jm1im1 = jm1i  - 1;
 	int jp1im1 = jp1i - 1;
 	int jm1ip1 = jm1i + 1;
	
	
	

	if (i > 0 && j > 0)
	{
		if (i < dev_width && j < dev_heightm1)
		{

			
			
			float uaf1= dev_advua[ji]   -0.25f*(dev_cor[j]*dev_d[ji]*(dev_va[jp1i]+dev_va[ji])
												+dev_cor[j]*dev_d[jim1]*(dev_va[jp1im1]+dev_va[jim1]) )
			+0.5f*dev_grav*dev_dy[j]/dev_aru[j]*(dev_d[ji]+dev_d[jim1])
			*( (1.0f-2.0f*dev_alpha)*(dev_el[ji]-dev_el[jim1])
			  +dev_alpha*(dev_elb[ji]-dev_elb[jim1]+dev_elf[ji]-dev_elf[jim1]) )
			+dev_wusurf[ji]-dev_wubot[ji];
			
			dev_uaf[ji]=
			((dev_h[ji]+dev_elb[ji]+dev_h[jim1]+dev_elb[jim1])*dev_uab[ji]
			 -4.e0*dev_dte*uaf1)  /(dev_h[ji]+dev_elf[ji]+dev_h[jim1]+dev_elf[jim1]);
		}
		
		if (i < dev_widthm1 && j < dev_height)
		{
			float vaf1=dev_advva[ji]
			+.25f*(  dev_cor[j]*dev_d[ji]*(dev_ua[jip1]+dev_ua[ji])
				   +dev_cor[j-1]*dev_d[jm1i]*(dev_ua[jm1ip1]+dev_ua[jm1i]) )
			+0.5f*dev_grav*dev_dx[j]/dev_arv[j]*(dev_d[ji]+dev_d[jm1i])
			*( (1.0f-2.0f*dev_alpha)*(dev_el[ji]-dev_el[jm1i])
			  +dev_alpha*(dev_elb[ji]-dev_elb[jm1i]+dev_elf[ji]-dev_elf[jm1i]) )
			+ dev_wvsurf[ji]-dev_wvbot[ji];
			
			dev_vaf[ji]= ((dev_h[ji]+dev_elb[ji]+dev_h[jm1i]+dev_elb[jm1i])*dev_vab[ji]
						  -4.0f*dev_dte*vaf1) /(dev_h[ji]+dev_elf[ji]+dev_h[jm1i]+dev_elf[jm1i]);
			
		}
		
	}
	
}


__global__ void bcond_2_j()
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	
	int j1 =  j * dev_width;
	int j2 =  j1 + 1;
	int j3 =  j1 + 2;
	int jl = j1 + dev_widthm1;
	int jlm1 = jl - 1;
	
	float gae;
	
	if (j > 0 && j < dev_heightm1)
	{
		if(dev_dum[jl] > 0.5f)
		{
			gae = dev_dte*sqrtf(dev_grav*dev_h[jl])/dev_dx[j];
			
			dev_uaf[jl] = gae*dev_ua[jlm1]+(1.0f-gae)*dev_ua[jl];
		}
		else
		{
			dev_uaf[jl] = 0.0f;
		}
		
		dev_vaf[jl]=0.0f;
		
		if(dev_dum[j2] > 0.5f)
		{
			gae = dev_dte*sqrtf(dev_grav*dev_h[j2])/dev_dx[j];
			dev_uaf[j2]=gae*dev_ua[j3]+(1.0f-gae)*dev_ua[j2];
		}
		else
		{
			dev_uaf[j2]=0.0f;
		}
		
		dev_uaf[j1]=dev_uaf[j2];
		dev_vaf[j1]=0.0f;

	}
}

__global__ void bcond_2_i()
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	int jli = dev_width * (dev_heightm1) + i;
	int jlm1i = jli - dev_width;
	
	int j1i = i;
	
	int j2i = dev_width + j1i;
	
	int j3i = dev_width + j2i;
	
	float gae;
	
	if (i > 0 && i< dev_widthm1)
	{
		if (dev_dvm[jli] > 0.5f)
		{
			gae = dev_dte * sqrtf(dev_grav * dev_h[jli]) / dev_dy[dev_heightm1];
			
			dev_vaf[jli] = gae * dev_va[jlm1i]+(1.0f-gae)*dev_va[jli];
		}
		else
		{
			dev_vaf[jli]=0.0f;
		}
		
		dev_uaf[jli]=0.0;
		
		if (dev_dvm[j2i] > 0.5f)
		{
			gae=dev_dte*sqrtf(dev_grav*dev_h[j2i])/dev_dy[0];
			
			dev_vaf[j2i]=gae*dev_va[j3i]+(1.-gae)*dev_va[j2i];
		}
		else
		{
			dev_vaf[j2i]=0.0f;
		}
		
		
		dev_vaf[j1i]=dev_vaf[j1i];
		dev_uaf[j1i]=0.0f;
	}
}


__global__ void bcond_2_ji()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	
	
	if (i > 0 && j > 0 && i < dev_width && j < dev_height)
	{
		dev_uaf[ji] = dev_uaf[ji] * dev_dum[ji];
		dev_vaf[ji] = dev_vaf[ji] * dev_dvm[ji];
	}
 

}

__global__ void tps_and_other_arrays_4()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	
	
	if (i > 0 && j > 0 && i < dev_width && j < dev_height)
	{
		dev_tps[ji] = hypotf(dev_uaf[ji], dev_vaf[ji]);  ///sqrtf(dev_uaf[ji]*dev_uaf[ji] + dev_vaf[ji]*dev_vaf[ji]);
		
		/*if (dev_tps[ji] > dev_vmaxl)
		{
			dev_should_stop = 1;
		}*/
		
		dev_ua[ji]=dev_ua[ji]+0.5f*dev_smoth*(dev_uab[ji]-2.0f*dev_ua[ji]+dev_uaf[ji]);
		dev_va[ji]=dev_va[ji]+0.5f*dev_smoth*(dev_vab[ji]-2.0f*dev_va[ji]+dev_vaf[ji]);
		dev_el[ji]=dev_el[ji]+0.5f*dev_smoth*(dev_elb[ji]-2.0f*dev_el[ji]+dev_elf[ji]);
		//dev_elb[ji]=dev_el[ji];  // OP
		//dev_el[ji]=dev_elf[ji];  // OP
		dev_d[ji]=dev_h[ji]+dev_elf[ji];
		//dev_uab[ji]=dev_ua[ji];  // OP
		//dev_ua[ji]=dev_uaf[ji];  // OP
		//dev_vab[ji]=dev_va[ji];  // OP
		//dev_va[ji]=dev_vaf[ji];  // OP
		
	}
	
	
}

__global__ void swap_arrays_5()
{
	float * t;
	
	t = dev_elb;
	dev_elb = dev_el;
	dev_el = dev_elf;
	dev_elf = t;
	
	
	t = dev_uab;
	dev_uab = dev_ua;
	dev_ua = dev_uaf;
	dev_uaf = t;
	
	t = dev_vab;
	dev_vab = dev_va;
	dev_va = dev_vaf;
	dev_vaf = t;
}


__global__ void adv_fluxes_1()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	int jip1 = ji + 1;
	int jim1 = ji - 1;
	int jm1i = ji - dev_width;
	int jm1im1 = jm1i  - 1;
 
	
	
	if (i > 0 && j > 0)
	{
		if (i < dev_widthm1 && j < dev_height)
		{
			dev_fluxua[ji]=dev_dy[j]*(.125f*((dev_d[jip1]+dev_d[ji])*dev_ua[jip1]
										 +(dev_d[ji]+dev_d[jim1])*dev_ua[ji])
								  *(dev_ua[jip1]+dev_ua[ji])
								  -dev_d[ji]*2.0f*dev_aam2d*(dev_uab[jip1]-dev_uab[ji])/dev_dx[j]);
		}
		
		
		if (i < dev_width && j < dev_height)
		{
			dev_tps[ji]=(dev_d[ji]+dev_d[jim1]+dev_d[jm1i]+dev_d[jm1im1])
			*dev_aam2d
			*((dev_uab[ji]-dev_uab[jm1i])
			  /(4.0f*dev_dy[j])
			  +(dev_vab[ji]-dev_vab[jim1])
			  /(4.0f*dev_dx[j]) );
			
			dev_fluxva[ji]=(.125f*((dev_d[ji]+dev_d[jm1i])*dev_va[ji]
								 +(dev_d[jim1]+dev_d[jm1im1])*dev_va[jim1])
						  *(dev_ua[ji]+dev_ua[jm1i])
						  -dev_tps[ji])*dev_dx[j];
			
		}

	}
}




__global__ void adv_advua_1()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	int jim1 = ji - 1;
 	int jp1i = ji + dev_width;
	
	if (i > 0 && j > 0 && i < dev_widthm1 && j < dev_heightm1)
	{
		dev_advua[ji]=(dev_fluxua[ji]-dev_fluxua[jim1]
					 +dev_fluxva[jp1i]-dev_fluxva[ji])/dev_aru[j];
		
	}
}

__global__ void adv_fluxes_2()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	int jim1 = ji - 1;
	int jp1i = ji + dev_width;
	int jm1i = ji - dev_width;
	int jm1im1 = jm1i  - 1;
 
	
	
	if (i > 0 && j > 0)
	{
		if (i < dev_width && j < dev_heightm1)
		{
			dev_fluxva[ji]=dev_dx[j]*(.125f*((dev_d[jp1i]+dev_d[ji])
										 *dev_va[jp1i]+(dev_d[ji]+dev_d[jm1i])*dev_va[ji])
								  *(dev_va[jp1i]+dev_va[ji])
								  -dev_d[ji]*2.0f*dev_aam2d*(dev_vab[jp1i]-dev_vab[ji])/dev_dy[j]);
		}
		
		
		if (i < dev_width && j < dev_height)
		{
			dev_fluxua[ji]=(.125f*((dev_d[ji]+dev_d[jim1])*dev_ua[ji]
								 +(dev_d[jm1i]+dev_d[jm1im1])*dev_ua[jm1i])*
						  (dev_va[jim1]+dev_va[ji])
						  -dev_tps[ji])*dev_dy[j];
			
		}
		
	}
}


__global__ void adv_advva_2()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	int jip1 = ji + 1;
	int jm1i = ji - dev_width;
	
	if (i > 0 && j > 0 && i < dev_widthm1 && j < dev_heightm1)
	{
		dev_advva[ji]=(dev_fluxua[jip1]-dev_fluxua[ji]
					 +dev_fluxva[ji]-dev_fluxva[jm1i])/dev_arv[j];
		
	}
}

__global__ void adv_bot_3()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
 	int jp1i = ji + dev_width;
 	int jip1 = ji + 1;
 	int jim1 = ji - 1;
 	int jm1i = ji - dev_width;
 
 	int jp1im1 = jp1i - 1;
 	int jm1ip1 = jm1i + 1;
	
	if (i > 0 && j > 0 && i < dev_widthm1 && j < dev_heightm1)
	{
		dev_wubot[ji]=-0.5f*(dev_cbc[ji]+dev_cbc[jim1])
		* hypotf(dev_uab[ji], 0.25f*(dev_vab[ji] +dev_vab[jp1i]+dev_vab[jim1]+dev_vab[jp1im1]))
		*dev_uab[ji];
		
		dev_wvbot[ji]=-0.5f*(dev_cbc[ji]+dev_cbc[jm1i])
		* hypotf(.25f*(dev_uab[ji]+dev_uab[jip1]+dev_uab[jm1i]+dev_uab[jm1ip1]), dev_vab[ji])
		* dev_vab[ji];
	}
}



__global__ void dev_fill_station_data(int khour)
{
	int n = blockDim.x * blockIdx.x + threadIdx.x;
	
	int ji;
	
	if (n < dev_nstations)
	{
		ji = (dev_stations_y[n] - 1) * dev_width + dev_stations_x[n] - 1;
		
		dev_station_elves[(khour-1) * dev_nstations + n] = dev_el[ji];
		
		//printf("set st to %f", dev_el[ji]);
		
		/*if(n==0)
		{
			printf("khour is %d elf is %f\n", khour, dev_station_elves[khour * dev_nstations + n]);
		}*/
	}
	

}


__global__ void dev_statistics_1(float ftim)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	
	if (i < dev_width && j < dev_height)
	{
		dev_sel[ji] += dev_el[ji];
		dev_ssel[ji] += dev_el[ji]*dev_el[ji];
	}
	
	//float btim = 1.0f - ftim;
	
	//float fa = (btim * dev_fb[ji] + ftim * dev_ff[ji] - 100.0f)/10.0f;
	
	
	
	//dev_sfa[ji] += fa;
	//dev_ssfa[ji] += fa*fa;
	

	//dev_sfel[ji] += dev_el[ji] * fa;
	
	
}

__global__ void dev_statistics_finalize(float nstat)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	int ji = j * dev_width + i;
	
	
	
	if (i < dev_width && j < dev_height)
	{
		//dev_sfa[ji] /= nstat;
		dev_sel[ji] /= nstat;
		
		//dev_sfar[ji] /= nstat;
		
		float prev_ssel = dev_ssel[ji];
		
		//dev_ssfa[ji] = (dev_ssfa[ji]/nstat - dev_sfa[ji]*dev_sfa[ji]) * 10000.0f;
		dev_ssel[ji] = (dev_ssel[ji]/nstat - dev_sel[ji]*dev_sel[ji]) * 10000.0f;
		
		
		if (dev_ssel[ji] >= 8570.0f)
		{
			printf("something wrong at i=%d, j=%d, with nstat=%f, sel=%f, prev=%f\n", i, j, nstat, dev_sel[ji], prev_ssel);
		}

	}
	
}



float * KaspyCycler::getElves()
{
	return g_el;
}

float * KaspyCycler::getSurface()
{
	return g_fsm;
}


void KaspyCycler::findElves()
{
	/// DO CUDA REDUCTION instead of copying back to host mem



	DeviceReduce::Min(d_temp_storage, temp_storage_bytes, g_elf, g_elf_r, F_DATA_SIZE);
	hipMemcpy(&m_fVars->elfmin, g_elf_r,  sizeof(float), hipMemcpyDeviceToHost);
	
	DeviceReduce::Max(d_temp_storage, temp_storage_bytes, g_elf, g_elf_r, F_DATA_SIZE);
	hipMemcpy(&m_fVars->elfmax, g_elf_r,  sizeof(float), hipMemcpyDeviceToHost);
	
	
}


void KaspyCycler::sendDataToGPU()
{
	//int ewidth = ((int)m_pitch) / sizeof(float);
	int wm1 = m_width - 1 ;
	int hm1 = m_height - 1 ;
	float dte = (float)m_fVars->dte;
	float dte2 = (float)m_fVars->dte * 2.0f;
	float tide_l = (float)m_fVars->tide_l;
	
	float aam2d = m_fArrays->aam2d;
	
	float xmi = m_fVars->xmi;
	float xma = m_fVars->xma;
	float ymi = m_fVars->ymi;
	float yma = m_fVars->yma;
	
	//float xki = m_fWindData->xki;
	//float xka = m_fWindData->xka;
	//float yki = m_fWindData->yki;
	//float yka = m_fWindData->xka;
	
	
	if ( (hipMemcpyToSymbol(HIP_SYMBOL(dev_width), &m_width, sizeof(int))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_height), &m_height, sizeof(int))  == hipSuccess)
		&&(hipMemcpyToSymbol(HIP_SYMBOL(dev_widthm1), &wm1, sizeof(int))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_heightm1), &hm1, sizeof(int))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_dte), &dte, sizeof(float))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_dte2), &dte2, sizeof(float))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_tide_l), &tide_l, sizeof(float))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_aam2d), &aam2d, sizeof(float))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_xmi), &xmi, sizeof(float))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_xma), &xma, sizeof(float))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ymi), &ymi, sizeof(float))  == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_yma), &yma, sizeof(float))  == hipSuccess)
		//&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_xki), &xki, sizeof(float))  == hipSuccess)
		//&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_xka), &xka, sizeof(float))  == hipSuccess)
		//&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_yki), &yki, sizeof(float))  == hipSuccess)
		//&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_yka), &yka, sizeof(float))  == hipSuccess)
		//&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ewidth), &ewidth,  sizeof(int))  == hipSuccess)
		)
	{
		printf("GPU constant memory filled\n");
		
		
		//int test_i = 0;
		//int test_f = 0;
		
		
	}
	else
	{
		printf("GPU memory copy error (error code %s)!\n", hipGetErrorString(hipGetLastError()));
		
		deinit_device();
		
		exit(-1);
	}
	
	
	size_t s_data_size =  m_height * m_width *  sizeof(float);

	size_t press_data_size =  m_fWindData->ky *  m_fWindData->kx * m_fWindData->kt * sizeof(float);
	size_t uwd_data_size =  m_fWindData->kyu *  m_fWindData->kxu * m_fWindData->ktu * sizeof(float);
	size_t vwd_data_size =  m_fWindData->kyv *  m_fWindData->kxv * m_fWindData->ktv * sizeof(float);
	
	//printf("have m_h ADDR as %llx\n", (unsigned long long)m_h);

	
	if ( (hipMemcpy(g_fbu,&m_fFloats->fbu[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fbv,&m_fFloats->fbv[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_ffu,&m_fFloats->ffu[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_ffv,&m_fFloats->ffv[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fb,&m_fFloats->fb[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_ff,&m_fFloats->ff[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fxb,&m_fFloats->fxb[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fxf,&m_fFloats->fxf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fyb,&m_fFloats->fyb[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fyf,&m_fFloats->fyf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_wusurf,&m_fArrays->wusurf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_wvsurf,&m_fArrays->wvsurf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_dum,&m_fArrays->dum[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_dvm,&m_fArrays->dvm[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_d, &m_fArrays->d[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		
		&& (hipMemcpy(g_fluxua,&m_fArrays->fluxua[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fluxva,&m_fArrays->fluxva[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		
		&& (hipMemcpy(g_ua,&m_fArrays->ua[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_va,&m_fArrays->va[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_uab,&m_fArrays->uab[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_vab,&m_fArrays->vab[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_uaf,&m_fArrays->uaf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_vaf,&m_fArrays->vaf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_el,&m_fArrays->el[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_elb,&m_fArrays->elb[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_elf,&m_fArrays->elf[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_fsm,&m_fArrays->fsm[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_tps,&m_fArrays->tps[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_advua,&m_fArrays->advua[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_advva,&m_fArrays->advva[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_wubot,&m_fArrays->wubot[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_wvbot,&m_fArrays->wvbot[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_cbc,&m_fArrays->cbc[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_h, &m_fArrays->h[0][0], s_data_size, hipMemcpyHostToDevice) == hipSuccess)
		
		&& (hipMemcpy(g_press,&m_press[0],press_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_uwd,&m_uwd[0],uwd_data_size, hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_vwd,&m_vwd[0],vwd_data_size, hipMemcpyHostToDevice) == hipSuccess)
		
		
		
		&& (hipMemcpy(g_cor, &m_fArrays->cor[0], m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_aru, &m_fArrays->aru[0], m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_arv, &m_fArrays->arv[0],  m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_art, &m_fArrays->art[0],  m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_dx, &m_fArrays->dx[0], m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_dy, &m_fArrays->dy[0], m_height * sizeof(float), hipMemcpyHostToDevice) == hipSuccess)
		
		&& (hipMemcpy(g_stations_x, m_stations_x, m_stations * sizeof(int), hipMemcpyHostToDevice) == hipSuccess)
		&& (hipMemcpy(g_stations_y, m_stations_y, m_stations * sizeof(int), hipMemcpyHostToDevice) == hipSuccess)
		)
		
	{
		printf("GPU memory filled\n");
	}
	else
	{
		printf("GPU memory copy error (error code %s)!\n", hipGetErrorString(hipGetLastError()));

		deinit_device();
		
		exit(-1);
	}
	
	
}

void KaspyCycler::writeStatistics()
{
	size_t s_data_size =  m_height * m_width *  sizeof(float);
	
	dim3 threadsPerSquareBlock(initValues->m_cuda_threads_2d_x, initValues->m_cuda_threads_2d_y);
	
	dim3 numSquareBlocks((m_width + threadsPerSquareBlock.x - 1) / threadsPerSquareBlock.x, (m_height + threadsPerSquareBlock.y - 1) / threadsPerSquareBlock.y);
	
	
	float * host_buf =  (float *) malloc(s_data_size );
	
	float * gpu_buf =  g_sel;
	
	const char * stat_filename = "sel.grd\0                           ";
	
	
	if (host_buf)
	{
		printf("have nstat %d!\n", m_nstat);
		dev_statistics_finalize<<< numSquareBlocks, threadsPerSquareBlock>>>((float)m_nstat);
		
		
		hipError_t err = hipMemcpy(host_buf, gpu_buf,  s_data_size, hipMemcpyDeviceToHost);
		
		if (err == hipSuccess)
		{
			// CALL WRITEGRD(IM,JM,IM,SSEL,xmi,xma,ymi,yma,NAME)
			
			WRITEGRD(&m_width, &m_height, &m_width, host_buf, &m_fVars->xmi, &m_fVars->xma,&m_fVars->ymi, &m_fVars->yma,stat_filename);

		}
		else
		{
			fprintf(stderr, "Failed to update statistics data  (error code %s)!\n", hipGetErrorString(err));
		}
		
		
		
		free(host_buf);
	}
	else
	{
		printf("memory allocation failed!\n");
		
		deinit_device();
		
		exit(-1);
	}
	
}

void KaspyCycler::getDataToCPU()
{
	float * h_el =  &m_fArrays->el[0][0];
	

	hipMemcpyFromSymbol(&g_el, HIP_SYMBOL(dev_el), sizeof(float *), 0,hipMemcpyDeviceToHost);
	
	
	hipError_t err = hipMemcpy(h_el, g_el,  m_height * m_width * sizeof(float), hipMemcpyDeviceToHost);
	
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to update host array EL  (error code %s)!\n", hipGetErrorString(err));
		
		deinit_device();
		
		exit(-1);
	}
	
	
	err = hipMemcpy(m_station_elves, g_station_elves,  (m_duration-1) * m_stations * sizeof(float), hipMemcpyDeviceToHost);
	
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to update station data  (error code %s)!\n", hipGetErrorString(err));
		
		deinit_device();
		
		exit(-1);
	}
}




void KaspyCycler::makeWsurf()
{
	//hipError_t err;
	
	float hours =  (m_duration - 1 )*m_fVars->dht;
	
	
	int n_iterations = int(hours * 3600.0/m_fVars->dte) + 2;
	
	
	int iold = 0;
	
	int ihour_s = int(600.0 / m_fVars->dte);
	
	float timeh;
	float timeh6;
	float ftim;
	int itimeh;
	int itime6;
	int itime6_old  = 0;
	
    int pressSize = m_fWindData->kx * m_fWindData->ky;
    int windUSize = m_fWindData->kxu * m_fWindData->kyu;
    int windVSize = m_fWindData->kxv * m_fWindData->kyv;
	
	
	int threadsPerBlock = initValues->m_cuda_threads_1d;
	
	int blocksPerGridJ = (m_height + threadsPerBlock - 1) / threadsPerBlock;
	int blocksPerGridI = (m_width + threadsPerBlock - 1) / threadsPerBlock;
	int blocksPerStations = (m_stations + threadsPerBlock - 1) / threadsPerBlock;
	
	dim3 threadsPerSquareBlock(initValues->m_cuda_threads_2d_x, initValues->m_cuda_threads_2d_y);
	
	dim3 numSquareBlocks((m_width + threadsPerSquareBlock.x - 1) / threadsPerSquareBlock.x, (m_height + threadsPerSquareBlock.y - 1) / threadsPerSquareBlock.y);
	
	
	
	for (int i=0; i<n_iterations; i++)
	{
		m_fVars->timeh = i * m_fVars->dte / 3600.0;
		
		m_fVars->timeh6 = (m_fVars->timeh / m_fVars->dht) + 1.0;
		
		timeh = (float)m_fVars->timeh;
		timeh6 = (float)m_fVars->timeh6;
		
		itimeh=int(timeh);
		
		ftim = fmodf(timeh6, 1.0f);
		
		itime6 = (int)timeh6;
		
		
		if ( (i % ihour_s) == 1)
		{
			findElves();
			printf("elves t=%f level=%f,%f \n", timeh, m_fVars->elfmin, m_fVars->elfmax);
		}
		
		
		if (itimeh > iold)
		{
			iold=itimeh;
		
			
			dev_fill_station_data<<< blocksPerStations, threadsPerBlock>>>(itimeh);
			
			/// STATISTICS HERE

			//
			dev_statistics_1<<< numSquareBlocks, threadsPerSquareBlock>>>(ftim);
			
			m_nstat++;
			
			//printf("STTT %d\n", m_nstat);
		}
		
		
		
		
		
		
		
		if (itime6 > itime6_old)
		{
			itime6_old = itime6;
			
			float * p_temp;
			
			p_temp = g_fxb;
			g_fxb = g_fxf;
			g_fxf = p_temp;
			
			p_temp = g_fyb;
			g_fyb = g_fyf;
			g_fyf = p_temp;
			
			p_temp = g_fb;
			g_fb = g_ff;
			g_ff = p_temp;
			
			p_temp = g_fbu;
			g_fbu = g_ffu;
			g_ffu = p_temp;
			
			p_temp = g_fbv;
			g_fbv = g_ffv;
			g_ffv = p_temp;
			
			hipMemcpyToSymbol(HIP_SYMBOL(dev_fxf), &g_fxf, sizeof(float *));
			hipMemcpyToSymbol(HIP_SYMBOL(dev_fxb), &g_fxb, sizeof(float *));
			hipMemcpyToSymbol(HIP_SYMBOL(dev_fyf), &g_fyf, sizeof(float *));
			hipMemcpyToSymbol(HIP_SYMBOL(dev_fyb), &g_fyb, sizeof(float *));
			hipMemcpyToSymbol(HIP_SYMBOL(dev_ff), &g_ff, sizeof(float *));
			hipMemcpyToSymbol(HIP_SYMBOL(dev_fb), &g_fb, sizeof(float *));
			hipMemcpyToSymbol(HIP_SYMBOL(dev_ffu), &g_ffu, sizeof(float *));
			hipMemcpyToSymbol(HIP_SYMBOL(dev_fbu), &g_fbu, sizeof(float *));
			hipMemcpyToSymbol(HIP_SYMBOL(dev_ffv), &g_ffv, sizeof(float *));
			hipMemcpyToSymbol(HIP_SYMBOL(dev_fbv), &g_fbv, sizeof(float *));
			
			
			
			g_press0 = g_press + (itime6 - 1) * pressSize;
			getWindPressure('p');
			
			g_uwd0 =  g_uwd + (itime6 - 1) * windUSize;
			getWindPressure('u');
			
			g_vwd0 = g_vwd + (itime6 - 1) * windVSize;
			getWindPressure('v');
			
			//printf("PPP\n");
		}
		
		
		surf_and_flux_1<<<numSquareBlocks, threadsPerSquareBlock>>>(ftim);
		
		/*err = hipGetLastError();
		
		if (err != hipSuccess)
		{
			printf("error calling surf_and_flux_1 kernel!  (error code %s)!\n", hipGetErrorString(err));
			return;
		}*/
		
		elf_and_flux_2<<<numSquareBlocks, threadsPerSquareBlock>>>();
		
		/*if (hipGetLastError() != hipSuccess)
		{
			printf("error calling elf_and_flux_2 kernel! \n");
		}*/
		
		/// BCOND 1
		
		
		bcond_1_j<<< blocksPerGridJ, threadsPerBlock>>>();
		
		/*if (hipGetLastError() != hipSuccess)
		{
			printf("error calling bcond_1_j kernel! \n");
		}*/
		
		bcond_1_i<<< blocksPerGridI, threadsPerBlock>>>();
		
		/*if (hipGetLastError() != hipSuccess)
		{
			printf("error calling bcond_1_i kernel! \n");
		}*/
		
		bcond_1_ji<<< numSquareBlocks, threadsPerSquareBlock>>>();
		
		/*if (hipGetLastError() != hipSuccess)
		{
			printf("error calling bcond_1_ji kernel! \n");
		}*/
		
		
		
		if (i % 10 == 9)
		{
			//ADVAVE()
			//       ADVUA=0 ?
			//		FLUXUA=0 ?
			
			
			adv_fluxes_1<<< numSquareBlocks, threadsPerSquareBlock>>>();
			
			/*if (hipGetLastError() != hipSuccess)
			{
				printf("error calling adv_fluxes_1 kernel! \n");
			}*/
			
			adv_advua_1<<< numSquareBlocks, threadsPerSquareBlock>>>();
			
			/*if (hipGetLastError() != hipSuccess)
			{
				printf("error calling adv_advua_1 kernel! \n");
			}*/
			
			adv_fluxes_2<<< numSquareBlocks, threadsPerSquareBlock>>>();
			
			if (hipGetLastError() != hipSuccess)
			{
				printf("error calling adv_fluxes_2 kernel! \n");
			}
			
			adv_advva_2<<< numSquareBlocks, threadsPerSquareBlock>>>();
			
			/*if (hipGetLastError() != hipSuccess)
			{
				printf("error calling adv_advva_2 kernel! \n");
			}*/
			
			
		 adv_bot_3<<< numSquareBlocks, threadsPerSquareBlock>>>();
			
			/*if (hipGetLastError() != hipSuccess)
			{
				printf("error calling adv_bot_3 kernel! \n");
			}*/
			
			
			// END ADVAVE();
		}
		
		
		uaf_and_vaf_3<<<numSquareBlocks, threadsPerSquareBlock>>>();
		
		/*if (hipGetLastError() != hipSuccess)
		{
			printf("error calling uaf_and_vaf_3 kernel! \n");
		}*/
		
		
	 bcond_2_j<<< blocksPerGridJ, threadsPerBlock>>>();
		
		/*if (hipGetLastError() != hipSuccess)
		{
			printf("error calling bcond_2_j kernel! \n");
		}*/
		
	 bcond_2_i<<< blocksPerGridI, threadsPerBlock>>>();
		
		/*if (hipGetLastError() != hipSuccess)
		{
			printf("error calling bcond_2_i kernel! \n");
		}*/
		
		bcond_2_ji<<< numSquareBlocks, threadsPerSquareBlock>>>();
		
		/*if (hipGetLastError() != hipSuccess)
		{
			printf("error calling bcond_2_ji kernel! \n");
		}*/
		
		tps_and_other_arrays_4<<<numSquareBlocks, threadsPerSquareBlock>>>();
		
		/*if (hipGetLastError() != hipSuccess)
		{
			printf("error calling tps_and_other_arrays_4 kernel! \n");
		}*/
		
		
		swap_arrays_5<<<1, 1>>>();
		
		/*if (hipGetLastError() != hipSuccess)
		{
			printf("error calling swap_arrays_5 kernel! \n");
		}*/

	}
	
	
	getDataToCPU();
	
	
	FILE * hnd = fopen(initValues->m_output_stations, "w");
	
	if (hnd!= NULL)
	{
		for (int i=1; i<m_duration; i++)
		{
			fprintf(hnd, "elves t=%d ", i);
			
			for (int k=0; k<m_stations; k++)
			{
				fprintf(hnd, " %9.3f ", m_station_elves[(i-1) * m_stations + k]  );
			}
						
			fprintf(hnd, " \n");
		}
		
		fclose(hnd);
	}
	
	

	
	

}


void KaspyCycler::getWindPressure(char uv)
{
	int kx, ky, kd, nx, ny;//, nd;

	float xki, xka, yki, yka, xmi, xma, ymi, yma;

	
	float * zero = 0;
	
	if (uv == 'u')
	{
		kx = m_fWindData->kxu;
		ky = m_fWindData->kyu;
		//pk = g_uwd0;
		
		xki = m_fWindData->xkui;
		xka = m_fWindData->xkua;
		yki = m_fWindData->ykui;
		yka = m_fWindData->ykua;
		
		hipMemcpyToSymbol(HIP_SYMBOL(dev_p), &g_ffu, sizeof(float *));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_pk), &g_uwd0, sizeof(float *));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_px), &zero, sizeof(float *));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_py), &zero, sizeof(float *));
		
		//p = g_ffu;
	}
	else if (uv == 'v')
	{
		kx = m_fWindData->kxv;
		ky = m_fWindData->kyv;
		//pk = g_vwd0;
		
		xki = m_fWindData->xkvi;
		xka = m_fWindData->xkva;
		yki = m_fWindData->ykvi;
		yka = m_fWindData->ykva;
		
		//p = g_ffv;
		hipMemcpyToSymbol(HIP_SYMBOL(dev_p), &g_ffv, sizeof(float *));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_pk), &g_vwd0, sizeof(float *));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_px), &zero, sizeof(float *));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_py), &zero, sizeof(float *));
		
	}
	else if (uv == 'p')
	{
		kx = m_fWindData->kx;
		ky = m_fWindData->ky;
		//float kd = kx;
		//pk = g_press0;
		xki = m_fWindData->xki;
		xka = m_fWindData->xka;
		yki = m_fWindData->yki;
		yka = m_fWindData->yka;

		hipMemcpyToSymbol(HIP_SYMBOL(dev_p), &g_ff, sizeof(float *));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_pk), &g_press0, sizeof(float *));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_px), &g_fxf, sizeof(float *));
		hipMemcpyToSymbol(HIP_SYMBOL(dev_py), &g_fyf, sizeof(float *));
		
		
		//p = g_ff;
		//px = g_fxf;
		//py = g_fyf;
	}
	else
	{
		// don't know what to do
		return;
	}
	
	kd = kx;
	
	nx = F_DATA_WIDTH;
	ny = F_DATA_HEIGHT;
	//nd = F_DATA_WIDTH;
	
	xmi = m_fVars->xmi;
	xma = m_fVars->xma;
	ymi = m_fVars->ymi;
	yma = m_fVars->yma;
	
	//float c1=3.1415926/180.0;
	//float c2=111111.0f;
	
	
	float dky=(yka-yki)/(ky-1.0f);
	float dkx=(xka-xki)/(kx-1.0f);
 
	float dy=(yma-ymi)/(ny-1.0f);
	float dx=(xma-xmi)/(nx-1.0f);
	

	
	int threadsPerBlock = initValues->m_cuda_threads_1d;

	
	dim3 threadsPerSquareBlock(initValues->m_cuda_threads_2d_x, initValues->m_cuda_threads_2d_y);
	
	
	dim3 numSquareBlocks((kx  + threadsPerSquareBlock.x ) / threadsPerSquareBlock.x, (ky  + threadsPerSquareBlock.y ) / threadsPerSquareBlock.y);
	
	
	dev_pkk_ij<<<numSquareBlocks, threadsPerSquareBlock>>>(kx, ky, kd);
	
	int blocksPerGridJ = (ky + threadsPerBlock) / threadsPerBlock;
	int blocksPerGridI = (kx + 1 + threadsPerBlock) / threadsPerBlock;
	
	dev_pkk_j<<<threadsPerBlock, blocksPerGridJ>>>(kx, ky);
	dev_pkk_i<<<threadsPerBlock, blocksPerGridI>>>(kx, ky);
	


	
	dim3 numNBlocks(((nx) + threadsPerSquareBlock.x - 1) / threadsPerSquareBlock.x, ((ny) + threadsPerSquareBlock.y - 1) / threadsPerSquareBlock.y);
	
	
	dev_bicubic<<<numNBlocks, threadsPerSquareBlock>>>(kx + 2, ky + 2, 50);

	
	dev_make_p<<<numNBlocks, threadsPerSquareBlock>>>(nx, ny, kx, ky, dx, dy, dkx, dky, xki, yki);
	


}








int KaspyCycler::init_device()
{
	int device_count = 0;
	
	size_t square_size = m_height*m_width * sizeof(float);
	
	//setbuf(stdout,NULL);
	//printf("\n\nIII device is %d\n\n", m_gpu_device);
	//setbuf(stdout,NULL);
	
	if (m_gpu_device >= 0)
	{
		// already initialized
		printf("CUDA device is already initiaized\n");
		
		return m_gpu_device;
	}
	
	
	hipGetDeviceCount(&device_count);
	
	for (int i = 0 ; i < device_count ; ++i)
	{
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, i);
		
		if (properties.major > 1 || (properties.major == 1 && properties.minor >= 1))
		{
			m_gpu_device = i;
			
			printf("Running on GPU %d (%s) \n",i ,properties.name);
			break;
		}
		else
		{
			printf("GPU %d (%s) does not support CUDA Dynamic Parallelism\n", i ,properties.name);
		}
	}
	
	
	if (m_gpu_device == -1)
	{
		printf("No suitable device found!\n");
		return m_gpu_device;
	}
	
	if (hipSetDevice(m_gpu_device) == hipSuccess)
	{
		printf("device set OK\n");
	}
	else
	{
		printf("unable to set device!\n");
		m_gpu_device = -1;
	}
	
	
	
	// Allocate GPU memory.
	if ( (hipMalloc((void **)&g_fbu, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_fbv, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_ffu, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_ffv, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_fb, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_ff, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_fxb, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_fxf, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_fyb, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_fyf, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_wusurf, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_wvsurf, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_dum, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_dvm, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_d, square_size) == hipSuccess)
		
		&& (hipMalloc((void **)&g_fluxua, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_fluxva, square_size) == hipSuccess)
		
		&& (hipMalloc((void **)&g_ua, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_va, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_uab, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_vab, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_uaf, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_vaf, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_el, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_elb, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_elf, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_elf_r, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_fsm, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_tps, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_advua, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_advva, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_wubot, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_wvbot, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_cbc, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_h, square_size) == hipSuccess)
		
		&& (hipMalloc((void **)&g_press,  m_fWindData->ky *  m_fWindData->kx * m_fWindData->kt * sizeof(float)) == hipSuccess)
		
		&& (hipMalloc((void **)&g_uwd, m_fWindData->kyu * m_fWindData->kxu * m_fWindData->ktu * sizeof(float)) == hipSuccess)
		&& (hipMalloc((void **)&g_vwd, m_fWindData->kyv * m_fWindData->kxv * m_fWindData->ktv * sizeof(float)) == hipSuccess)

		
		&& (hipMalloc((void **)&g_cor, m_height * sizeof(float)) == hipSuccess)
		&& (hipMalloc((void **)&g_aru, m_height * sizeof(float)) == hipSuccess)
		&& (hipMalloc((void **)&g_arv, m_height * sizeof(float)) == hipSuccess)
		&& (hipMalloc((void **)&g_art, m_height * sizeof(float)) == hipSuccess)
		
		&& (hipMalloc((void **)&g_dx, m_height * sizeof(float)) == hipSuccess)
		&& (hipMalloc((void **)&g_dy, m_height * sizeof(float)) == hipSuccess)
		
		&& (hipMalloc((void **)&g_stations_x, m_stations * sizeof(int)) == hipSuccess)
		&& (hipMalloc((void **)&g_stations_y, m_stations * sizeof(int)) == hipSuccess)
		&& (hipMalloc((void **)&g_station_elves, (m_duration -1) *  m_stations * sizeof(float)) == hipSuccess)
		
		&& (hipMalloc((void **)&g_sel, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_ssel, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_sfel, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_sfa, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_ssfa, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_sfar, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_ssfar, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_sfelr, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_su, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_sv, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_ssu, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_ssv, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_ssuv, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_ssue, square_size) == hipSuccess)
		&& (hipMalloc((void **)&g_ssve, square_size) == hipSuccess)
		
		
		)
	{
		printf("GPU memory allocated\n");
		
	}
	else
	{
		printf("GPU memory allocation error!\n");
		deinit_device();
		return m_gpu_device;
	}
	
	
	if ( (hipMemset (g_sel, 0, square_size) == hipSuccess)
		&& (hipMemset (g_ssel, 0, square_size) == hipSuccess)
		&& (hipMemset (g_sfel, 0, square_size) == hipSuccess)
		&& (hipMemset (g_sfa, 0, square_size) == hipSuccess)
		&& (hipMemset (g_ssfa, 0, square_size) == hipSuccess)
		&& (hipMemset (g_sfar, 0, square_size) == hipSuccess)
		&& (hipMemset (g_ssfar, 0, square_size) == hipSuccess)
		&& (hipMemset (g_sfelr, 0, square_size) == hipSuccess)
		&& (hipMemset (g_su, 0, square_size) == hipSuccess)
		&& (hipMemset (g_sv, 0, square_size) == hipSuccess)
		&& (hipMemset (g_ssu, 0, square_size) == hipSuccess)
		&& (hipMemset (g_ssv, 0, square_size) == hipSuccess)
		&& (hipMemset (g_ssuv, 0, square_size) == hipSuccess)
		&& (hipMemset (g_ssue, 0, square_size) == hipSuccess)
		&& (hipMemset (g_ssve, 0, square_size) == hipSuccess)
		
		
		)
	{
		printf("GPU memory zeroed\n");
		
	}
	else
	{
		printf("GPU memory fill error!\n");
		deinit_device();
		return m_gpu_device;
	}
	
	
	
	d_temp_storage = 0;
	temp_storage_bytes = 0;
	
	DeviceReduce::Min(d_temp_storage, temp_storage_bytes, g_elf, g_elf_r, F_DATA_SIZE);
	g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes);
	
	
	if ( (hipMemcpyToSymbol(HIP_SYMBOL(dev_fbu), &g_fbu, sizeof(g_fbu)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fbv), &g_fbv, sizeof(g_fbv)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ffu), &g_ffu, sizeof(g_ffu)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ffv), &g_ffv, sizeof(g_ffv)) == hipSuccess)
		
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fxb), &g_fxb, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fxf), &g_fxf, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fyb), &g_fyb, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fyf), &g_fyf, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fb), &g_fb, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ff), &g_ff, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_wusurf), &g_wusurf, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_wvsurf), &g_wvsurf, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_dum), &g_dum, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_dvm), &g_dvm, sizeof(float *)) == hipSuccess)

		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_d), &g_d, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_dx), &g_dx, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_dy), &g_dy, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fluxua), &g_fluxua, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fluxva), &g_fluxva, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ua), &g_ua, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_va), &g_va, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_uab), &g_uab, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_vab), &g_vab, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_uaf), &g_uaf, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_vaf), &g_vaf, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_el), &g_el, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_elf), &g_elf, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_elf_r), &g_elf_r, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_elb), &g_elb, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_fsm), &g_fsm, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_tps), &g_tps, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_advua), &g_advua, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_advva), &g_advva, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_aru), &g_aru, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_arv), &g_arv, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_art), &g_art, sizeof(float *)) == hipSuccess)

		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_wubot), &g_wubot, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_wvbot), &g_wvbot, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_cbc), &g_cbc, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_cor), &g_cor, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_h), &g_h, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_press), &g_press, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_uwd), &g_uwd, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_vwd), &g_vwd, sizeof(float *)) == hipSuccess)
		
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_stations_x), &g_stations_x, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_stations_y), &g_stations_y, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_station_elves), &g_station_elves, sizeof(float *)) == hipSuccess)
		
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_nstations), &m_stations, sizeof(int)) == hipSuccess)
		
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_sel), &g_sel, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ssel), &g_ssel, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_sfel), &g_sfel, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_sfa), &g_sfa, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ssfa), &g_ssfa, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_sfar), &g_sfar, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ssfar), &g_ssfar, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_sfelr), &g_sfelr, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_su), &g_su, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_sv), &g_sv, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ssu), &g_ssu, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ssv), &g_ssv, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ssuv), &g_ssuv, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ssue), &g_ssue, sizeof(float *)) == hipSuccess)
		&& (hipMemcpyToSymbol(HIP_SYMBOL(dev_ssve), &g_ssve, sizeof(float *)) == hipSuccess)

		
		
		)
	{
		printf("Device pointers initialized\n");
	}
	else
	{
		printf("Pointer init error!\n");
		deinit_device();
		return m_gpu_device;
	}
	
	
	
	return m_gpu_device;
}


void KaspyCycler::deinit_device()
{
	if (m_gpu_device >=0)
	{
		
		if (g_fbu)
		{
			hipFree(g_fbu);
		}
		
		if (g_fbv)
		{
			hipFree(g_fbv);
		}
		
		if (g_ffu)
		{
			hipFree(g_ffu);
		}
		
		if (g_ffv)
		{
			hipFree(g_ffv);
		}
		
		
		if (g_fb)
		{
			hipFree(g_fb);
		}
		
		if (g_ff)
		{
			hipFree(g_ff);
		}
		
		
		if (g_fxb)
		{
			hipFree(g_fxb);
		}
		
		if (g_fxf)
		{
			hipFree(g_fxf);
		}
		
		if (g_fyb)
		{
			hipFree(g_fyb);
		}
		
		if (g_fyf)
		{
			hipFree(g_fyf);
		}
		
		if (g_wusurf)
		{
			hipFree(g_wusurf);
		}
		
		if (g_wvsurf)
		{
			hipFree(g_wvsurf);
		}
		
		if (g_dum)
		{
			hipFree(g_dum);
		}
		
		if (g_dvm)
		{
			hipFree(g_dvm);
		}
		
		if (g_d)
		{
			hipFree(g_d);
		}
		
		if (g_fluxua)
		{
			hipFree(g_fluxua);
		}
		
		if (g_fluxva)
		{
			hipFree(g_fluxva);
		}
		
		if (g_ua)
		{
			hipFree(g_ua);
		}
		
		if (g_va)
		{
			hipFree(g_va);
		}
		
		if (g_uaf)
		{
			hipFree(g_uaf);
		}
		
		if (g_vaf)
		{
			hipFree(g_vaf);
		}
		
		if (g_uab)
		{
			hipFree(g_uab);
		}
		
		if (g_vab)
		{
			hipFree(g_vab);
		}
		
		if (g_el)
		{
			hipFree(g_el);
		}
		
		if (g_elb)
		{
			hipFree(g_elb);
		}
		
		if (g_elf)
		{
			hipFree(g_elf);
		}
		
		if (g_elf_r)
		{
			hipFree(g_elf_r);
		}
		
		if (g_fsm)
		{
			hipFree(g_fsm);
		}
		
		if (g_tps)
		{
			hipFree(g_tps);
		}
		
		if (g_advua)
		{
			hipFree(g_advua);
		}
		
		if (g_advva)
		{
			hipFree(g_advva);
		}
		
		if (g_aru)
		{
			hipFree(g_aru);
		}
		
		if (g_arv)
		{
			hipFree(g_arv);
		}
		
		if (g_art)
		{
			hipFree(g_art);
		}
		
		if (g_wubot)
		{
			hipFree(g_wubot);
		}
		
		if (g_wvbot)
		{
			hipFree(g_wvbot);
		}
		
		if (g_cbc)
		{
			hipFree(g_cbc);
		}
		
		if (g_h)
		{
			hipFree(g_h);
		}
		
		if (g_cor)
		{
			hipFree(g_cor);
		}
		
		if (g_press)
		{
			hipFree(g_press);
		}
		
		
		if (g_uwd)
		{
			hipFree(g_uwd);
		}
		
		if (g_vwd)
		{
			hipFree(g_vwd);
		}
		
		if (g_sel)
		{
			hipFree(g_sel);
		}
		
		
		if (g_ssel)
		{
			hipFree(g_ssel);
		}
		
		if (g_sfel)
		{
			hipFree(g_sfel);
		}
		
		
		if (g_sfa)
		{
			hipFree(g_sfa);
		}
		if (g_ssfa)
		{
			hipFree(g_ssfa);
		}
		if (g_sfar)
		{
			hipFree(g_sfar);
		}
		if (g_ssfar)
		{
			hipFree(g_ssfar);
		}
		if (g_sfelr)
		{
			hipFree(g_sfelr);
		}
		
		if (g_su)
		{
			hipFree(g_su);
		}
		if (g_sv)
		{
			hipFree(g_sv);
		}
		if (g_ssu)
		{
			hipFree(g_ssu);
		}
		if (g_ssv)
		{
			hipFree(g_ssv);
		}
		if (g_ssuv)
		{
			hipFree(g_ssuv);
		}
		if (g_ssue)
		{
			hipFree(g_ssue);
		}
		if (g_ssve)
		{
			hipFree(g_ssve);
		}
		
		if (d_temp_storage)
		{
			g_allocator.DeviceFree(d_temp_storage);
		}
		
		if (hipDeviceReset() == hipSuccess)
		{
			printf("GPU device reset ok\n");
		}
		
		m_gpu_device = -1;
	}
}




