#include "hip/hip_runtime.h"
//
//  KaspyCycler.cpp
//  kaspy_cuda
//
//  Created by Andrei Koulikov on 24.05.16.
//
//

#include "KaspyCycler.h"







void KaspyCycler::findElves()
{
    //printf("arrays is set to %llxd \n", (long long)m_fArrays );
    
    
    float * elves = &(m_fArrays->elf[0][0]);
    
    float elf_min = elves[0];
    float elf_max = elves[0];
    
    for (int i=1; i<F_DATA_SIZE; i++)
    {
        if (elves[i] > elf_max)
        {
            elf_max = elves[i];
        }
        
        if (elves[i] < elf_min)
        {
            elf_min = elves[i];
        }
    }
    
    //printf("C SAYS: time is %f, elf min is %f, elf max is %f \n",m_fVars->timeh, elf_min, elf_max);
}




/*
 DO J=2,JMM1
 DO I=2,IMM1
 uw=(btim*fbu(i,j)+ftim*ffu(i,j))
 vw=(btim*fbv(i,j)+ftim*ffv(i,j))
 speed=sqrt(uw**2+vw**2) !******************************************************
 !      speed=0
 windc=1.0e-3*(0.8+speed*0.065)*ro_ratio*speed
 WUSURF(I,J)=-windc*uw
 1 	*.25*(DUM(I,J+1)+DUM(I+1,J)+DUM(I-1,J)+DUM(I,J-1))+
 2  0.5*(d(i,j)+d(i-1,j))*(btim*FxB(i,j)+ftim*FxF(i,j))
 WVSURF(I,J)=-windc*vw
 1 	*.25*(DVM(I,J+1)+DVM(I+1,J)+DVM(I-1,J)+DVM(I,J-1))+
 2  0.5*(d(i,j)+d(i,j-1))*(btim*FyB(i,j)+ftim*FyF(i,j))
 end do
 end do
 */
void KaspyCycler::makeWsurf(float ro_ratio)
{
    float ftim = fmodf((float)m_fVars->timeh6, 1.0f);
    float btim = 1.0f - ftim;
    float uw, vw, speed, windc;
    
    int ji, jp1i, jip1, jim1, jm1i;
    
    float * g_fbu = &m_fFloats->fbu[0][0];
    float * g_fbv = &m_fFloats->fbv[0][0];
    float * g_ffu = &m_fFloats->ffu[0][0];
    float * g_ffv = &m_fFloats->ffv[0][0];

    float * g_fxb = &m_fFloats->fxb[0][0];
    float * g_fxf = &m_fFloats->fxf[0][0];
    float * g_fyb = &m_fFloats->fyb[0][0];
    float * g_fyf = &m_fFloats->fyf[0][0];
   
    
    float * g_wusurf = &m_fArrays->wusurf[0][0];
    float * g_wvsurf = &m_fArrays->wvsurf[0][0];
    
    float * g_dum = &m_fArrays->dum[0][0];
    float * g_dvm = &m_fArrays->dvm[0][0];
    
    float * g_d = &m_fArrays->d[0][0];
    
    
    for (int j=1; j<(m_height-1); j++ )
    {
        for (int i=1; i<(m_width-1); i++ )
        {
            ji = j * m_width + i;
            jp1i = ji + m_width;
            jip1 = ji + 1;
            jim1 = ji - 1;
            jm1i = ji - m_width;
            
            uw = btim * (g_fbu[ji]) + ftim * (g_ffu[ji]);
            vw = btim * (g_fbv[ji]) + ftim * (g_ffv[ji]);
            
            speed = sqrtf(uw*uw + vw*vw);
            windc = 0.001f * (0.8f + speed * 0.065f) * ro_ratio * speed;
            
            g_wusurf[ji] = -windc * uw *
            0.25f * (g_dum[jp1i]+g_dum[jip1]+g_dum[jim1]+g_dum[jm1i])
            + 0.5f * (g_d[ji] + g_d[jim1]) * (btim * g_fxb[ji] + ftim * g_fxf[ji]);
            
            g_wvsurf[ji] = -windc * vw *
            0.25f * (g_dvm[jp1i]+g_dvm[jip1]+g_dvm[jim1]+g_dvm[jm1i])
            + 0.5f * (g_d[ji] + g_d[jm1i]) * (btim * g_fyb[ji] + ftim * g_fyf[ji]);
        }
    }
}
