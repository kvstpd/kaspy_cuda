#include "hip/hip_runtime.h"
//
//  KaspyCycler.cpp
//  kaspy_cuda
//
//  Created by Andrei Koulikov on 24.05.16.
//
//

#include "KaspyCycler.h"







void KaspyCycler::findElves()
{
    //printf("arrays is set to %llxd \n", (long long)m_fArrays );
    
    
    float * elves = &(m_fArrays->elf[0][0]);
    
    float elf_min = elves[0];
    float elf_max = elves[0];
    
    for (int i=1; i<F_DATA_SIZE; i++)
    {
        if (elves[i] > elf_max)
        {
            elf_max = elves[i];
        }
        
        if (elves[i] < elf_min)
        {
            elf_min = elves[i];
        }
    }
    
    //printf("C SAYS: time is %f, elf min is %f, elf max is %f \n",m_fVars->timeh, elf_min, elf_max);
}



void KaspyCycler::loadData()
{
    g_fbu = &m_fFloats->fbu[0][0];
    g_fbv = &m_fFloats->fbv[0][0];
    g_ffu = &m_fFloats->ffu[0][0];
    g_ffv = &m_fFloats->ffv[0][0];
    
    g_fxb = &m_fFloats->fxb[0][0];
    g_fxf = &m_fFloats->fxf[0][0];
    g_fyb = &m_fFloats->fyb[0][0];
    g_fyf = &m_fFloats->fyf[0][0];

    
    g_fb = &m_fFloats->fb[0][0];
    g_ff = &m_fFloats->ff[0][0];
   
    
    g_wusurf = &m_fArrays->wusurf[0][0];
    g_wvsurf = &m_fArrays->wvsurf[0][0];
    
    g_dum = &m_fArrays->dum[0][0];
    g_dvm = &m_fArrays->dvm[0][0];
    
    g_d = &m_fArrays->d[0][0];
}




/*
 DO J=2,JMM1
 DO I=2,IMM1
 uw=(btim*fbu(i,j)+ftim*ffu(i,j))
 vw=(btim*fbv(i,j)+ftim*ffv(i,j))
 speed=sqrt(uw**2+vw**2) !******************************************************
 !      speed=0
 windc=1.0e-3*(0.8+speed*0.065)*ro_ratio*speed
 WUSURF(I,J)=-windc*uw
 1 	*.25*(DUM(I,J+1)+DUM(I+1,J)+DUM(I-1,J)+DUM(I,J-1))+
 2  0.5*(d(i,j)+d(i-1,j))*(btim*FxB(i,j)+ftim*FxF(i,j))
 WVSURF(I,J)=-windc*vw
 1 	*.25*(DVM(I,J+1)+DVM(I+1,J)+DVM(I-1,J)+DVM(I,J-1))+
 2  0.5*(d(i,j)+d(i,j-1))*(btim*FyB(i,j)+ftim*FyF(i,j))
 end do
 end do
 */
void KaspyCycler::makeWsurf(float ro_ratio)
{
    m_fVars->timeh6 = (m_fVars->timeh / m_fVars->dht) + 1.0f;

    float timeh6 = m_fVars->timeh6;
    
    int pressSize = m_fWindData->kx * m_fWindData->ky;
    int windUSize = m_fWindData->kxu * m_fWindData->kyu;
    int windVSize = m_fWindData->kxv * m_fWindData->kyv;
    
    itime6 = (int)timeh6;

    ftim = (timeh6 - itime6);
    btim = 1.0f - ftim;
    
    if (itime6 > itime6_old)
    {
        itime6_old = itime6;
        
        memcpy(g_fxb, g_fxf, F_DATA_SIZE * sizeof(float));
        memcpy(g_fyb, g_fyf, F_DATA_SIZE * sizeof(float));
        memcpy(g_fb, g_ff, F_DATA_SIZE * sizeof(float));
        memcpy(g_fbu, g_ffu, F_DATA_SIZE * sizeof(float));
        memcpy(g_fbv, g_ffv, F_DATA_SIZE * sizeof(float));

        setbuf(stdout,NULL);
        
        //printf("press size is %d\n", pressSize );
        
        //printf("press 000 is %f press0 00 is %f\n", m_press[0], m_press0[0]);

        //printf("copy pressure from %#018llx to %#018llx\n", m_press, m_press0);

        memcpy(m_press0, m_press + (itime6 - 1) * pressSize, pressSize * sizeof(float));
        memcpy(m_uwd0, m_uwd + (itime6 - 1) * windUSize, windUSize * sizeof(float));
        memcpy(m_vwd0, m_vwd + (itime6 - 1) * windVSize, windVSize * sizeof(float));
        
    }
    
        /*press0(:,:)=press(:,:,itime6)
        call getnewpressureVAR(kx,ky,XKI,XKA,YKI,YKA,PRESS0,
                               1 FF,fxf,fyf)
        uwd0(:,:)=uwd(:,:,itime6)
        call getnewwindVAR(kxu,kyu,XKUI,XKUA,YKUI,YKUA,uwd0,ffu)
        vwd0(:,:)=vwd(:,:,itime6)
        call getnewwindVAR(kxv,kyv,XKVI,XKVA,YKVI,YKVA,vwd0,ffv)*/


            
    float uw, vw, speed, windc;
    int ji, jp1i, jip1, jim1, jm1i;

    
    
    
    ftim = fmodf((float)m_fVars->timeh6, 1.0f);
    btim = 1.0f - ftim;
    
    for (int j=1; j<(m_height-1); j++ )
    {
        for (int i=1; i<(m_width-1); i++ )
        {
            ji = j * m_width + i;
            jp1i = ji + m_width;
            jip1 = ji + 1;
            jim1 = ji - 1;
            jm1i = ji - m_width;
            
            uw = btim * (g_fbu[ji]) + ftim * (g_ffu[ji]);
            vw = btim * (g_fbv[ji]) + ftim * (g_ffv[ji]);
            
            speed = sqrtf(uw*uw + vw*vw);
            windc = 0.001f * (0.8f + speed * 0.065f) * ro_ratio * speed;
            
            g_wusurf[ji] = -windc * uw *
            0.25f * (g_dum[jp1i]+g_dum[jip1]+g_dum[jim1]+g_dum[jm1i])
            + 0.5f * (g_d[ji] + g_d[jim1]) * (btim * g_fxb[ji] + ftim * g_fxf[ji]);
            
            g_wvsurf[ji] = -windc * vw *
            0.25f * (g_dvm[jp1i]+g_dvm[jip1]+g_dvm[jim1]+g_dvm[jm1i])
            + 0.5f * (g_d[ji] + g_d[jm1i]) * (btim * g_fyb[ji] + ftim * g_fyf[ji]);
        }
    }
}
